#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 2 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/types.h>
#include <unistd.h>

#include "config.h"
#include "mesh_constants_cuda.h"



/* ----------------------------------------------------------------------------------------------- */

// ELASTIC simulations

/* ----------------------------------------------------------------------------------------------- */

//fortran code snippet...
/*
  ! gets global number of that receiver
  irec = number_receiver_global(irec_local)

  ! gets local receiver interpolators
  ! (1-D Lagrange interpolators)
  hxir(:) = hxir_store(irec_local,:)
  hetar(:) = hetar_store(irec_local,:)
  hgammar(:) = hgammar_store(irec_local,:)
*/

/* ----------------------------------------------------------------------------------------------- */

// unused...
/*
__device__ double my_atomicAdd(double* address, double val) {

    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do{
      assumed = old;
      old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
*/

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_elastic_seismogram_kernel(int nrec_local,
                                                         realw* field,
                                                         int* d_ibool,
                                                         realw* hxir, realw* hgammar,
                                                         realw* seismograms,
                                                         realw* cosrot,
                                                         realw* sinrot,
                                                         int* number_receiver_global,
                                                         int* ispec_selected_rec) {


  int irec_local = blockIdx.x + blockIdx.y*gridDim.x;
  int tx = threadIdx.x;
  int J = (tx/NGLLX);
  int I = (tx-J*NGLLX);

  __shared__ realw sh_dxd[NGLL2_PADDED];
  __shared__ realw sh_dzd[NGLL2_PADDED];


  if (irec_local < nrec_local) {

    int irec = number_receiver_global[irec_local]-1;
    int ispec = ispec_selected_rec[irec]-1;

   sh_dxd[tx] = 0;
   sh_dzd[tx] = 0;


  if (tx < NGLL2) {

    int iglob = d_ibool[tx+NGLL2_PADDED*ispec]-1;

    realw hlagrange = hxir[irec_local + nrec_local*I]*hgammar[irec_local + nrec_local*J];
    sh_dxd[tx] = hlagrange*field[0+2*iglob];
    sh_dzd[tx] = hlagrange*field[1+2*iglob];
    __syncthreads();}

for (unsigned int s=1; s<NGLL2_PADDED ; s *= 2) {
  if (tx % (2*s) == 0){ sh_dxd[tx] += sh_dxd[tx + s];sh_dzd[tx] += sh_dzd[tx + s];}
  __syncthreads();
}

  if (tx == 0) {seismograms[irec_local] = cosrot[irec_local]*sh_dxd[0]  + sinrot[irec_local]*sh_dzd[0];}
  if (tx == 1) {seismograms[irec_local+nrec_local] = cosrot[irec_local]*sh_dzd[0]  - sinrot[irec_local]*sh_dxd[0];}
}

}
/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_acoustic_seismogram_kernel(int nrec_local,
                                                         realw* pressure,
                                                         int* d_ibool,
                                                         realw* hxir, realw* hgammar,
                                                         realw* seismograms,
                                                         int* number_receiver_global,
                                                         int* ispec_selected_rec) {
  int irec_local = blockIdx.x + blockIdx.y*gridDim.x;
  int tx = threadIdx.x;
  int J = (tx/NGLLX);
  int I = (tx-J*NGLLX);

  __shared__ realw sh_dxd[NGLL2_PADDED];



  if (irec_local < nrec_local) {

    int irec = number_receiver_global[irec_local]-1;
    int ispec = ispec_selected_rec[irec]-1;

   sh_dxd[tx] = 0;
realw hlagrange;
int iglob;
  if (tx < NGLL2) {

    iglob = d_ibool[tx+NGLL2_PADDED*ispec]-1;

    hlagrange = hxir[irec_local + nrec_local*I]*hgammar[irec_local + nrec_local*J];
    sh_dxd[tx] = hlagrange*pressure[iglob];
    __syncthreads();}

for (unsigned int s=1; s<NGLL2_PADDED ; s *= 2) {
  if (tx % (2*s) == 0) sh_dxd[tx] += sh_dxd[tx + s];
  __syncthreads();}


// Signe moins car pression = -minus_pressure
  if (tx == 0) {seismograms[irec_local] = -sh_dxd[0];}
  if (tx == 1) {seismograms[irec_local+nrec_local] = 0;}

    }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_seismograms_cuda,
              COMPUTE_SEISMOGRAMS_CUDA)(long* Mesh_pointer_f,
                                        int* seismotypef,
                                        double* sisux, double* sisuz,
                                        int* seismo_currentf,
                                        int* NSTEP_BETWEEN_OUTPUT_SEISMOSf,
                                        int * ELASTIC_SIMULATION,int * ACOUSTIC_SIMULATION,
                                        int* USE_TRICK_FOR_BETTER_PRESSURE) {

// compute_seismograms
  TRACE("\tcompute_seismograms");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); // get Mesh from fortran integer wrapper
  int seismotype = *seismotypef;
  int seismo_current = *seismo_currentf - 1 ;
  int NSTEP_BETWEEN_OUTPUT_SEISMOS = *NSTEP_BETWEEN_OUTPUT_SEISMOSf;
  int i;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->nrec_local,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(NGLL2_PADDED,1,1);

  switch (seismotype){

  case 1 :  //Deplacement
  if (! *ELASTIC_SIMULATION) printf("\nWrong type of seismogram for a pure fluid simulation, use pressure in seismotype\n");

  compute_elastic_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(       mp->nrec_local,
                                                                                  mp->d_displ,
                                                                                  mp->d_ibool,
                                                                                  mp->d_xir_store_loc, mp->d_gammar_store_loc,
                                                                                  mp->d_seismograms,
                                                                                  mp->d_cosrot,
                                                                                  mp->d_sinrot,
                                                                                  mp->d_number_receiver_global,
                                                                                  mp->d_ispec_selected_rec
                                                                                  );

  break;

  case 2 :  //Vitesse
  if (! *ELASTIC_SIMULATION) printf("\nWrong type of seismogram for a pure fluid simulation, use pressure in seismotype\n");

  compute_elastic_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(       mp->nrec_local,
                                                                                  mp->d_veloc,
                                                                                  mp->d_ibool,
                                                                                  mp->d_xir_store_loc, mp->d_gammar_store_loc,
                                                                                  mp->d_seismograms,
                                                                                  mp->d_cosrot,
                                                                                  mp->d_sinrot,
                                                                                  mp->d_number_receiver_global,
                                                                                  mp->d_ispec_selected_rec
                                                                                  );
  break;

  case 3 :  //Acceleration
  if (! *ELASTIC_SIMULATION) printf("\nWrong type of seismogram for a pure fluid simulation, use pressure in seismotype\n");

  compute_elastic_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(       mp->nrec_local,
                                                                                  mp->d_accel,
                                                                                  mp->d_ibool,
                                                                                  mp->d_xir_store_loc, mp->d_gammar_store_loc,
                                                                                  mp->d_seismograms,
                                                                                  mp->d_cosrot,
                                                                                  mp->d_sinrot,
                                                                                  mp->d_number_receiver_global,
                                                                                  mp->d_ispec_selected_rec
                                                                                  );
  break;

  case 4 :  //Pression
  if (! *ACOUSTIC_SIMULATION) printf("\nWrong type of seismogram for a pure elasticsimulation, use displ veloc or accel in seismotype\n");

  if (*USE_TRICK_FOR_BETTER_PRESSURE)
  compute_acoustic_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(      mp->nrec_local,
                                                                                  mp->d_minus_int_int_pressure_acoustic,
                                                                                  mp->d_ibool,
                                                                                  mp->d_xir_store_loc, mp->d_gammar_store_loc,
                                                                                  mp->d_seismograms,
                                                                                  mp->d_number_receiver_global,
                                                                                  mp->d_ispec_selected_rec
                                                                                  );
  else
  compute_acoustic_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(      mp->nrec_local,
                                                                                  mp->d_minus_pressure_acoustic,
                                                                                  mp->d_ibool,
                                                                                  mp->d_xir_store_loc, mp->d_gammar_store_loc,
                                                                                  mp->d_seismograms,
                                                                                  mp->d_number_receiver_global,
                                                                                  mp->d_ispec_selected_rec
                                                                                  );

  break;
  }

  int size = mp->nrec_local;

  // (hipMemcpy implicitly synchronizes all other cuda operations)
  print_CUDA_error_if_any(hipMemcpy(mp->h_seismograms,mp->d_seismograms,
                                    sizeof(realw)*2* size,hipMemcpyDeviceToHost),72001);

  for (i=0;i<size;i++)
   { sisux[seismo_current + NSTEP_BETWEEN_OUTPUT_SEISMOS * i ] = (double)*(mp->h_seismograms+i);
     sisuz[seismo_current + NSTEP_BETWEEN_OUTPUT_SEISMOS * i ] = (double)*(mp->h_seismograms+i+size);
   }

}
