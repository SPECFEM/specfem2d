#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This software is governed by the CeCILL license under French law and
! abiding by the rules of distribution of free software. You can use,
! modify and/or redistribute the software under the terms of the CeCILL
! license as circulated by CEA, CNRS and Inria at the following URL
! "http://www.cecill.info".
!
! As a counterpart to the access to the source code and rights to copy,
! modify and redistribute granted by the license, users are provided only
! with a limited warranty and the software's author, the holder of the
! economic rights, and the successive licensors have only limited
! liability.
!
! In this respect, the user's attention is drawn to the risks associated
! with loading, using, modifying and/or developing or reproducing the
! software by the user in light of its specific status of free software,
! that may mean that it is complicated to manipulate, and that also
! therefore means that it is reserved for developers and experienced
! professionals having in-depth computer knowledge. Users are therefore
! encouraged to load and test the software's suitability as regards their
! requirements in conditions enabling the security of their systems and/or
! data to be ensured and, more generally, to use and operate it in the
! same conditions as regards security.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================

*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/types.h>
#include <unistd.h>

#include "config.h"
#include "mesh_constants_cuda.h"



/* ----------------------------------------------------------------------------------------------- */

// ELASTIC simulations

/* ----------------------------------------------------------------------------------------------- */

//fortran code snippet...
/*
  ! gets global number of that receiver
  irec = number_receiver_global(irec_local)

  ! gets local receiver interpolators
  ! (1-D Lagrange interpolators)
  hxir(:) = hxir_store(irec_local,:)
  hetar(:) = hetar_store(irec_local,:)
  hgammar(:) = hgammar_store(irec_local,:)
*/

/* ----------------------------------------------------------------------------------------------- */

// unused...
/*
__device__ double my_atomicAdd(double* address, double val) {

    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do{
      assumed = old;
      old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
*/

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_elastic_seismogram_kernel(int nrec_local,
                                                         realw* field,
                                                         int* d_ibool,
                                                         realw* hxir, realw* hgammar,
                                                         realw* seismograms,
                                                         realw* cosrot,
                                                         realw* sinrot,
                                                         int* number_receiver_global,
                                                         int* ispec_selected_rec) {


  int irec_local = blockIdx.x + blockIdx.y*gridDim.x;
  int tx = threadIdx.x;
  int J = (tx/NGLLX);
  int I = (tx-J*NGLLX);

  __shared__ realw sh_dxd[NGLL2_PADDED];
  __shared__ realw sh_dzd[NGLL2_PADDED];


  if(irec_local < nrec_local) {

    int irec = number_receiver_global[irec_local]-1;
    int ispec = ispec_selected_rec[irec]-1;

   sh_dxd[tx] = 0;
   sh_dzd[tx] = 0;


  if ( tx < NGLL2 ){

    int iglob = d_ibool[tx+NGLL2_PADDED*ispec]-1;

    realw hlagrange = hxir[irec_local + nrec_local*I]*hgammar[irec_local + nrec_local*J];
    sh_dxd[tx] = hlagrange*field[0+2*iglob];
    sh_dzd[tx] = hlagrange*field[1+2*iglob];
    __syncthreads();}

for (unsigned int s=1; s<NGLL2_PADDED ; s *= 2) {
  if(tx % (2*s) == 0){ sh_dxd[tx] += sh_dxd[tx + s];sh_dzd[tx] += sh_dzd[tx + s];}
  __syncthreads();
}

  if ( tx == 0 ){seismograms[irec_local] = cosrot[irec_local]*sh_dxd[0]  + sinrot[irec_local]*sh_dzd[0];}
  if ( tx == 1 ){seismograms[irec_local+nrec_local] = cosrot[irec_local]*sh_dzd[0]  - sinrot[irec_local]*sh_dxd[0];}
}

}
/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_acoustic_seismogram_kernel(int nrec_local,
                                                         realw* pressure,
                                                         int* d_ibool,
                                                         realw* hxir, realw* hgammar,
                                                         realw* seismograms,
                                                         int* number_receiver_global,
                                                         int* ispec_selected_rec) {
  int irec_local = blockIdx.x + blockIdx.y*gridDim.x;
  int tx = threadIdx.x;
  int J = (tx/NGLLX);
  int I = (tx-J*NGLLX);

  __shared__ realw sh_dxd[NGLL2_PADDED];



  if(irec_local < nrec_local) {

    int irec = number_receiver_global[irec_local]-1;
    int ispec = ispec_selected_rec[irec]-1;

   sh_dxd[tx] = 0;
realw hlagrange;
int iglob;
  if ( tx < NGLL2 ){

    iglob = d_ibool[tx+NGLL2_PADDED*ispec]-1;

    hlagrange = hxir[irec_local + nrec_local*I]*hgammar[irec_local + nrec_local*J];
    sh_dxd[tx] = hlagrange*pressure[iglob];
    __syncthreads();}

for (unsigned int s=1; s<NGLL2_PADDED ; s *= 2) {
  if(tx % (2*s) == 0) sh_dxd[tx] += sh_dxd[tx + s];
  __syncthreads();}


// Signe moins car pression = -potential_dot_dot
  if ( tx == 0 ){seismograms[irec_local] = -sh_dxd[0];}
  if ( tx == 1 ){seismograms[irec_local+nrec_local] = 0;}

    }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_seismograms_cuda,
              COMPUTE_SEISMOGRAMS_CUDA)(long* Mesh_pointer_f,int* seismotypef,double* sisux, double* sisuz,int* seismo_currentf,
                                   int* NSTEP_BETWEEN_OUTPUT_SEISMOSf,int * any_elastic_glob,int * any_acoustic_glob) {

// compute_seismograms
  TRACE("\tcompute_seismograms");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); // get Mesh from fortran integer wrapper
  int seismotype = *seismotypef;
  int seismo_current = *seismo_currentf - 1 ;
  int NSTEP_BETWEEN_OUTPUT_SEISMOS =*NSTEP_BETWEEN_OUTPUT_SEISMOSf;
  int i;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->nrec_local,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(NGLL2_PADDED,1,1);


 switch (seismotype){


  case 1 :  //Deplacement

  if (! *any_elastic_glob) printf("\nWrong type of seismogram for a pure fluid simulation, use pressure in seismotype\n");
  compute_elastic_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(       mp->nrec_local,
                                                                                  mp->d_displ,
                                                                                  mp->d_ibool,
                                                                                  mp->d_xir_store_loc, mp->d_gammar_store_loc,
                                                                                  mp->d_seismograms,
                                                                                  mp->d_cosrot,
                                                                                  mp->d_sinrot,
                                                                                  mp->d_number_receiver_global,
                                                                                  mp->d_ispec_selected_rec
                                                                                  );

  break;

  case 2 :  //Vitesse
  if (! *any_elastic_glob) printf("\nWrong type of seismogram for a pure fluid simulation, use pressure in seismotype\n");
  compute_elastic_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(       mp->nrec_local,
                                                                                  mp->d_veloc,
                                                                                  mp->d_ibool,
                                                                                  mp->d_xir_store_loc, mp->d_gammar_store_loc,
                                                                                  mp->d_seismograms,
                                                                                  mp->d_cosrot,
                                                                                  mp->d_sinrot,
                                                                                  mp->d_number_receiver_global,
                                                                                  mp->d_ispec_selected_rec
                                                                                  );
  break;

  case 3 :  //Acceleration
  if (! *any_elastic_glob) printf("\nWrong type of seismogram for a pure fluid simulation, use pressure in seismotype\n");
  compute_elastic_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(       mp->nrec_local,
                                                                                  mp->d_accel,
                                                                                  mp->d_ibool,
                                                                                  mp->d_xir_store_loc, mp->d_gammar_store_loc,
                                                                                  mp->d_seismograms,
                                                                                  mp->d_cosrot,
                                                                                  mp->d_sinrot,
                                                                                  mp->d_number_receiver_global,
                                                                                  mp->d_ispec_selected_rec
                                                                                  );
  break;

 case 4 :  //Pression

  if (! *any_acoustic_glob) printf("\nWrong type of seismogram for a pure elasticsimulation, use displ veloc or accel in seismotype\n");
  compute_acoustic_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(      mp->nrec_local,
                                                                                  mp->d_potential_dot_dot_acoustic,
                                                                                  mp->d_ibool,
                                                                                  mp->d_xir_store_loc, mp->d_gammar_store_loc,
                                                                                  mp->d_seismograms,
                                                                                  mp->d_number_receiver_global,
                                                                                  mp->d_ispec_selected_rec
                                                                                  );


  break;



}



 int size = mp->nrec_local;


  // (hipMemcpy implicitly synchronizes all other cuda operations)
  print_CUDA_error_if_any(hipMemcpy(mp->h_seismograms,mp->d_seismograms,
                                    sizeof(realw)*2* size,hipMemcpyDeviceToHost),72001);




  for (i=0;i<size;i++)
   { sisux[seismo_current + NSTEP_BETWEEN_OUTPUT_SEISMOS * i ] = (double)*(mp->h_seismograms+i);
     sisuz[seismo_current + NSTEP_BETWEEN_OUTPUT_SEISMOS * i ] = (double)*(mp->h_seismograms+i+size);
   }



}
