#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This software is governed by the CeCILL license under French law and
! abiding by the rules of distribution of free software. You can use,
! modify and/or redistribute the software under the terms of the CeCILL
! license as circulated by CEA, CNRS and Inria at the following URL
! "http://www.cecill.info".
!
! As a counterpart to the access to the source code and rights to copy,
! modify and redistribute granted by the license, users are provided only
! with a limited warranty and the software's author, the holder of the
! economic rights, and the successive licensors have only limited
! liability.
!
! In this respect, the user's attention is drawn to the risks associated
! with loading, using, modifying and/or developing or reproducing the
! software by the user in light of its specific status of free software,
! that may mean that it is complicated to manipulate, and that also
! therefore means that it is reserved for developers and experienced
! professionals having in-depth computer knowledge. Users are therefore
! encouraged to load and test the software's suitability as regards their
! requirements in conditions enabling the security of their systems and/or
! data to be ensured and, more generally, to use and operate it in the
! same conditions as regards security.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================

*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

// ASSEMBLY - mpi data transfer between CPU-GPU

/* ----------------------------------------------------------------------------------------------- */

// prepares a device array with with all inter-element edge-nodes -- this
// is followed by a memcpy and MPI operations

__global__ void prepare_boundary_accel_on_device(realw* d_accel, realw* d_send_accel_buffer,
                                                 const int ninterface_el,
                                                 const int max_nibool_interfaces_ext_mesh,
                                                 const int* d_nibool_interfaces_ext_mesh,
                                                 const int* d_ibool_interfaces_ext_mesh,
                                                 const int* inum_inter_elastic) {

  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;
  int ientry,iglob,num_int;

  for( int iinterface=0; iinterface < ninterface_el; iinterface++) {

     num_int=inum_inter_elastic[iinterface]-1;

      if( id < d_nibool_interfaces_ext_mesh[num_int] ) {


      // entry in interface array
      ientry = id + max_nibool_interfaces_ext_mesh*num_int;
      // global index in wavefield
      iglob = d_ibool_interfaces_ext_mesh[ientry] - 1;

      d_send_accel_buffer[2*ientry] = d_accel[2*iglob];
      d_send_accel_buffer[2*ientry + 1 ] = d_accel[2*iglob + 1];

    }
  }

}

/* ----------------------------------------------------------------------------------------------- */

// prepares and transfers the inter-element edge-nodes to the host to be MPI'd
// (elements on boundary)
extern "C"
void FC_FUNC_(transfer_boun_accel_from_device,
              TRANSFER_BOUN_ACCEL_FROM_DEVICE)(long* Mesh_pointer,
                                               realw* send_accel_buffer,
                                               const int* FORWARD_OR_ADJOINT){
TRACE("\ttransfer_boun_accel_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // checks if anything to do
  if( mp->size_mpi_buffer > 0 ){

    int blocksize = BLOCKSIZE_TRANSFER;
    int size_padded = ((int)ceil(((double)mp->max_nibool_interfaces_ext_mesh)/((double)blocksize)))*blocksize;

    int num_blocks_x, num_blocks_y;
    get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

    dim3 grid(num_blocks_x,num_blocks_y);
    dim3 threads(blocksize,1,1);

    // Cuda timing
    //hipEvent_t start, stop;
    //start_timing_cuda(&start,&stop);

    if(*FORWARD_OR_ADJOINT == 1) {

      prepare_boundary_accel_on_device<<<grid,threads,0,mp->compute_stream>>>(mp->d_accel,mp->d_send_accel_buffer,
                                                                              mp->ninterface_elastic,
                                                                              mp->max_nibool_interfaces_ext_mesh,
                                                                              mp->d_nibool_interfaces_ext_mesh,
                                                                              mp->d_ibool_interfaces_ext_mesh,
                                                                              mp->d_inum_interfaces_elastic);


      // synchronizes
      //synchronize_cuda();
      // explicitly waits until previous compute stream finishes
      // (hipMemcpy implicitly synchronizes all other cuda operations)
      hipStreamSynchronize(mp->compute_stream);

      // copies buffer from GPU to CPU host
      print_CUDA_error_if_any(hipMemcpy(send_accel_buffer,mp->d_send_accel_buffer,
                              mp->size_mpi_buffer*sizeof(realw),hipMemcpyDeviceToHost),97001);

    }
    else if(*FORWARD_OR_ADJOINT == 3) {
      prepare_boundary_accel_on_device<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_accel,mp->d_b_send_accel_buffer,
                                                                              mp->ninterface_elastic,
                                                                              mp->max_nibool_interfaces_ext_mesh,
                                                                              mp->d_nibool_interfaces_ext_mesh,
                                                                              mp->d_ibool_interfaces_ext_mesh,
                                                                              mp->d_inum_interfaces_elastic);
      // synchronizes
      //synchronize_cuda();
      // explicitly waits until previous compute stream finishes
      // (hipMemcpy implicitly synchronizes all other cuda operations)
      hipStreamSynchronize(mp->compute_stream);

      // copies buffer from GPU to CPU host
      print_CUDA_error_if_any(hipMemcpy(send_accel_buffer,mp->d_b_send_accel_buffer,
                              mp->size_mpi_buffer*sizeof(realw),hipMemcpyDeviceToHost),97002);
    }

    // Cuda timing
    // finish timing of kernel+memcpy
    //stop_timing_cuda(&start,&stop,"prepare_boundary_accel_on_device");
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("transfer_boun_accel_from_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_boundary_from_device_a,
              TRANSFER_BOUNDARY_FROM_DEVICE_A)(long* Mesh_pointer) {

// asynchronous transfer from device to host

  TRACE("\ttransfer_boundary_from_device_a");

  Mesh* mp = (Mesh*)(*Mesh_pointer); // get Mesh from fortran integer wrapper

  if( mp->size_mpi_buffer > 0 ){

    int blocksize = BLOCKSIZE_TRANSFER;
    int size_padded = ((int)ceil(((double)mp->max_nibool_interfaces_ext_mesh)/((double)blocksize)))*blocksize;

    int num_blocks_x, num_blocks_y;
    get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

    dim3 grid(num_blocks_x,num_blocks_y);
    dim3 threads(blocksize,1,1);

    prepare_boundary_accel_on_device<<<grid,threads,0,mp->compute_stream>>>(mp->d_accel,mp->d_send_accel_buffer,
                                                                            mp->ninterface_elastic,
                                                                            mp->max_nibool_interfaces_ext_mesh,
                                                                            mp->d_nibool_interfaces_ext_mesh,
                                                                            mp->d_ibool_interfaces_ext_mesh,
                                                                            mp->d_inum_interfaces_elastic);
    // waits until kernel is finished before starting async memcpy
    //synchronize_cuda();
    // waits until previous compute stream finishes
    hipStreamSynchronize(mp->compute_stream);

    hipMemcpyAsync(mp->h_send_accel_buffer,mp->d_send_accel_buffer,
                    mp->size_mpi_buffer*sizeof(realw),hipMemcpyDeviceToHost,mp->copy_stream);
  }
}



extern "C"
void FC_FUNC_(prepare_boundary_on_device,
              PREPARE_BOUNDARY_ON_DEVICE)(long* Mesh_pointer) {

// asynchronous transfer from device to host

  TRACE("\ttransfer_boundary_from_device_a");

  Mesh* mp = (Mesh*)(*Mesh_pointer); // get Mesh from fortran integer wrapper

  if( mp->size_mpi_buffer > 0 ){

    int blocksize = BLOCKSIZE_TRANSFER;
    int size_padded = ((int)ceil(((double)mp->max_nibool_interfaces_ext_mesh)/((double)blocksize)))*blocksize;

    int num_blocks_x, num_blocks_y;
    get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

    dim3 grid(num_blocks_x,num_blocks_y);
    dim3 threads(blocksize,1,1);

    prepare_boundary_accel_on_device<<<grid,threads,0,mp->compute_stream>>>(mp->d_accel,mp->d_send_accel_buffer,
                                                                            mp->ninterface_elastic,
                                                                            mp->max_nibool_interfaces_ext_mesh,
                                                                            mp->d_nibool_interfaces_ext_mesh,
                                                                            mp->d_ibool_interfaces_ext_mesh,
                                                                            mp->d_inum_interfaces_elastic);
    // waits until kernel is finished before starting async memcpy
    //synchronize_cuda();
    // waits until previous compute stream finishes
    hipStreamSynchronize(mp->compute_stream);


  }
}



/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_boundary_to_device_a,
              TRANSFER_BOUNDARY_TO_DEVICE_A)(long* Mesh_pointer,
                                             realw* buffer_recv_vector_ext_mesh,
                                             const int* max_nibool_interfaces_ext_mesh) {

// asynchronous transfer from host to device

  TRACE("transfer_boundary_to_device_a");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  if( mp->size_mpi_buffer > 0 ){
    // copy on host memory
    memcpy(mp->h_recv_accel_buffer,buffer_recv_vector_ext_mesh,mp->size_mpi_buffer*sizeof(realw));

    // asynchronous copy to GPU using copy_stream
    hipMemcpyAsync(mp->d_send_accel_buffer,mp->h_recv_accel_buffer,
                    mp->size_mpi_buffer*sizeof(realw),hipMemcpyHostToDevice,mp->copy_stream);
  }
}


/* ----------------------------------------------------------------------------------------------- */

// Assembly

/* ----------------------------------------------------------------------------------------------- */

__global__ void assemble_boundary_accel_on_device(realw* d_accel, realw* d_send_accel_buffer,
                                                  const int ninterface_el,
                                                  const int max_nibool_interfaces_ext_mesh,
                                                  const int* d_nibool_interfaces_ext_mesh,
                                                  const int* d_ibool_interfaces_ext_mesh,
                                                  const int* inum_inter_elastic) {

  //int bx = blockIdx.y*gridDim.x+blockIdx.x;
  //int tx = threadIdx.x;
  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;

  int ientry,iglob,num_int;

  for( int iinterface=0; iinterface < ninterface_el; iinterface++) {

     num_int=inum_inter_elastic[iinterface]-1;

     if( id < d_nibool_interfaces_ext_mesh[num_int] ) {

      // entry in interface array
      ientry = id + max_nibool_interfaces_ext_mesh*num_int;
      // global index in wavefield
      iglob = d_ibool_interfaces_ext_mesh[ientry] - 1;

      // for testing atomic operations against not atomic operations (0.1ms vs. 0.04 ms)
      // d_accel[3*(iglob)] += d_send_accel_buffer[3*(ientry)];
      // d_accel[3*(iglob)+1] += d_send_accel_buffer[3*(ientry)+1];
      // d_accel[3*(iglob)+2] += d_send_accel_buffer[3*(ientry)+2];

      atomicAdd(&d_accel[2*iglob],d_send_accel_buffer[2*ientry]);
      atomicAdd(&d_accel[2*iglob + 1],d_send_accel_buffer[2*ientry + 1]);
    }
  }
  // ! This step is done via previous function transfer_and_assemble...
  // ! do iinterface = 1, num_interfaces_ext_mesh
  // !   do ipoin = 1, nibool_interfaces_ext_mesh(iinterface)
  // !     array_val(:,ibool_interfaces_ext_mesh(ipoin,iinterface)) = &
  // !          array_val(:,ibool_interfaces_ext_mesh(ipoin,iinterface)) + buffer_recv_vector_ext_mesh(:,ipoin,iinterface)
  // !   enddo
  // ! enddo
}


/* ----------------------------------------------------------------------------------------------- */

// FORWARD_OR_ADJOINT == 1 for accel, and == 3 for b_accel
extern "C"
void FC_FUNC_(transfer_asmbl_accel_to_device,
              TRANSFER_ASMBL_ACCEL_TO_DEVICE)(long* Mesh_pointer,
                                              realw* buffer_recv_vector_ext_mesh,
                                              const int* max_nibool_interfaces_ext_mesh,
                                              const int* nibool_interfaces_ext_mesh,
                                              const int* ibool_interfaces_ext_mesh,
                                              const int* FORWARD_OR_ADJOINT) {
TRACE("\ttransfer_asmbl_accel_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  if( mp->size_mpi_buffer > 0 ){

    //daniel: todo - check if this copy is only needed for adjoint simulation, otherwise it is called asynchronously?
    if(*FORWARD_OR_ADJOINT == 1 ){
      // Wait until previous copy stream finishes. We assemble while other compute kernels execute.
      hipStreamSynchronize(mp->copy_stream);
    }
    else if(*FORWARD_OR_ADJOINT == 3 ){
      // explicitly synchronizes
      // (hipMemcpy implicitly synchronizes all other cuda operations)
      synchronize_cuda();

      print_CUDA_error_if_any(hipMemcpy(mp->d_b_send_accel_buffer, buffer_recv_vector_ext_mesh,
                              mp->size_mpi_buffer*sizeof(realw),hipMemcpyHostToDevice),97001);
    }




    int blocksize = BLOCKSIZE_TRANSFER;
    int size_padded = ((int)ceil(((double)mp->max_nibool_interfaces_ext_mesh)/((double)blocksize)))*blocksize;

    int num_blocks_x, num_blocks_y;
    get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

    dim3 grid(num_blocks_x,num_blocks_y);
    dim3 threads(blocksize,1,1);

    //double start_time = get_time();
    // hipEvent_t start, stop;
    // realw time;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord( start, 0 );

    if(*FORWARD_OR_ADJOINT == 1) {
      //assemble forward accel


      assemble_boundary_accel_on_device<<<grid,threads,0,mp->compute_stream>>>(mp->d_accel, mp->d_send_accel_buffer,
                                                                               mp->ninterface_elastic,
                                                                               mp->max_nibool_interfaces_ext_mesh,
                                                                               mp->d_nibool_interfaces_ext_mesh,
                                                                               mp->d_ibool_interfaces_ext_mesh,
                                                                               mp->d_inum_interfaces_elastic);


    }
    else if(*FORWARD_OR_ADJOINT == 3) {
      //assemble adjoint accel
      assemble_boundary_accel_on_device<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_accel, mp->d_b_send_accel_buffer,
                                                                               mp->ninterface_elastic,
                                                                               mp->max_nibool_interfaces_ext_mesh,
                                                                               mp->d_nibool_interfaces_ext_mesh,
                                                                               mp->d_ibool_interfaces_ext_mesh,
                                                                               mp->d_inum_interfaces_elastic);
    }

    // hipEventRecord( stop, 0 );
    // hipEventSynchronize( stop );
    // hipEventElapsedTime( &time, start, stop );
    // hipEventDestroy( start );
    // hipEventDestroy( stop );
    // printf("Boundary Assemble Kernel Execution Time: %f ms\n",time);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //double end_time = get_time();
  //printf("Elapsed time: %e\n",end_time-start_time);
  exit_on_cuda_error("transfer_asmbl_accel_to_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(sync_copy_from_device,
              SYNC_copy_FROM_DEVICE)(long* Mesh_pointer,
                                     int* iphase,
                                     realw* send_buffer) {

  TRACE("sync_copy_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); // get Mesh from fortran integer wrapper

  // Wait until async-memcpy of outer elements is finished and start MPI.
  if( *iphase != 2 ){ exit_on_cuda_error("sync_copy_from_device must be called for iphase == 2"); }

  if( mp->size_mpi_buffer > 0 ){
    // waits for asynchronous copy to finish
    hipStreamSynchronize(mp->copy_stream);

    // There have been problems using the pinned-memory with MPI, so
    // we copy the buffer into a non-pinned region.
    memcpy(send_buffer,mp->h_send_accel_buffer,mp->size_mpi_buffer*sizeof(float));
  }
  // memory copy is now finished, so non-blocking MPI send can proceed
}

