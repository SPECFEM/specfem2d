/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                              CNRS, France
!                       and Princeton University, USA
!                 (there are currently many more authors!)
!                           (c) October 2017
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/

#include "mesh_constants_cuda.h"

/* ----------------------------------------------------------------------------------------------- */

// Transfer functions

/* ----------------------------------------------------------------------------------------------- */



/* ----------------------------------------------------------------------------------------------- */

// for ELASTIC simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_fields_el_to_device,
              TRANSFER_FIELDS_EL_TO_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer) {

  TRACE("transfer_fields_el_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(mp->d_displ,displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),40003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_veloc,veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),40004);
  print_CUDA_error_if_any(hipMemcpy(mp->d_accel,accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40005);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_fields_el_from_device,
              TRANSFER_FIELDS_EL_FROM_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer) {

  TRACE("transfer_fields_el_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
  print_CUDA_error_if_any(hipMemcpy(veloc,mp->d_veloc,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40008);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_fields_to_device,
              TRANSFER_B_FIELDS_TO_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,
                                           long* Mesh_pointer) {

  TRACE("transfer_b_fields_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(mp->d_b_displ,b_displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),41006);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_veloc,b_veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),41007);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_accel,b_accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),41008);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_fields_from_device,
              TRANSFER_B_FIELDS_FROM_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,long* Mesh_pointer) {

  TRACE("transfer_b_fields_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(b_displ,mp->d_b_displ,sizeof(realw)*(*size),hipMemcpyDeviceToHost),42006);
  print_CUDA_error_if_any(hipMemcpy(b_veloc,mp->d_b_veloc,sizeof(realw)*(*size),hipMemcpyDeviceToHost),42007);
  print_CUDA_error_if_any(hipMemcpy(b_accel,mp->d_b_accel,sizeof(realw)*(*size),hipMemcpyDeviceToHost),42008);

}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_accel_to_device,
              TRNASFER_ACCEL_TO_DEVICE)(int* size, realw* accel,long* Mesh_pointer) {

  TRACE("transfer_accel_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(mp->d_accel,accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40016);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_accel_from_device,
              TRANSFER_ACCEL_FROM_DEVICE)(int* size, realw* accel,long* Mesh_pointer) {

  TRACE("transfer_accel_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40026);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_accel_from_device,
              TRNASFER_B_ACCEL_FROM_DEVICE)(int* size, realw* b_accel,long* Mesh_pointer) {

  TRACE("transfer_b_accel_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(b_accel,mp->d_b_accel,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40036);

}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_displ_from_device,
              TRANSFER_B_DISPL_FROM_DEVICE)(int* size, realw* displ,long* Mesh_pointer) {

  TRACE("transfer_b_displ_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_b_displ,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40056);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_displ_from_device,
              TRANSFER_DISPL_FROM_DEVICE)(int* size, realw* displ,long* Mesh_pointer) {

  TRACE("transfer_displ_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40066);

}



// JC JC here we will need to add GPU support for the new C-PML routines

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_kernels_el_to_host,
              TRANSFER_KERNELS_EL_TO_HOST)(long* Mesh_pointer,
                                            realw* h_rho_kl,
                                            realw* h_mu_kl,
                                            realw* h_kappa_kl,
                                            int* NSPEC_AB) {
  TRACE("transfer_kernels_el_to_host");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(h_rho_kl,mp->d_rho_kl,*NSPEC_AB*NGLL2*sizeof(realw),
                                     hipMemcpyDeviceToHost),40101);

    print_CUDA_error_if_any(hipMemcpy(h_mu_kl,mp->d_mu_kl,*NSPEC_AB*NGLL2*sizeof(realw),
                                       hipMemcpyDeviceToHost),40102);
    print_CUDA_error_if_any(hipMemcpy(h_kappa_kl,mp->d_kappa_kl,*NSPEC_AB*NGLL2*sizeof(realw),
                                       hipMemcpyDeviceToHost),40103);

}


/* ----------------------------------------------------------------------------------------------- */

// for ACOUSTIC simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_fields_ac_to_device,
              TRANSFER_FIELDS_AC_TO_DEVICE)(int* size,
                                            realw* potential_acoustic,
                                            realw* potential_dot_acoustic,
                                            realw* potential_dot_dot_acoustic,
                                            long* Mesh_pointer) {

  TRACE("transfer_fields_ac_to_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(mp->d_potential_acoustic,potential_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyHostToDevice),50110);
  print_CUDA_error_if_any(hipMemcpy(mp->d_potential_dot_acoustic,potential_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyHostToDevice),50120);
  print_CUDA_error_if_any(hipMemcpy(mp->d_potential_dot_dot_acoustic,potential_dot_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyHostToDevice),50130);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_fields_ac_to_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_fields_ac_to_device,
              TRANSFER_B_FIELDS_AC_TO_DEVICE)(int* size,
                                              realw* b_potential_acoustic,
                                              realw* b_potential_dot_acoustic,
                                              realw* b_potential_dot_dot_acoustic,
                                              long* Mesh_pointer) {

  TRACE("transfer_b_fields_ac_to_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(mp->d_b_potential_acoustic,b_potential_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyHostToDevice),51110);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_potential_dot_acoustic,b_potential_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyHostToDevice),51120);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_potential_dot_dot_acoustic,b_potential_dot_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyHostToDevice),51130);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_b_fields_ac_to_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_fields_ac_from_device,
              TRANSFER_FIELDS_AC_FROM_DEVICE)(int* size,
                                              realw* potential_acoustic,
                                              realw* potential_dot_acoustic,
                                              realw* potential_dot_dot_acoustic,
                                              long* Mesh_pointer) {
  TRACE("transfer_fields_ac_from_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  //print_CUDA_error_if_any(hipStreamSynchronize(mp->compute_stream),52110);

  print_CUDA_error_if_any(hipMemcpy(potential_acoustic,mp->d_potential_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),52111);
  print_CUDA_error_if_any(hipMemcpy(potential_dot_acoustic,mp->d_potential_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),52121);
  print_CUDA_error_if_any(hipMemcpy(potential_dot_dot_acoustic,mp->d_potential_dot_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),52131);





#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_fields_ac_from_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_fields_ac_from_device,
              TRANSFER_B_FIELDS_AC_FROM_DEVICE)(int* size,
                                                realw* b_potential_acoustic,
                                                realw* b_potential_dot_acoustic,
                                                realw* b_potential_dot_dot_acoustic,
                                                long* Mesh_pointer) {
  TRACE("transfer_b_fields_ac_from_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(b_potential_acoustic,mp->d_b_potential_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),53111);
  print_CUDA_error_if_any(hipMemcpy(b_potential_dot_acoustic,mp->d_b_potential_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),53121);
  print_CUDA_error_if_any(hipMemcpy(b_potential_dot_dot_acoustic,mp->d_b_potential_dot_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),53131);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_b_fields_ac_from_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_potential_ac_from_device,
              TRANSFER_B_POTENTIAL_AC_FROM_DEVICE)(int* size,
                                                realw* b_potential_acoustic,
                                                long* Mesh_pointer) {
  TRACE("transfer_b_potential_ac_from_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);
  print_CUDA_error_if_any(hipMemcpy(b_potential_acoustic,mp->d_b_potential_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),53132);
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_b_potential_ac_from_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_potential_ac_to_device,
              TRANSFER_B_POTENTIAL_AC_TO_DEVICE)(int* size,
                                                 realw* b_potential_acoustic,
                                                 long* Mesh_pointer) {
  TRACE("transfer_b_potential_ac_to_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_potential_acoustic,b_potential_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyHostToDevice),53133);
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_b_potential_ac_to_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_dot_dot_from_device,
              TRNASFER_DOT_DOT_FROM_DEVICE)(int* size, realw* potential_dot_dot_acoustic,long* Mesh_pointer) {

  TRACE("transfer_dot_dot_from_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(potential_dot_dot_acoustic,mp->d_potential_dot_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),50041);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_dot_dot_from_device,
              TRNASFER_B_DOT_DOT_FROM_DEVICE)(int* size, realw* b_potential_dot_dot_acoustic,long* Mesh_pointer) {

  TRACE("transfer_b_dot_dot_from_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(b_potential_dot_dot_acoustic,mp->d_b_potential_dot_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),50042);

}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_kernels_ac_to_host,
              TRANSFER_KERNELS_AC_TO_HOST)(long* Mesh_pointer,realw* h_rho_ac_kl,realw* h_kappa_ac_kl,int* NSPEC_AB) {

  TRACE("transfer_kernels_ac_to_host");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = *NSPEC_AB*NGLL2;

  // copies kernel values over to CPU host
  print_CUDA_error_if_any(hipMemcpy(h_rho_ac_kl,mp->d_rho_ac_kl,size*sizeof(realw),
                                     hipMemcpyDeviceToHost),54101);
  print_CUDA_error_if_any(hipMemcpy(h_kappa_ac_kl,mp->d_kappa_ac_kl,size*sizeof(realw),
                                     hipMemcpyDeviceToHost),54102);
}

/* ----------------------------------------------------------------------------------------------- */

// for Hess kernel calculations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_kernels_hess_el_tohost,
              TRANSFER_KERNELS_HESS_EL_TOHOST)(long* Mesh_pointer,realw* h_hess_kl,int* NSPEC_AB) {

  TRACE("transfer_kernels_hess_el_tohost");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(h_hess_kl,mp->d_hess_el_kl,NGLL2*(*NSPEC_AB)*sizeof(realw),
                                     hipMemcpyDeviceToHost),70201);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_kernels_hess_ac_tohost,
              TRANSFER_KERNELS_HESS_AC_TOHOST)(long* Mesh_pointer,realw* h_hess_ac_kl,int* NSPEC_AB) {

  TRACE("transfer_kernels_hess_ac_tohost");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(h_hess_ac_kl,mp->d_hess_ac_kl,NGLL2*(*NSPEC_AB)*sizeof(realw),
                                     hipMemcpyDeviceToHost),70202);
}

//For UNDO_ATTENUATION

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_viscoacoustic_b_var_to_device,
              TRANSFER_VISCOACOUSTIC_b_VAR_TO_DEVICE)(int* size,
                                                      realw* b_e1_acous_sf,
                                                      realw* b_sum_forces_old,
                                                      long* Mesh_pointer) {

  TRACE("transfer_viscoacoustic_var_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_sum_forces_old,b_sum_forces_old,sizeof(realw)*(*size),hipMemcpyHostToDevice),70203);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_e1_acous,b_e1_acous_sf,sizeof(realw)*(*size)*N_SLS,hipMemcpyHostToDevice),70204);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_viscoacoustic_var_from_device,
              TRANSFER_VISCOACOUSTIC_VAR_FROM_DEVICE)(int* size,
                                                      realw* e1_acous_sf,
                                                      realw* sum_forces_old,
                                                      long* Mesh_pointer) {

  TRACE("transfer_viscoacoustic_var_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(sum_forces_old,mp->d_sum_forces_old,sizeof(realw)*(*size),hipMemcpyDeviceToHost),70205);
  print_CUDA_error_if_any(hipMemcpy(e1_acous_sf,mp->d_e1_acous,sizeof(realw)*(*size)*N_SLS,hipMemcpyDeviceToHost),70206);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_async_pot_ac_from_device,
              TRANSFER_ASYNC_POT_AC_FROM_DEVICE)(realw* pot_buffer,long* Mesh_pointer) {

  TRACE("transfer_async_pot_ac_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // waits for previous transfer to finish
  print_CUDA_error_if_any(hipStreamSynchronize(mp->compute_stream),70207);
//  print_CUDA_error_if_any(hipStreamSynchronize(mp->copy_stream_no_backward),70207);

  hipStreamWaitEvent(mp->compute_stream,mp->transfer_is_complete1,0);
  // adds the copy of d_potential_acoustic to the compute_stream stream to make sure it will be not overwritten by this same stream in further operations
  print_CUDA_error_if_any(hipMemcpyAsync(mp->d_potential_acoustic_buffer,mp->d_potential_acoustic,sizeof(realw)*mp->NGLOB_AB,hipMemcpyDeviceToDevice,mp->compute_stream),70208);
  // We create an event to know when the GPU buffer is ready for the transfer GPU ==> CPU
  hipEventRecord(mp->transfer_is_complete2,mp->compute_stream);
  hipStreamWaitEvent(mp->copy_stream_no_backward,mp->transfer_is_complete2,0);

  print_CUDA_error_if_any(hipMemcpyAsync(pot_buffer,mp->d_potential_acoustic_buffer,sizeof(realw)*mp->NGLOB_AB,hipMemcpyDeviceToHost,mp->copy_stream_no_backward),70209);

  hipEventRecord(mp->transfer_is_complete1,mp->copy_stream_no_backward);
//  print_CUDA_error_if_any(hipStreamSynchronize(mp->compute_stream),70207);
//  print_CUDA_error_if_any(hipStreamSynchronize(mp->copy_stream_no_backward),70207);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_async_pot_ac_to_device,
              TRANSFER_ASYNC_POT_AC_TO_DEVICE)(realw* pot_buffer,
                                               long* Mesh_pointer) {
  TRACE("transfer_async_pot_ac_to_device");
  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipStreamSynchronize(mp->compute_stream),70207);
//  print_CUDA_error_if_any(hipStreamSynchronize(mp->copy_stream_no_backward),70207);

  hipStreamWaitEvent(mp->compute_stream,mp->transfer_is_complete1,0);

  print_CUDA_error_if_any(hipMemcpyAsync(mp->d_b_potential_acoustic,mp->d_potential_acoustic_buffer,sizeof(realw)*mp->NGLOB_AB,hipMemcpyDeviceToDevice,mp->compute_stream),70211);

  hipEventRecord(mp->transfer_is_complete2,mp->compute_stream);
  hipStreamWaitEvent(mp->copy_stream_no_backward,mp->transfer_is_complete2,0);
  print_CUDA_error_if_any(hipMemcpyAsync(mp->d_potential_acoustic_buffer,pot_buffer,sizeof(realw)*mp->NGLOB_AB,hipMemcpyHostToDevice,mp->copy_stream_no_backward),70212);
//  print_CUDA_error_if_any(hipStreamSynchronize(mp->compute_stream),70207);
//  print_CUDA_error_if_any(hipStreamSynchronize(mp->copy_stream_no_backward),70207);

  hipEventRecord(mp->transfer_is_complete1,mp->copy_stream_no_backward);
}

/* ----------------------------------------------------------------------------------------------- */



// unused...

/* ----------------------------------------------------------------------------------------------- */
/*
extern "C"
void FC_FUNC_(transfer_compute_kernel_answers_from_device,
              TRANSFER_COMPUTE_KERNEL_ANSWERS_FROM_DEVICE)(long* Mesh_pointer,
                                                           realw* rho_kl,int* size_rho,
                                                           realw* mu_kl, int* size_mu,
                                                           realw* kappa_kl, int* size_kappa) {
TRACE("transfer_compute_kernel_answers_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  hipMemcpy(rho_kl,mp->d_rho_kl,*size_rho*sizeof(realw),hipMemcpyDeviceToHost);
  if (! mp->anisotropic_kl) {
    hipMemcpy(mu_kl,mp->d_mu_kl,*size_mu*sizeof(realw),hipMemcpyDeviceToHost);
    hipMemcpy(kappa_kl,mp->d_kappa_kl,*size_kappa*sizeof(realw),hipMemcpyDeviceToHost);
  }
}
*/

/* ----------------------------------------------------------------------------------------------- */
/*
extern "C"
void FC_FUNC_(transfer_compute_kernel_fields_from_device,
              TRANSFER_COMPUTE_KERNEL_FIELDS_FROM_DEVICE)(long* Mesh_pointer,
                                                          realw* accel, int* size_accel,
                                                          realw* b_displ, int* size_b_displ,
                                                          realw* epsilondev_xx,realw* epsilondev_yy,realw* epsilondev_xy,
                                                          realw* epsilondev_xz,realw* epsilondev_yz,
                                                          int* size_epsilondev,
                                                          realw* b_epsilondev_xx,realw* b_epsilondev_yy,realw* b_epsilondev_xy,
                                                          realw* b_epsilondev_xz,realw* b_epsilondev_yz,
                                                          int* size_b_epsilondev,
                                                          realw* rho_kl,int* size_rho,
                                                          realw* mu_kl, int* size_mu,
                                                          realw* kappa_kl, int* size_kappa,
                                                          realw* epsilon_trace_over_3,
                                                          realw* b_epsilon_trace_over_3,
                                                          int* size_epsilon_trace_over_3) {
TRACE("transfer_compute_kernel_fields_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  hipMemcpy(accel,mp->d_accel,*size_accel*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(b_displ,mp->d_b_displ,*size_b_displ*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_xx,mp->d_epsilondev_xx,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_yy,mp->d_epsilondev_yy,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_xy,mp->d_epsilondev_xy,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_xz,mp->d_epsilondev_xz,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_yz,mp->d_epsilondev_yz,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(b_epsilondev_xx,mp->d_b_epsilondev_xx,*size_b_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(b_epsilondev_yy,mp->d_b_epsilondev_yy,*size_b_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(b_epsilondev_xy,mp->d_b_epsilondev_xy,*size_b_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(b_epsilondev_xz,mp->d_b_epsilondev_xz,*size_b_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(b_epsilondev_yz,mp->d_b_epsilondev_yz,*size_b_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(rho_kl,mp->d_rho_kl,*size_rho*sizeof(realw),hipMemcpyDeviceToHost);

  if (! mp->anisotropic_kl) {
    hipMemcpy(mu_kl,mp->d_mu_kl,*size_mu*sizeof(realw),hipMemcpyDeviceToHost);
    hipMemcpy(kappa_kl,mp->d_kappa_kl,*size_kappa*sizeof(realw),hipMemcpyDeviceToHost);
  }

  hipMemcpy(epsilon_trace_over_3,mp->d_epsilon_trace_over_3,*size_epsilon_trace_over_3*sizeof(realw),
       hipMemcpyDeviceToHost);
  hipMemcpy(b_epsilon_trace_over_3,mp->d_b_epsilon_trace_over_3,*size_epsilon_trace_over_3*sizeof(realw),
       hipMemcpyDeviceToHost);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_compute_kernel_fields_from_device");
#endif
}
*/

