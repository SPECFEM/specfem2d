#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                              CNRS, France
!                       and Princeton University, USA
!                 (there are currently many more authors!)
!                           (c) October 2017
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/

#include "mesh_constants_cuda.h"


#ifdef USE_TEXTURES_FIELDS
realw_texture d_potential_tex;
realw_texture d_potential_dot_dot_tex;
//backward/reconstructed
realw_texture d_b_potential_tex;
realw_texture d_b_potential_dot_dot_tex;

//note: texture variables are implicitly static, and cannot be passed as arguments to cuda kernels;
//      thus, 1) we thus use if-statements (FORWARD_OR_ADJOINT) to determine from which texture to fetch from
//            2) we use templates
//      since if-statements are a bit slower as the variable is only known at runtime, we use option 2)

// templates definitions
template<int FORWARD_OR_ADJOINT> __device__ float texfetch_potential(int x);
template<int FORWARD_OR_ADJOINT> __device__ float texfetch_potential_dot_dot(int x);

// templates for texture fetching
// FORWARD_OR_ADJOINT == 1 <- forward arrays
template<> __device__ float texfetch_potential<1>(int x) { return tex1Dfetch(d_potential_tex, x); }
template<> __device__ float texfetch_potential_dot_dot<1>(int x) { return tex1Dfetch(d_potential_dot_dot_tex, x); }
// FORWARD_OR_ADJOINT == 3 <- backward/reconstructed arrays
template<> __device__ float texfetch_potential<3>(int x) { return tex1Dfetch(d_b_potential_tex, x); }
template<> __device__ float texfetch_potential_dot_dot<3>(int x) { return tex1Dfetch(d_b_potential_dot_dot_tex, x); }

#endif

#ifdef USE_TEXTURES_CONSTANTS
// already defined in compute_forces_viscoelastic_cuda.cu
extern realw_texture d_hprime_xx_tex;
//extern realw_texture d_hprimewgll_xx_tex;
extern realw_texture d_wxgll_xx_tex;
#endif


// note on performance optimizations:
//
//   performance tests done:
//   - registers: we were trying to reduce the number of registers, as this is the main limiter for the
//                occupancy of the kernel. however, there is only little difference in register pressure for one "general" kernel
//                or multiple "spezialized" kernels. reducing registers is mainly achieved through the launch_bonds() directive.
//   - branching: we were trying to reduce code branches, such as the if-active check in earlier code versions.
//                reducing the branching helps the compiler to better optimize the executable.
//   - memory accesses: the global memory accesses are avoiding texture reads for coalescent arrays, as this is
//                still faster. thus we were using no __ldg() loads or __restricted__ pointer usage,
//                as those implicitly lead the compiler to use texture reads.
//   - arithmetic intensity: ratio of floating-point operations vs. memory accesses is still low for our kernels.
//                tests with using a loop over elements to re-use the constant arrays (like hprime, wgllwgll,..) and thus
//                increasing the arithmetic intensity failed because the number of registers increased as well.
//                this increased register pressure reduced the occupancy and slowed down the kernel performance.
//   - hiding memory latency: to minimize waiting times to retrieve a memory value from global memory, we put
//                some more calculations into the same code block before calling syncthreads(). this should help the
//                compiler to move independent calculations to wherever it can overlap it with memory access operations.
//                note, especially the if (gravity )-block locations are very sensitive
//                for optimal register usage and compiler optimizations
//

/* ----------------------------------------------------------------------------------------------- */

// KERNEL 2 - acoustic compute forces kernel

/* ----------------------------------------------------------------------------------------------- */

template<int FORWARD_OR_ADJOINT> __global__ void
#ifdef USE_LAUNCH_BOUNDS
// adds compiler specification
__launch_bounds__(NGLL2_PADDED,LAUNCH_MIN_BLOCKS_ACOUSTIC)
#endif
Kernel_2_acoustic_impl(const int nb_blocks_to_compute,
                       const int* d_ibool,
                       const int* d_phase_ispec_inner_acoustic,
                       const int num_phase_ispec_acoustic,
                       const int d_iphase,
                       realw_const_p d_potential_acoustic,
                       realw_p d_potential_dot_dot_acoustic,
                       realw_const_p d_b_potential_acoustic,
                       realw_p d_b_potential_dot_dot_acoustic,
                       const int nb_field,
                       const realw* d_xix, const realw* d_xiz,
                       const realw* d_gammax,const realw* d_gammaz,
                       realw_const_p d_hprime_xx,
                       realw_const_p d_hprimewgll_xx,
                       realw_const_p d_wxgll,
                       const realw* d_rhostore){

  // block-id == number of local element id in phase_ispec array
  int bx = blockIdx.y*gridDim.x+blockIdx.x;

  // thread-id == GLL node id
  // note: use only NGLL^2 = 25 active threads, plus 7 inactive/ghost threads,
  //       because we used memory padding from NGLL^2 = 25 to 32 to get coalescent memory accesses;
  //       to avoid execution branching and the need of registers to store an active state variable,
  //       the thread ids are put in valid range
  int tx = threadIdx.x;

  int I,J;
  int iglob,offset;

  realw temp1l,temp3l;
  realw xixl,xizl,gammaxl,gammazl;

  realw dpotentialdxl,dpotentialdzl;
  realw rho_invl_times_jacobianl;

  realw sum_terms;

  __shared__ realw s_dummy_loc[2*NGLL2];

  __shared__ realw s_temp1[NGLL2];
  __shared__ realw s_temp3[NGLL2];

  __shared__ realw sh_hprime_xx[NGLL2];
  __shared__ realw sh_hprimewgll_xx[NGLL2];
  __shared__ realw sh_wxgll[NGLLX];



// arithmetic intensity: ratio of number-of-arithmetic-operations / number-of-bytes-accessed-on-DRAM
//
// hand-counts on floating-point operations: counts addition/subtraction/multiplication/division
//                                           no counts for operations on indices in for-loops (compiler will likely unrool loops)
//
//                                           counts accesses to global memory, but no shared memory or register loads/stores
//                                           float has 4 bytes

//         counts floating-point operations (FLOP) per thread
//         counts global memory accesses in bytes (BYTES) per block
// 2 FLOP
//
// 0 BYTES

  // checks if anything to do
  if (bx >= nb_blocks_to_compute ) return;

// counts:
// + 1 FLOP
//
// + 0 BYTE

  // local padded index
  offset = (d_phase_ispec_inner_acoustic[bx + num_phase_ispec_acoustic*(d_iphase-1)]-1)*NGLL2_PADDED + tx;

  // global index
  iglob = d_ibool[offset] - 1;


// counts:
// + 7 FLOP
//
// + 2 float * 32 threads = 256 BYTE

#ifdef USE_TEXTURES_FIELDS
  s_dummy_loc[tx] = texfetch_potential<FORWARD_OR_ADJOINT>(iglob);
  if (nb_field==2) s_dummy_loc[NGLL2+tx]=texfetch_potential<3>(iglob);
#else
  // changing iglob indexing to match fortran row changes fast style
  s_dummy_loc[tx] = d_potential_acoustic[iglob];
  if (nb_field==2) s_dummy_loc[NGLL2+tx]=d_b_potential_acoustic[iglob];
#endif


// counts:
// + 0 FLOP
//
// + 1 float * 25 threads = 100 BYTE

  // local index
  J = (tx/NGLLX);
  I = (tx-J*NGLLX);

// counts:
// + 3 FLOP
//
// + 0 BYTES

  // note: loads mesh values here to give compiler possibility to overlap memory fetches with some computations;
  //       arguments defined as realw* instead of const realw* __restrict__ to avoid that the compiler
  //       loads all memory by texture loads (arrays accesses are coalescent, thus no need for texture reads)
  //
  // calculates laplacian
  xixl = get_global_cr( &d_xix[offset] );
  xizl = d_xiz[offset];
  gammaxl = d_gammax[offset];
  gammazl = d_gammaz[offset];

  rho_invl_times_jacobianl = 1.f /(d_rhostore[offset] * (xixl*gammazl-gammaxl*xizl));

// counts:
// + 5 FLOP
//
// + 5 float * 32 threads = 160 BYTE

  // loads hprime into shared memory

#ifdef USE_TEXTURES_CONSTANTS
  sh_hprime_xx[tx] = tex1Dfetch(d_hprime_xx_tex,tx);
#else
  sh_hprime_xx[tx] = d_hprime_xx[tx];
#endif
  // loads hprimewgll into shared memory
  sh_hprimewgll_xx[tx] = d_hprimewgll_xx[tx];

  if (threadIdx.x < NGLLX){
#ifdef USE_TEXTURES_CONSTANTS
    sh_wxgll[tx] = tex1Dfetch(d_wxgll_xx_tex,tx);
#else
    // changing iglob indexing to match fortran row changes fast style
    sh_wxgll[tx] = d_wxgll[tx];
#endif
  }


// counts:
// + 0 FLOP
//
// + 2 * 1 float * 25 threads = 200 BYTE

  for (int k=0 ; k < nb_field ; k++) {

    // synchronize all the threads (one thread for each of the NGLL grid points of the
    // current spectral element) because we need the whole element to be ready in order
    // to be able to compute the matrix products along cut planes of the 3D element below
    __syncthreads();

    // computes first matrix product
    temp1l = 0.f;
    temp3l = 0.f;

    for (int l=0;l<NGLLX;l++) {

      //assumes that hprime_xx = hprime_yy = hprime_zz
      // 1. cut-plane along xi-direction
      temp1l += s_dummy_loc[NGLL2*k+J*NGLLX+l] * sh_hprime_xx[l*NGLLX+I];
      // 3. cut-plane along gamma-direction
      temp3l += s_dummy_loc[NGLL2*k+l*NGLLX+I] * sh_hprime_xx[l*NGLLX+J];
    }

// counts:
// + NGLLX * 2 * 6 FLOP = 60 FLOP
//
// + 0 BYTE

    // compute derivatives of ux, uy and uz with respect to x, y and z
    // derivatives of potential
    dpotentialdxl = xixl*temp1l +  gammaxl*temp3l;
    dpotentialdzl = xizl*temp1l +  gammazl*temp3l;

// counts:
// + 2 * 3 FLOP = 6 FLOP
//
// + 0 BYTE

    // form the dot product with the test vector
    s_temp1[tx] = sh_wxgll[J]*rho_invl_times_jacobianl  * (dpotentialdxl*xixl  + dpotentialdzl*xizl)  ;
    s_temp3[tx] = sh_wxgll[I]*rho_invl_times_jacobianl  * (dpotentialdxl*gammaxl + dpotentialdzl*gammazl)  ;

// counts:
// + 2 * 6 FLOP = 12 FLOP
//
// + 2 BYTE

    // synchronize all the threads (one thread for each of the NGLL grid points of the
    // current spectral element) because we need the whole element to be ready in order
    // to be able to compute the matrix products along cut planes of the 3D element below
    __syncthreads();

    sum_terms = 0.f;
    for (int l=0;l<NGLLX;l++) {
      //assumes hprimewgll_xx = hprimewgll_zz
      sum_terms -= s_temp1[J*NGLLX+l] * sh_hprimewgll_xx[I*NGLLX+l] + s_temp3[l*NGLLX+I] * sh_hprimewgll_xx[J*NGLLX+l];
    }

// counts:
// + NGLLX * 11 FLOP = 55 FLOP
//
// + 0 BYTE

    // assembles potential array
    if (k==0) {
      atomicAdd(&d_potential_dot_dot_acoustic[iglob],sum_terms);
    } else {
      atomicAdd(&d_b_potential_dot_dot_acoustic[iglob],sum_terms);
    }
// counts:
// + 1 FLOP
//
// + 1 float * 25 threads = 100 BYTE

// -----------------
// total of: 149 FLOP per thread
//           ~ 32 * 149 = 4768 FLOP per block
//
//           818 BYTE DRAM accesses per block
//
//           -> arithmetic intensity: 4768 FLOP / 818 BYTES ~ 5.83 FLOP/BYTE (hand-count)
  }
}

/* ----------------------------------------------------------------------------------------------- */

// KERNEL 2 - viscoacoustic compute forces kernel

/* ----------------------------------------------------------------------------------------------- */

template<int FORWARD_OR_ADJOINT> __global__ void
#ifdef USE_LAUNCH_BOUNDS
// adds compiler specification
__launch_bounds__(NGLL2_PADDED,LAUNCH_MIN_BLOCKS_ACOUSTIC)
#endif
Kernel_2_viscoacoustic_impl(const int nb_blocks_to_compute,
                            const int* d_ibool,
                            const int* d_phase_ispec_inner_acoustic,
                            const int num_phase_ispec_acoustic,
                            const int d_iphase,
                            realw_const_p d_potential_acoustic,
                            realw_p d_potential_dot_dot_acoustic,
                            const realw* d_xix, const realw* d_xiz,
                            const realw* d_gammax,const realw* d_gammaz,
                            realw_const_p d_hprime_xx,
                            realw_const_p d_hprimewgll_xx,
                            realw_const_p d_wxgll,
                            const realw* d_rhostore,
                            realw_p d_e1_acous,
                            const realw* d_A_newmark,
                            const realw* d_B_newmark,
                            realw_p d_sum_forces_old){

  // block-id == number of local element id in phase_ispec array
  int bx = blockIdx.y*gridDim.x+blockIdx.x;
  int tx = threadIdx.x;
  int I,J;
  int iglob,offset,offset_align,i_sls;

  realw temp1l,temp3l;
  realw xixl,xizl,gammaxl,gammazl;
  realw dpotentialdxl,dpotentialdzl;
  realw rho_invl_times_jacobianl;
  realw sum_terms;
  realw sum_forces_old,forces_attenuation,a_newmark;
  realw e1_acous_load[N_SLS];

  __shared__ realw s_dummy_loc[NGLL2];
  __shared__ realw s_temp1[NGLL2];
  __shared__ realw s_temp3[NGLL2];
  __shared__ realw sh_hprime_xx[NGLL2];
  __shared__ realw sh_hprimewgll_xx[NGLL2];
  __shared__ realw sh_wxgll[NGLLX];

  if (bx >= nb_blocks_to_compute ) return;

  I =d_phase_ispec_inner_acoustic[bx + num_phase_ispec_acoustic*(d_iphase-1)]-1;
  offset = I*NGLL2_PADDED + tx;
  offset_align = I*NGLL2 + tx;
  iglob = d_ibool[offset] - 1;

#ifdef USE_TEXTURES_FIELDS
  s_dummy_loc[tx] = texfetch_potential<FORWARD_OR_ADJOINT>(iglob);
#else
  s_dummy_loc[tx] = d_potential_acoustic[iglob];
#endif

  // local index
  J = (tx/NGLLX);
  I = (tx-J*NGLLX);

  xixl = get_global_cr( &d_xix[offset] );
  xizl = d_xiz[offset];
  gammaxl = d_gammax[offset];
  gammazl = d_gammaz[offset];

  rho_invl_times_jacobianl = 1.f /(d_rhostore[offset] * (xixl*gammazl-gammaxl*xizl));

  for (i_sls=0;i_sls<N_SLS;i_sls++)  e1_acous_load[i_sls] = d_e1_acous[N_SLS*offset_align+i_sls];

#ifdef USE_TEXTURES_CONSTANTS
  sh_hprime_xx[tx] = tex1Dfetch(d_hprime_xx_tex,tx);
#else
  sh_hprime_xx[tx] = d_hprime_xx[tx];
#endif
  // loads hprimewgll into shared memory
  sh_hprimewgll_xx[tx] = d_hprimewgll_xx[tx];

  if (threadIdx.x < NGLLX){
#ifdef USE_TEXTURES_CONSTANTS
    sh_wxgll[tx] = tex1Dfetch(d_wxgll_xx_tex,tx);
#else
    sh_wxgll[tx] = d_wxgll[tx];
#endif
  }

  __syncthreads();

  // computes first matrix product
  temp1l = 0.f;
  temp3l = 0.f;

  for (int l=0;l<NGLLX;l++) {
    //assumes that hprime_xx = hprime_yy = hprime_zz
    // 1. cut-plane along xi-direction
    temp1l += s_dummy_loc[J*NGLLX+l] * sh_hprime_xx[l*NGLLX+I];
    // 3. cut-plane along gamma-direction
    temp3l += s_dummy_loc[l*NGLLX+I] * sh_hprime_xx[l*NGLLX+J];
  }

  dpotentialdxl = xixl*temp1l +  gammaxl*temp3l;
  dpotentialdzl = xizl*temp1l +  gammazl*temp3l;
  s_temp1[tx] = sh_wxgll[J]*rho_invl_times_jacobianl  * (dpotentialdxl*xixl  + dpotentialdzl*xizl)  ;
  s_temp3[tx] = sh_wxgll[I]*rho_invl_times_jacobianl  * (dpotentialdxl*gammaxl + dpotentialdzl*gammazl)  ;

  __syncthreads();

  sum_terms = 0.f;
  for (int l=0;l<NGLLX;l++) {
    //assumes hprimewgll_xx = hprimewgll_zz
    sum_terms -= s_temp1[J*NGLLX+l] * sh_hprimewgll_xx[I*NGLLX+l] + s_temp3[l*NGLLX+I] * sh_hprimewgll_xx[J*NGLLX+l];
  }

  sum_forces_old = d_sum_forces_old[offset_align];
  forces_attenuation = 0.f;

  for (i_sls=0;i_sls<N_SLS;i_sls++){
    a_newmark = d_A_newmark[N_SLS * offset_align + i_sls];
    e1_acous_load[i_sls] = a_newmark * a_newmark * e1_acous_load[i_sls] + d_B_newmark[N_SLS * offset_align + i_sls] * (sum_terms + a_newmark * sum_forces_old);
    forces_attenuation += e1_acous_load[i_sls];
    d_e1_acous[N_SLS*offset_align+i_sls] = e1_acous_load[i_sls];
  }

  d_sum_forces_old[offset_align] = sum_terms;
  sum_terms += forces_attenuation;

  atomicAdd(&d_potential_dot_dot_acoustic[iglob],sum_terms);
}




/* ----------------------------------------------------------------------------------------------- */

void Kernel_2_acoustic(int nb_blocks_to_compute, Mesh* mp, int d_iphase,
                       int* d_ibool,
                       realw* d_xix,realw* d_xiz,
                       realw* d_gammax,realw* d_gammaz,
                       realw* d_rhostore,
                       int ATTENUATION_VISCOACOUSTIC,
                       int compute_wavefield_1,
                       int compute_wavefield_2) {

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("before acoustic kernel Kernel 2");
#endif

  // if the grid can handle the number of blocks, we let it be 1D
  int blocksize = NGLL2;

  int num_blocks_x, num_blocks_y, nb_field;
  get_blocks_xy(nb_blocks_to_compute,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // Cuda timing
  hipEvent_t start, stop;
  if (CUDA_TIMING) {
    start_timing_cuda(&start,&stop);
  }

  if (compute_wavefield_1 && compute_wavefield_2){
    nb_field=2;
  }else{
    nb_field=1;
  }
  if ( ! ATTENUATION_VISCOACOUSTIC){
    if (nb_field==2){
      // forward wavefields -> FORWARD_OR_ADJOINT == 1
      Kernel_2_acoustic_impl<1><<<grid,threads,0,mp->compute_stream>>>(nb_blocks_to_compute,
                                                                       d_ibool,
                                                                       mp->d_phase_ispec_inner_acoustic,
                                                                       mp->num_phase_ispec_acoustic,
                                                                       d_iphase,
                                                                       mp->d_potential_acoustic, mp->d_potential_dot_dot_acoustic,
                                                                       mp->d_b_potential_acoustic,mp->d_b_potential_dot_dot_acoustic,
                                                                       nb_field,
                                                                       d_xix, d_xiz,
                                                                       d_gammax, d_gammaz,
                                                                       mp->d_hprime_xx,
                                                                       mp->d_hprimewgll_xx,
                                                                       mp->d_wxgll,
                                                                       d_rhostore);
    }else{ // nb_field==1
      if (compute_wavefield_1){
        // forward wavefields -> FORWARD_OR_ADJOINT == 1
        Kernel_2_acoustic_impl<1><<<grid,threads,0,mp->compute_stream>>>(nb_blocks_to_compute,
                                                                         d_ibool,
                                                                         mp->d_phase_ispec_inner_acoustic,
                                                                         mp->num_phase_ispec_acoustic,
                                                                         d_iphase,
                                                                         mp->d_potential_acoustic, mp->d_potential_dot_dot_acoustic,
                                                                         mp->d_b_potential_acoustic,mp->d_b_potential_dot_dot_acoustic,
                                                                         nb_field,
                                                                         d_xix, d_xiz,
                                                                         d_gammax, d_gammaz,
                                                                         mp->d_hprime_xx,
                                                                         mp->d_hprimewgll_xx,
                                                                         mp->d_wxgll,
                                                                         d_rhostore);
      }
      if (compute_wavefield_2){
        // this run only happens with UNDO_ATTENUATION_AND_OR_PML on
        // adjoint wavefields -> FORWARD_OR_ADJOINT == 3
        Kernel_2_acoustic_impl<3><<<grid,threads,0,mp->compute_stream>>>(nb_blocks_to_compute,
                                                                         d_ibool,
                                                                         mp->d_phase_ispec_inner_acoustic,
                                                                         mp->num_phase_ispec_acoustic,
                                                                         d_iphase,
                                                                         mp->d_b_potential_acoustic, mp->d_b_potential_dot_dot_acoustic,
                                                                         mp->d_b_potential_acoustic,mp->d_b_potential_dot_dot_acoustic,
                                                                         nb_field,
                                                                         d_xix, d_xiz,
                                                                         d_gammax, d_gammaz,
                                                                         mp->d_hprime_xx,
                                                                         mp->d_hprimewgll_xx,
                                                                         mp->d_wxgll,
                                                                         d_rhostore);
      } //compute_wavefield_1
    } //nb_field
  }else{ // ATTENUATION_VISCOACOUSTIC== .true. below
    if (compute_wavefield_1) {
      Kernel_2_viscoacoustic_impl<1><<<grid,threads,0,mp->compute_stream>>>(nb_blocks_to_compute,
                                                                            d_ibool,
                                                                            mp->d_phase_ispec_inner_acoustic,
                                                                            mp->num_phase_ispec_acoustic,
                                                                            d_iphase,
                                                                            mp->d_potential_acoustic, mp->d_potential_dot_dot_acoustic,
                                                                            d_xix, d_xiz,
                                                                            d_gammax, d_gammaz,
                                                                            mp->d_hprime_xx,
                                                                            mp->d_hprimewgll_xx,
                                                                            mp->d_wxgll,
                                                                            d_rhostore,
                                                                            mp->d_e1_acous,
                                                                            mp->d_A_newmark_acous,
                                                                            mp->d_B_newmark_acous,
                                                                            mp->d_sum_forces_old);
    }
    if (compute_wavefield_2) {
      Kernel_2_viscoacoustic_impl<3><<<grid,threads,0,mp->compute_stream>>>(nb_blocks_to_compute,
                                                                            d_ibool,
                                                                            mp->d_phase_ispec_inner_acoustic,
                                                                            mp->num_phase_ispec_acoustic,
                                                                            d_iphase,
                                                                            mp->d_b_potential_acoustic, mp->d_b_potential_dot_dot_acoustic,
                                                                            d_xix, d_xiz,
                                                                            d_gammax, d_gammaz,
                                                                            mp->d_hprime_xx,
                                                                            mp->d_hprimewgll_xx,
                                                                            mp->d_wxgll,
                                                                            d_rhostore,
                                                                            mp->d_b_e1_acous,
                                                                            mp->d_A_newmark_acous,
                                                                            mp->d_B_newmark_acous,
                                                                            mp->d_b_sum_forces_old);
    }
  } // ATTENUATION_VISCOACOUSTIC



  // Cuda timing
  if (CUDA_TIMING) {
    realw flops,time;
    stop_timing_cuda(&start,&stop,"Kernel_2_acoustic_impl",&time);
    // time in seconds
    time = time / 1000.;
    flops = 15559 * nb_blocks_to_compute;
    printf("  performance: %f GFlop/s\n", flops/time * 1.e-9);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("kernel Kernel_2");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// main compute_forces_acoustic CUDA routine

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_forces_acoustic_cuda,
              COMPUTE_FORCES_ACOUSTIC_CUDA)(long* Mesh_pointer,
                                            int* iphase,
                                            int* nspec_outer_acoustic,
                                            int* nspec_inner_acoustic,
                                            int* ATTENUATION_VISCOACOUSTIC,
                                            int* compute_wavefield_1,
                                            int* compute_wavefield_2) {
  TRACE("compute_forces_acoustic_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); // get Mesh from fortran integer wrapper
  int num_elements;

  if (*iphase == 1)
    num_elements = *nspec_outer_acoustic;
  else
    num_elements = *nspec_inner_acoustic;
  if (num_elements == 0) return;

  // no mesh coloring: uses atomic updates
  Kernel_2_acoustic(num_elements, mp, *iphase,
                    mp->d_ibool,
                    mp->d_xix,mp->d_xiz,
                    mp->d_gammax,mp->d_gammaz,
                    mp->d_rhostore,
                    *ATTENUATION_VISCOACOUSTIC,
                    *compute_wavefield_1,
                    *compute_wavefield_2);
}



/* ----------------------------------------------------------------------------------------------- */

/* KERNEL for enforce free surface */

/* ----------------------------------------------------------------------------------------------- */


__global__ void enforce_free_surface_cuda_kernel(realw_p potential_acoustic,
                                                 realw_p potential_dot_acoustic,
                                                 realw_p potential_dot_dot_acoustic,
                                                 const int num_free_surface_faces,
                                                 const int* free_surface_ispec,
                                                 const int* free_surface_ij,
                                                 const int* d_ibool,
                                                 const int* ispec_is_acoustic) {
  // gets spectral element face id
  int iface = blockIdx.x + gridDim.x*blockIdx.y;

  // for all faces on free surface
  if (iface < num_free_surface_faces) {

    int ispec = free_surface_ispec[iface]-1;

    // checks if element is in acoustic domain
    if (ispec_is_acoustic[ispec]) {

      // gets global point index
      int igll = threadIdx.x + threadIdx.y*blockDim.x;

      int i = free_surface_ij[INDEX3(NDIM,NGLLX,0,igll,iface)] - 1; // (1,igll,iface)
      int j = free_surface_ij[INDEX3(NDIM,NGLLX,1,igll,iface)] - 1;

      int iglob = d_ibool[INDEX3_PADDED(NGLLX,NGLLX,i,j,ispec)] - 1;

      // sets potentials to zero at free surface
      potential_acoustic[iglob] = 0.f;
      potential_dot_acoustic[iglob] = 0.f;
      potential_dot_dot_acoustic[iglob] = 0.f;
    }
  }
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(acoustic_enforce_free_surf_cuda,
              ACOUSTIC_ENFORCE_FREE_SURF_CUDA)(long* Mesh_pointer,int* compute_wavefield_1,int* compute_wavefield_2) {

  TRACE("acoustic_enforce_free_surf_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // does not absorb free surface, thus we enforce the potential to be zero at surface

  // checks if anything to do
  if (mp->num_free_surface_faces == 0) return;

  // block sizes
  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->num_free_surface_faces,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y,1);
  dim3 threads(NGLLX,1,1);


  // sets potentials to zero at free surface
  if (*compute_wavefield_1) {
  enforce_free_surface_cuda_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_potential_acoustic,
                                                                          mp->d_potential_dot_acoustic,
                                                                          mp->d_potential_dot_dot_acoustic,
                                                                          mp->num_free_surface_faces,
                                                                          mp->d_free_surface_ispec,
                                                                          mp->d_free_surface_ijk,
                                                                          mp->d_ibool,
                                                                          mp->d_ispec_is_acoustic);
  }
  // for backward/reconstructed potentials
  if (*compute_wavefield_2) {
    enforce_free_surface_cuda_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_potential_acoustic,
                                                                            mp->d_b_potential_dot_acoustic,
                                                                            mp->d_b_potential_dot_dot_acoustic,
                                                                            mp->num_free_surface_faces,
                                                                            mp->d_free_surface_ispec,
                                                                            mp->d_free_surface_ijk,
                                                                            mp->d_ibool,
                                                                            mp->d_ispec_is_acoustic);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("enforce_free_surface_cuda");
#endif
}


