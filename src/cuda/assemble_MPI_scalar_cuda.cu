#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This software is governed by the CeCILL license under French law and
! abiding by the rules of distribution of free software. You can use,
! modify and/or redistribute the software under the terms of the CeCILL
! license as circulated by CEA, CNRS and Inria at the following URL
! "http://www.cecill.info".
!
! As a counterpart to the access to the source code and rights to copy,
! modify and redistribute granted by the license, users are provided only
! with a limited warranty and the software's author, the holder of the
! economic rights, and the successive licensors have only limited
! liability.
!
! In this respect, the user's attention is drawn to the risks associated
! with loading, using, modifying and/or developing or reproducing the
! software by the user in light of its specific status of free software,
! that may mean that it is complicated to manipulate, and that also
! therefore means that it is reserved for developers and experienced
! professionals having in-depth computer knowledge. Users are therefore
! encouraged to load and test the software's suitability as regards their
! requirements in conditions enabling the security of their systems and/or
! data to be ensured and, more generally, to use and operate it in the
! same conditions as regards security.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================

*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

// ASSEMBLY - mpi data transfer between CPU-GPU

/* ----------------------------------------------------------------------------------------------- */

// prepares a device array with with all inter-element edge-nodes -- this
// is followed by a memcpy and MPI operations
__global__ void prepare_boundary_potential_on_device(realw* d_potential_dot_dot_acoustic,
                                                     realw* d_send_potential_dot_dot_buffer,
                                                     const int ninterface_ac,
                                                     const int max_nibool_interfaces_ext_mesh,
                                                     const int* d_nibool_interfaces_ext_mesh,
                                                     const int* d_ibool_interfaces_ext_mesh,
                                                     const int* inum_inter_acoustic) {

  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;
  int ientry,iglob,num_int;

  for(int iinterface=0; iinterface < ninterface_ac; iinterface++) {

   num_int=inum_inter_acoustic[iinterface]-1;

    if(id<d_nibool_interfaces_ext_mesh[num_int]) {

      // entry in interface array
      ientry = id + max_nibool_interfaces_ext_mesh*num_int;
      // global index in wavefield
      iglob = d_ibool_interfaces_ext_mesh[ientry] - 1;

      d_send_potential_dot_dot_buffer[ientry] = d_potential_dot_dot_acoustic[iglob];
    }
  }

}


/* ----------------------------------------------------------------------------------------------- */

// prepares and transfers the inter-element edge-nodes to the host to be MPI'd
extern "C"
void FC_FUNC_(transfer_boun_pot_from_device,
              TRANSFER_BOUN_POT_FROM_DEVICE)(long* Mesh_pointer,
                                             realw* send_potential_dot_dot_buffer,
                                             const int* FORWARD_OR_ADJOINT){

TRACE("transfer_boun_pot_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // checks if anything to do
  if( mp->size_mpi_buffer_potential > 0 ){

    int blocksize = BLOCKSIZE_TRANSFER;
    int size_padded = ((int)ceil(((double)(mp->max_nibool_interfaces_ext_mesh))/((double)blocksize)))*blocksize;

    int num_blocks_x, num_blocks_y;
    get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

    dim3 grid(num_blocks_x,num_blocks_y);
    dim3 threads(blocksize,1,1);

    if(*FORWARD_OR_ADJOINT == 1) {

     prepare_boundary_potential_on_device<<<grid,threads,0,mp->compute_stream>>>(mp->d_potential_dot_dot_acoustic,
                                                                                   mp->d_send_potential_dot_dot_buffer,
                                                                                   mp->ninterface_acoustic,
                                                                                   mp->max_nibool_interfaces_ext_mesh,
                                                                                   mp->d_nibool_interfaces_ext_mesh,
                                                                                   mp->d_ibool_interfaces_ext_mesh,
                                                                                   mp->d_inum_interfaces_acoustic);

      // synchronizes
      //synchronize_cuda();
      // explicitly waits until previous compute stream finishes
      // (hipMemcpy implicitly synchronizes all other cuda operations)
      print_CUDA_error_if_any(hipStreamSynchronize(mp->compute_stream),95);

      print_CUDA_error_if_any(hipMemcpy(send_potential_dot_dot_buffer,mp->d_send_potential_dot_dot_buffer,
                                         mp->size_mpi_buffer_potential*sizeof(realw),hipMemcpyDeviceToHost),98000);
    }
    else if(*FORWARD_OR_ADJOINT == 3) {
      // backward/reconstructed wavefield buffer
      prepare_boundary_potential_on_device<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_potential_dot_dot_acoustic,
                                                                                   mp->d_b_send_potential_dot_dot_buffer,
                                                                                   mp->ninterface_acoustic,
                                                                                   mp->max_nibool_interfaces_ext_mesh,
                                                                                   mp->d_nibool_interfaces_ext_mesh,
                                                                                   mp->d_ibool_interfaces_ext_mesh,
                                                                                   mp->d_inum_interfaces_acoustic);

      // synchronizes
      //synchronize_cuda();
      // explicitly waits until previous compute stream finishes
      // (hipMemcpy implicitly synchronizes all other cuda operations)
      hipStreamSynchronize(mp->compute_stream);

      print_CUDA_error_if_any(hipMemcpy(send_potential_dot_dot_buffer,mp->d_b_send_potential_dot_dot_buffer,
                                         mp->size_mpi_buffer_potential*sizeof(realw),hipMemcpyDeviceToHost),98001);
    }
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after prepare_boundary_potential_on_device");
#endif


  // finish timing of kernel+memcpy
  // hipEventRecord( stop, 0 );
  // hipEventSynchronize( stop );
  // hipEventElapsedTime( &time, start, stop );
  // hipEventDestroy( start );
  // hipEventDestroy( stop );
  // printf("boundary xfer d->h Time: %f ms\n",time);
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("transfer_boun_pot_from_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// Assembly

/* ----------------------------------------------------------------------------------------------- */


__global__ void assemble_boundary_potential_on_device(realw* d_potential_dot_dot_acoustic,
                                                      realw* d_send_potential_dot_dot_buffer,
                                                      const int ninterface_ac,
                                                      const int max_nibool_interfaces_ext_mesh,
                                                      const int* d_nibool_interfaces_ext_mesh,
                                                      const int* d_ibool_interfaces_ext_mesh,
                                                      const int* inum_inter_acoustic) {

  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;
  int ientry,iglob,num_int;

  for( int iinterface=0; iinterface < ninterface_ac; iinterface++) {
   num_int=inum_inter_acoustic[iinterface]-1;


    if(id<d_nibool_interfaces_ext_mesh[num_int]) {

      // entry in interface array
      ientry = id + max_nibool_interfaces_ext_mesh*num_int;
      // global index in wavefield
      iglob = d_ibool_interfaces_ext_mesh[ientry] - 1;

      // for testing atomic operations against not atomic operations (0.1ms vs. 0.04 ms)
      // d_potential_dot_dot_acoustic[3*(d_ibool_interfaces_ext_mesh[id+max_nibool_interfaces_ext_mesh*iinterface]-1)] +=
      // d_send_potential_dot_dot_buffer[3*(id + max_nibool_interfaces_ext_mesh*iinterface)];
    atomicAdd(&d_potential_dot_dot_acoustic[iglob],d_send_potential_dot_dot_buffer[ientry]);
    }
  }
  // ! This step is done via previous function transfer_and_assemble...
  // ! do iinterface = 1, num_interfaces_ext_mesh
  // !   do ipoin = 1, nibool_interfaces_ext_mesh(iinterface)
  // !     array_val(:,ibool_interfaces_ext_mesh(ipoin,iinterface)) = &
  // !          array_val(:,ibool_interfaces_ext_mesh(ipoin,iinterface)) + buffer_recv_vector_ext_mesh(:,ipoin,iinterface)
  // !   enddo
  // ! enddo
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_asmbl_pot_to_device,
              TRANSFER_ASMBL_POT_TO_DEVICE)(long* Mesh_pointer,
                                            realw* buffer_recv_scalar_ext_mesh,
                                            const int* FORWARD_OR_ADJOINT) {

TRACE("transfer_asmbl_pot_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // Cuda timing
  //hipEvent_t start, stop;
  //start_timing_cuda(&start,&stop);

  // checks if anything to do
  if( mp->size_mpi_buffer_potential > 0 ){


    // assembles on GPU
    int blocksize = BLOCKSIZE_TRANSFER;
    int size_padded = ((int)ceil(((double)mp->max_nibool_interfaces_ext_mesh)/((double)blocksize)))*blocksize;

    int num_blocks_x, num_blocks_y;
    get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

    dim3 grid(num_blocks_x,num_blocks_y);
    dim3 threads(blocksize,1,1);


    // synchronizes
    synchronize_cuda();

    if(*FORWARD_OR_ADJOINT == 1) {
      // copies buffer onto GPU
      print_CUDA_error_if_any(hipMemcpy(mp->d_send_potential_dot_dot_buffer, buffer_recv_scalar_ext_mesh,
                                         mp->size_mpi_buffer_potential*sizeof(realw), hipMemcpyHostToDevice),98010);

      //assemble forward field
      assemble_boundary_potential_on_device<<<grid,threads,0,mp->compute_stream>>>(mp->d_potential_dot_dot_acoustic,
                                                                                    mp->d_send_potential_dot_dot_buffer,
                                                                                    mp->ninterface_acoustic,
                                                                                    mp->max_nibool_interfaces_ext_mesh,
                                                                                    mp->d_nibool_interfaces_ext_mesh,
                                                                                    mp->d_ibool_interfaces_ext_mesh,
                                                                                   mp->d_inum_interfaces_acoustic);


    }
    else if(*FORWARD_OR_ADJOINT == 3) {
      // copies buffer onto GPU
      print_CUDA_error_if_any(hipMemcpy(mp->d_b_send_potential_dot_dot_buffer, buffer_recv_scalar_ext_mesh,
                                         mp->size_mpi_buffer_potential*sizeof(realw), hipMemcpyHostToDevice),98011);

      //assemble reconstructed/backward field
      assemble_boundary_potential_on_device<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_potential_dot_dot_acoustic,
                                                                                    mp->d_b_send_potential_dot_dot_buffer,
                                                                                    mp->ninterface_acoustic,
                                                                                    mp->max_nibool_interfaces_ext_mesh,
                                                                                    mp->d_nibool_interfaces_ext_mesh,
                                                                                    mp->d_ibool_interfaces_ext_mesh,
                                                                                   mp->d_inum_interfaces_acoustic);
    }
  }

  // Cuda timing
  //stop_timing_cuda(&start,&stop,"assemble_boundary_potential_on_device");

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("transfer_asmbl_pot_to_device");
#endif
}

