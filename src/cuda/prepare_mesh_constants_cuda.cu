#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This software is governed by the CeCILL license under French law and
! abiding by the rules of distribution of free software. You can use,
! modify and/or redistribute the software under the terms of the CeCILL
! license as circulated by CEA, CNRS and Inria at the following URL
! "http://www.cecill.info".
!
! As a counterpart to the access to the source code and rights to copy,
! modify and redistribute granted by the license, users are provided only
! with a limited warranty and the software's author, the holder of the
! economic rights, and the successive licensors have only limited
! liability.
!
! In this respect, the user's attention is drawn to the risks associated
! with loading, using, modifying and/or developing or reproducing the
! software by the user in light of its specific status of free software,
! that may mean that it is complicated to manipulate, and that also
! therefore means that it is reserved for developers and experienced
! professionals having in-depth computer knowledge. Users are therefore
! encouraged to load and test the software's suitability as regards their
! requirements in conditions enabling the security of their systems and/or
! data to be ensured and, more generally, to use and operate it in the
! same conditions as regards security.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================

*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#ifdef USE_MPI
#include <mpi.h>
#endif

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"
#include "prepare_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

// helper functions

/* ----------------------------------------------------------------------------------------------- */


// copies integer array from CPU host to GPU device
void copy_todevice_int(void** d_array_addr_ptr,int* h_array,int size){
  TRACE("  copy_todevice_int");

  // allocates memory on GPU
  //
  // note: hipMalloc uses a double-pointer, such that it can return an error code in case it fails
  //          we thus pass the address to the pointer above (as void double-pointer) to have it
  //          pointing to the correct pointer of the array here
  print_CUDA_error_if_any(hipMalloc((void**)d_array_addr_ptr,size*sizeof(int)),
                          12001);

  // copies values onto GPU
  //
  // note: hipMemcpy uses the pointer to the array, we thus re-cast the value of
  //          the double-pointer above to have the correct pointer to the array
  print_CUDA_error_if_any(hipMemcpy((int*) *d_array_addr_ptr,h_array,size*sizeof(int),hipMemcpyHostToDevice),
                          12002);
}

/* ----------------------------------------------------------------------------------------------- */

// copies integer array from CPU host to GPU device
void copy_todevice_realw(void** d_array_addr_ptr,realw* h_array,int size){
  TRACE("  copy_todevice_realw");

  // allocates memory on GPU
  print_CUDA_error_if_any(hipMalloc((void**)d_array_addr_ptr,size*sizeof(realw)),
                          22001);
  // copies values onto GPU
  print_CUDA_error_if_any(hipMemcpy((realw*) *d_array_addr_ptr,h_array,size*sizeof(realw),hipMemcpyHostToDevice),
                          22002);
}


/*
__global__ void check_field(int* ibool,int* nibool, int max_nibool,int num_interfaces_ext_mesh)
{


  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;
  int ientry,iglob;

  for(int iinterface=0; iinterface < num_interfaces_ext_mesh; iinterface++) {
    if(id<nibool[iinterface]) {

      // entry in interface array
      ientry = id + max_nibool*iinterface;
      // global index in wavefield
      iglob = ibool[ientry] - 1;

cuPrintf("valeurs de iglob %d, de l'indice d'entree %d,du nombre de points dans l'interface %d : %d\n", iglob,ientry,iinterface,nibool[iinterface]);
  }}
}*/
/* ----------------------------------------------------------------------------------------------- */

// GPU preparation

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_constants_device,
              PREPARE_CONSTANTS_DEVICE)(long* Mesh_pointer,
                                        int* h_NGLLX, int* NSPEC_AB, int* NGLOB_AB,
                                        realw* h_xix, realw* h_xiz,
                                        realw* h_gammax, realw* h_gammaz,
                                        realw* h_kappav, realw* h_muv,
                                        int* h_ibool,
                                        int* num_interfaces_ext_mesh, int* max_nibool_interfaces_ext_mesh,
                                        int* h_nibool_interfaces_ext_mesh, int* h_ibool_interfaces_ext_mesh,
                                        realw* h_hprime_xx, realw* h_hprimewgll_xx,
                                        realw* h_wxgll,
                                        int* ABSORBING_CONDITIONS,
                                        int* h_nspec_bottom,
                                        int* h_nspec_left,
                                        int* h_nspec_right,
                                        int* h_nspec_top,
                                        int* h_abs_boundary_ispec, int* h_abs_boundary_ij,
                                        realw* h_abs_boundary_normal,
                                        realw* h_abs_boundary_jacobian1Dw,
                                        int* h_num_abs_boundary_faces,
                                        int* h_cote_abs,
                                        int* h_ib_bottom,
                                        int* h_ib_left,
                                        int* h_ib_right,
                                        int* h_ib_top,
                                        int* h_ispec_is_inner,
                                        int* nsources_local_f,
                                        realw* h_sourcearrays, realw * h_source_time_function,
                                        int* NSTEP,
                                        int* h_ispec_selected_source,
                                        int* h_number_receiver_global, int* h_ispec_selected_rec,
                                        int* nrec,int* nrec_local,
                                        realw * h_cosrot,realw * h_sinrot,
                                        int* SIMULATION_TYPE,
                                        int* USE_MESH_COLORING_GPU_f,
                                        int* nspec_acoustic,int* nspec_elastic,
                                        int* h_myrank,
                                        int* SAVE_FORWARD,
                                        realw* h_xir_store, realw* h_gammar_store ) {

  TRACE("prepare_constants_device");

  // allocates mesh parameter structure
  Mesh* mp = (Mesh*) malloc( sizeof(Mesh) );
  if (mp == NULL) exit_on_error("error allocating mesh pointer");
  *Mesh_pointer = (long)mp;

  // checks if NGLLX == 5
  if( *h_NGLLX != NGLLX ){
    exit_on_error("NGLLX must be 5 for CUDA devices");
  }

  // sets processes mpi rank
  mp->myrank = *h_myrank;

  // sets global parameters
  mp->NSPEC_AB = *NSPEC_AB;
  mp->NGLOB_AB = *NGLOB_AB;

  // constants
  mp->simulation_type = *SIMULATION_TYPE;
  mp->absorbing_conditions = *ABSORBING_CONDITIONS;
  mp->save_forward = *SAVE_FORWARD;

  setConst_wxgll(h_wxgll,mp);


  // sets constant arrays
  setConst_hprime_xx(h_hprime_xx,mp);

  // setConst_hprime_zz(h_hprime_zz,mp); // only needed if NGLLX != NGLLY != NGLLZ

  setConst_hprimewgll_xx(h_hprimewgll_xx,mp);

  //setConst_hprimewgll_zz(h_hprimewgll_zz,mp); // only needed if NGLLX != NGLLY != NGLLZ


  // mesh
  // Assuming NGLLX=5. Padded is then 32 (5^2+3)
  int size_padded = NGLL2_PADDED * (mp->NSPEC_AB);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xix, size_padded*sizeof(realw)),1001);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xiz, size_padded*sizeof(realw)),1003);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammax, size_padded*sizeof(realw)),1007);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammaz, size_padded*sizeof(realw)),1009);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_kappav, size_padded*sizeof(realw)),1010);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_muv, size_padded*sizeof(realw)),1011);


  print_CUDA_error_if_any(hipMemcpy2D(mp->d_xix, NGLL2_PADDED*sizeof(realw),
                                       h_xix, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1501);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_xiz, NGLL2_PADDED*sizeof(realw),
                                       h_xiz, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1503);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_gammax, NGLL2_PADDED*sizeof(realw),
                                       h_gammax, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1507);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_gammaz, NGLL2_PADDED*sizeof(realw),
                                       h_gammaz, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1509);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_kappav, NGLL2_PADDED*sizeof(realw),
                                       h_kappav, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1510);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_muv, NGLL2_PADDED*sizeof(realw),
                                       h_muv, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1511);

  // global indexing (padded)
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibool, size_padded*sizeof(int)),1600);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_ibool, NGLL2_PADDED*sizeof(int),
                                       h_ibool, NGLL2*sizeof(int), NGLL2*sizeof(int),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1601);


  // prepare interprocess-edge exchange information
  mp->num_interfaces_ext_mesh = *num_interfaces_ext_mesh;
  mp->max_nibool_interfaces_ext_mesh = *max_nibool_interfaces_ext_mesh;
  if( mp->num_interfaces_ext_mesh > 0 ){
    copy_todevice_int((void**)&mp->d_nibool_interfaces_ext_mesh,h_nibool_interfaces_ext_mesh,
                      mp->num_interfaces_ext_mesh);
    copy_todevice_int((void**)&mp->d_ibool_interfaces_ext_mesh,h_ibool_interfaces_ext_mesh,
                      (mp->num_interfaces_ext_mesh)*(mp->max_nibool_interfaces_ext_mesh));


  int blocksize = BLOCKSIZE_TRANSFER;
    int size_padded = ((int)ceil(((double)(mp->max_nibool_interfaces_ext_mesh))/((double)blocksize)))*blocksize;


  }

  hipStreamCreate(&mp->compute_stream);
  // copy stream (needed to transfer mpi buffers)
  if( mp->num_interfaces_ext_mesh * mp->max_nibool_interfaces_ext_mesh > 0 ){
    hipStreamCreate(&mp->copy_stream);
  }


  // inner elements
  copy_todevice_int((void**)&mp->d_ispec_is_inner,h_ispec_is_inner,mp->NSPEC_AB);

  // absorbing boundaries
  mp->d_num_abs_boundary_faces = *h_num_abs_boundary_faces;
  if( mp->absorbing_conditions && mp->d_num_abs_boundary_faces > 0 ){
    copy_todevice_int((void**)&mp->d_abs_boundary_ispec,h_abs_boundary_ispec,mp->d_num_abs_boundary_faces);
    copy_todevice_int((void**)&mp->d_abs_boundary_ijk,h_abs_boundary_ij,
                      2*NGLL*(mp->d_num_abs_boundary_faces));
    copy_todevice_realw((void**)&mp->d_abs_boundary_normal,h_abs_boundary_normal,
                        NDIM*NGLL*(mp->d_num_abs_boundary_faces));
    copy_todevice_realw((void**)&mp->d_abs_boundary_jacobian2Dw,h_abs_boundary_jacobian1Dw,
                        NGLL*(mp->d_num_abs_boundary_faces));
    copy_todevice_int((void**)&mp->d_cote_abs,h_cote_abs,(mp->d_num_abs_boundary_faces));
    copy_todevice_int((void**)&mp->d_ib_left,h_ib_left,(mp->d_num_abs_boundary_faces));
    copy_todevice_int((void**)&mp->d_ib_right,h_ib_right,(mp->d_num_abs_boundary_faces));
    copy_todevice_int((void**)&mp->d_ib_top,h_ib_top,(mp->d_num_abs_boundary_faces));
    copy_todevice_int((void**)&mp->d_ib_bottom,h_ib_bottom,(mp->d_num_abs_boundary_faces));
      mp->d_nspec_bottom = *h_nspec_bottom;
      mp->d_nspec_left = *h_nspec_left;
      mp->d_nspec_right = *h_nspec_right;
      mp->d_nspec_top = *h_nspec_top;

  }

  // sources
  mp->nsources_local = *nsources_local_f;



  if( mp->nsources_local > 0){
    copy_todevice_realw((void**)&mp->d_source_time_function,h_source_time_function,(*NSTEP)*(mp->nsources_local));
    copy_todevice_realw((void**)&mp->d_sourcearrays,h_sourcearrays,mp->nsources_local*NDIM*NGLL2);
    copy_todevice_int((void**)&mp->d_ispec_selected_source,h_ispec_selected_source,mp->nsources_local);
    }



  // receiver stations
  mp->nrec_local = *nrec_local; // number of receiver located in this partition
  // note that:
  // size(number_receiver_global) = nrec_local
  // size(ispec_selected_rec) = nrec
  if( mp->nrec_local > 0 ){
    copy_todevice_int((void**)&mp->d_number_receiver_global,h_number_receiver_global,mp->nrec_local);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_seismograms,(mp->nrec_local)*sizeof(realw)*2),1303);
    print_CUDA_error_if_any(hipHostMalloc((void**)&(mp->h_seismograms),sizeof(float)*(mp->nrec_local)*2),8004);
    copy_todevice_realw((void**)&mp->d_cosrot,h_cosrot,mp->nrec_local);
    copy_todevice_realw((void**)&mp->d_sinrot,h_sinrot,mp->nrec_local);
  }
  copy_todevice_int((void**)&mp->d_ispec_selected_rec,h_ispec_selected_rec,(*nrec));


#ifdef USE_MESH_COLORING_GPU
  mp->use_mesh_coloring_gpu = 1;
  if( ! *USE_MESH_COLORING_GPU_f ) exit_on_error("error with USE_MESH_COLORING_GPU constant; please re-compile\n");
#else
  // mesh coloring
  // note: this here passes the coloring as an option to the kernel routines
  //          the performance seems to be the same if one uses the pre-processing directives above or not
  mp->use_mesh_coloring_gpu = *USE_MESH_COLORING_GPU_f;
#endif

  // number of elements per domain
  mp->nspec_acoustic = *nspec_acoustic;
  mp->nspec_elastic = *nspec_elastic;

  copy_todevice_realw((void**)&mp->d_xir_store_loc,h_xir_store,(*nrec_local)*NGLLX);

  copy_todevice_realw((void**)&mp->d_gammar_store_loc,h_gammar_store,(*nrec_local)*NGLLX);



  // JC JC here we will need to add GPU support for the new C-PML routines

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_constants_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// for ACOUSTIC simulations

/* ----------------------------------------------------------------------------------------------- */





extern "C"
void FC_FUNC_(prepare_fields_acoustic_device,
              PREPARE_FIELDS_ACOUSTIC_DEVICE)(long* Mesh_pointer,
                                              realw* rmass_acoustic, realw* rhostore, realw* kappastore,
                                              int* num_phase_ispec_acoustic, int* phase_ispec_inner_acoustic,
                                              int* ispec_is_acoustic,
                                              int* num_free_surface_faces,
                                              int* free_surface_ispec,
                                              int* free_surface_ijk,
                                              int* ELASTIC_SIMULATION,
                                              int* num_coupling_ac_el_faces,
                                              int* coupling_ac_el_ispec,
                                              int* coupling_ac_el_ijk,
                                              realw* coupling_ac_el_normal,
                                              realw* coupling_ac_el_jacobian2Dw,
                                              int * h_ninterface_acoustic,int * h_inum_interfaces_acoustic,
                                              int* num_colors_outer_acoustic,
                                              int* num_colors_inner_acoustic,
                                              int* num_elem_colors_acoustic) {

  TRACE("prepare_fields_acoustic_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer);

  // allocates arrays on device (GPU)
  int size = mp->NGLOB_AB;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_acoustic),sizeof(realw)*size),2001);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_dot_acoustic),sizeof(realw)*size),2002);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_dot_dot_acoustic),sizeof(realw)*size),2003);
  // initializes values to zero
  //print_CUDA_error_if_any(hipMemset(mp->d_potential_acoustic,0,sizeof(realw)*size),2007);
  //print_CUDA_error_if_any(hipMemset(mp->d_potential_dot_acoustic,0,sizeof(realw)*size),2007);
  //print_CUDA_error_if_any(hipMemset(mp->d_potential_dot_dot_acoustic,0,sizeof(realw)*size),2007);

  // mpi buffer
  mp->size_mpi_buffer_potential = (mp->num_interfaces_ext_mesh) * (mp->max_nibool_interfaces_ext_mesh);
  if( mp->size_mpi_buffer_potential > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_send_potential_dot_dot_buffer),mp->size_mpi_buffer_potential *sizeof(realw)),2004);
  }

  // mass matrix
  copy_todevice_realw((void**)&mp->d_rmass_acoustic,rmass_acoustic,mp->NGLOB_AB);

  // density
  // padded array
  // Assuming NGLLX==5. Padded is then 32 (5^2+3)
  int size_padded = NGLL2_PADDED * mp->NSPEC_AB;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rhostore),size_padded*sizeof(realw)),2006);
  // transfer constant element data with padding

  print_CUDA_error_if_any(hipMemcpy2D(mp->d_rhostore, NGLL2_PADDED*sizeof(realw),
                                       rhostore, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),2106);


  // non-padded array
  copy_todevice_realw((void**)&mp->d_kappastore,kappastore,NGLL2*mp->NSPEC_AB);

  // phase elements
  mp->num_phase_ispec_acoustic = *num_phase_ispec_acoustic;
  copy_todevice_int((void**)&mp->d_phase_ispec_inner_acoustic,phase_ispec_inner_acoustic,
                    2*mp->num_phase_ispec_acoustic);
  copy_todevice_int((void**)&mp->d_ispec_is_acoustic,ispec_is_acoustic,mp->NSPEC_AB);


    // allocate surface arrays
    mp->num_free_surface_faces = *num_free_surface_faces;
    if( mp->num_free_surface_faces > 0 ){
      copy_todevice_int((void**)&mp->d_free_surface_ispec,free_surface_ispec,mp->num_free_surface_faces);
      copy_todevice_int((void**)&mp->d_free_surface_ijk,free_surface_ijk,
                        2*NGLLX*mp->num_free_surface_faces);

  }

  // absorbing boundaries
  if( mp->absorbing_conditions && mp->d_num_abs_boundary_faces > 0 ){
    // absorb_field array used for file i/o
    if(mp->simulation_type == 3 || ( mp->simulation_type == 1 && mp->save_forward )){
      print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_potential_left,mp->d_nspec_left*sizeof(realw)*NGLLX),2201);
      print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_potential_right,mp->d_nspec_right*sizeof(realw)*NGLLX),2201);
      print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_potential_top,mp->d_nspec_top*sizeof(realw)*NGLLX),2201);
      print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_potential_bottom,mp->d_nspec_bottom*sizeof(realw)*NGLLX),2201);

    }
  }


  // coupling with elastic parts
  if( *ELASTIC_SIMULATION && *num_coupling_ac_el_faces > 0 ){
    copy_todevice_int((void**)&mp->d_coupling_ac_el_ispec,coupling_ac_el_ispec,(*num_coupling_ac_el_faces));
    copy_todevice_int((void**)&mp->d_coupling_ac_el_ijk,coupling_ac_el_ijk,2*NGLL*(*num_coupling_ac_el_faces));
    copy_todevice_realw((void**)&mp->d_coupling_ac_el_normal,coupling_ac_el_normal,
                        2*NGLL*(*num_coupling_ac_el_faces));
    copy_todevice_realw((void**)&mp->d_coupling_ac_el_jacobian2Dw,coupling_ac_el_jacobian2Dw,
                        NGLL*(*num_coupling_ac_el_faces));
  }

  // mesh coloring
  if( mp->use_mesh_coloring_gpu ){
    mp->num_colors_outer_acoustic = *num_colors_outer_acoustic;
    mp->num_colors_inner_acoustic = *num_colors_inner_acoustic;
    mp->h_num_elem_colors_acoustic = (int*) num_elem_colors_acoustic;
  }

  mp->ninterface_acoustic = *h_ninterface_acoustic;
  copy_todevice_int((void**)&mp->d_inum_interfaces_acoustic,h_inum_interfaces_acoustic,mp->num_interfaces_ext_mesh);



#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_acoustic_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_acoustic_adj_dev,
              PREPARE_FIELDS_ACOUSTIC_ADJ_DEV)(long* Mesh_pointer,
                                              int* APPROXIMATE_HESS_KL) {

  TRACE("prepare_fields_acoustic_adj_dev");

  Mesh* mp = (Mesh*)(*Mesh_pointer);

  // kernel simulations
  if( mp->simulation_type != 3 ) return;

  // allocates backward/reconstructed arrays on device (GPU)
  int size = mp->NGLOB_AB;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_acoustic),sizeof(realw)*size),3014);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_dot_acoustic),sizeof(realw)*size),3015);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_dot_dot_acoustic),sizeof(realw)*size),3016);
  // initializes values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_b_potential_acoustic,0,sizeof(realw)*size),3007);
  print_CUDA_error_if_any(hipMemset(mp->d_b_potential_dot_acoustic,0,sizeof(realw)*size),3007);
  print_CUDA_error_if_any(hipMemset(mp->d_b_potential_dot_dot_acoustic,0,sizeof(realw)*size),3007);



  // allocates kernels
  size = NGLL2*mp->NSPEC_AB;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_ac_kl),size*sizeof(realw)),3017);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_kappa_ac_kl),size*sizeof(realw)),3018);
  // initializes kernel values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_rho_ac_kl,0,size*sizeof(realw)),3019);
  print_CUDA_error_if_any(hipMemset(mp->d_kappa_ac_kl,0,size*sizeof(realw)),3020);

  // preconditioner
  if( *APPROXIMATE_HESS_KL ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_ac_kl),size*sizeof(realw)),3030);
    // initializes with zeros
    print_CUDA_error_if_any(hipMemset(mp->d_hess_ac_kl,0,size*sizeof(realw)),3031);
  }

  // mpi buffer
  if( mp->size_mpi_buffer_potential > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_send_potential_dot_dot_buffer),mp->size_mpi_buffer_potential*sizeof(realw)),3014);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_acoustic_adj_dev");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// for ELASTIC simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_elastic_device,
              PREPARE_FIELDS_ELASTIC_DEVICE)(long* Mesh_pointer,
                                             realw* rmassx, realw* rmassz,
                                             realw* rho_vp, realw* rho_vs,
                                             int* num_phase_ispec_elastic,
                                             int* phase_ispec_inner_elastic,
                                             int* ispec_is_elastic,
                                             int* h_nspec_left,
                                             int* h_nspec_right,
                                             int* h_nspec_top,
                                             int* h_nspec_bottom,
                                             int* ACOUSTIC_SIMULATION,
                                             int* num_colors_outer_elastic,
                                             int* num_colors_inner_elastic,
                                             int* num_elem_colors_elastic,
                                             int* ANISOTROPY,
                                             realw *c11store,realw *c12store,realw *c13store,
                                             realw *c15store,
                                             realw *c23store,
                                             realw *c25store,realw *c33store,
                                             realw *c35store,
                                             realw *c55store,int* h_ninterface_elastic,int * h_inum_interfaces_elastic ){



  TRACE("prepare_fields_elastic_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer);
  int size;


  // debug
  //printf("prepare_fields_elastic_device: rank %d - wavefield setup\n",mp->myrank);
  //synchronize_mpi();

  // elastic wavefields
  size = NDIM * mp->NGLOB_AB;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_displ),sizeof(realw)*size),4001);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_veloc),sizeof(realw)*size),4002);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_accel),sizeof(realw)*size),4003);
  // initializes values to zero
  //print_CUDA_error_if_any(hipMemset(mp->d_displ,0,sizeof(realw)*size),4007);
  //print_CUDA_error_if_any(hipMemset(mp->d_veloc,0,sizeof(realw)*size),4007);
  //print_CUDA_error_if_any(hipMemset(mp->d_accel,0,sizeof(realw)*size),4007);


  // debug
  //synchronize_mpi();

  // MPI buffer
  mp->size_mpi_buffer = NDIM * (mp->num_interfaces_ext_mesh) * (mp->max_nibool_interfaces_ext_mesh);
  if( mp->size_mpi_buffer > 0 ){
    // note: Allocate pinned mpi-buffers.
    //       MPI buffers use pinned memory allocated by hipHostMalloc, which
    //       enables the use of asynchronous memory copies from host <-> device
    // send buffer
    print_CUDA_error_if_any(hipHostMalloc((void**)&(mp->h_send_accel_buffer),sizeof(float)*(mp->size_mpi_buffer)),8004);
    //mp->send_buffer = (float*)malloc((mp->size_mpi_buffer)*sizeof(float));
    // adjoint
    //print_CUDA_error_if_any(hipHostMalloc((void**)&(mp->h_send_b_accel_buffer),sizeof(float)*(mp->size_mpi_buffer)),8004);
    // mp->b_send_buffer = (float*)malloc((size_mpi_buffer)*sizeof(float));
    // receive buffer
    print_CUDA_error_if_any(hipHostMalloc((void**)&(mp->h_recv_accel_buffer),sizeof(float)*(mp->size_mpi_buffer)),8004);
    mp->recv_buffer = (float*)malloc((mp->size_mpi_buffer)*sizeof(float));

    // non-pinned buffer
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_send_accel_buffer),mp->size_mpi_buffer*sizeof(realw)),4004);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_recv_accel_buffer),mp->size_mpi_buffer*sizeof(realw)),4004);


    // adjoint
    if( mp->simulation_type == 3 ){
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_send_accel_buffer),mp->size_mpi_buffer*sizeof(realw)),4004);
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_recv_accel_buffer),mp->size_mpi_buffer*sizeof(realw)),4004);
    }
  }

  // debug
  //printf("prepare_fields_elastic_device: rank %d - mass matrix\n",mp->myrank);
  //synchronize_mpi();

  // mass matrix
  copy_todevice_realw((void**)&mp->d_rmassx,rmassx,mp->NGLOB_AB);
  copy_todevice_realw((void**)&mp->d_rmassz,rmassz,mp->NGLOB_AB);

  // element indices
  copy_todevice_int((void**)&mp->d_ispec_is_elastic,ispec_is_elastic,mp->NSPEC_AB);

  // phase elements
  mp->num_phase_ispec_elastic = *num_phase_ispec_elastic;

  copy_todevice_int((void**)&mp->d_phase_ispec_inner_elastic,phase_ispec_inner_elastic,2*mp->num_phase_ispec_elastic);

  // debug
  //synchronize_mpi();



  // debug
  //synchronize_mpi();

  // absorbing conditions
  if( mp->absorbing_conditions && mp->d_num_abs_boundary_faces > 0){

    // debug
    //printf("prepare_fields_elastic_device: rank %d - absorbing boundary setup\n",mp->myrank);

    // non-padded arrays
    // rho_vp, rho_vs non-padded; they are needed for stacey boundary condition
    copy_todevice_realw((void**)&mp->d_rho_vp,rho_vp,NGLL2*mp->NSPEC_AB);
    copy_todevice_realw((void**)&mp->d_rho_vs,rho_vs,NGLL2*mp->NSPEC_AB);

    // absorb_field array used for file i/o

  if( mp->absorbing_conditions && mp->d_num_abs_boundary_faces > 0 ){
    // absorb_field array used for file i/o
    if(mp->simulation_type == 3 || ( mp->simulation_type == 1 && mp->save_forward )){
      mp->d_nspec_left = *h_nspec_left;
      print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_elastic_left,2*mp->d_nspec_left*sizeof(realw)*NGLLX),2201);

      mp->d_nspec_right = *h_nspec_right;
      print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_elastic_right,2*mp->d_nspec_right*sizeof(realw)*NGLLX),2201);

      mp->d_nspec_top = *h_nspec_top;
      print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_elastic_top,2*mp->d_nspec_top*sizeof(realw)*NGLLX),2201);

      mp->d_nspec_bottom = *h_nspec_bottom;
      print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_elastic_bottom,2*mp->d_nspec_bottom*sizeof(realw)*NGLLX),2201);


    }
  }
  }

  // debug
  //synchronize_mpi();

  // anisotropy
  if( *ANISOTROPY ){
    // debug
    //printf("prepare_fields_elastic_device: rank %d - attenuation setup\n",mp->myrank);
    //synchronize_mpi();

    // Assuming NGLLX==5. Padded is then 32 (5^2+3)
    int size_padded = NGLL2_PADDED * (mp->NSPEC_AB);

    // allocates memory on GPU
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c11store),size_padded*sizeof(realw)),4700);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c12store),size_padded*sizeof(realw)),4701);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c13store),size_padded*sizeof(realw)),4702);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c15store),size_padded*sizeof(realw)),4704);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c23store),size_padded*sizeof(realw)),4707);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c25store),size_padded*sizeof(realw)),4709);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c33store),size_padded*sizeof(realw)),4711);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c35store),size_padded*sizeof(realw)),4711);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c55store),size_padded*sizeof(realw)),4718);



    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c11store, NGLL2_PADDED*sizeof(realw),
                                         c11store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c12store, NGLL2_PADDED*sizeof(realw),
                                         c12store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c13store, NGLL2_PADDED*sizeof(realw),
                                         c13store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c15store, NGLL2_PADDED*sizeof(realw),
                                         c15store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c23store, NGLL2_PADDED*sizeof(realw),
                                         c23store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c25store, NGLL2_PADDED*sizeof(realw),
                                         c25store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c33store, NGLL2_PADDED*sizeof(realw),
                                         c33store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c35store, NGLL2_PADDED*sizeof(realw),
                                         c35store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c55store, NGLL2_PADDED*sizeof(realw),
                                         c55store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);

  }


  // mesh coloring
  if( mp->use_mesh_coloring_gpu ){
    mp->num_colors_outer_elastic = *num_colors_outer_elastic;
    mp->num_colors_inner_elastic = *num_colors_inner_elastic;
    mp->h_num_elem_colors_elastic = (int*) num_elem_colors_elastic;
  }


  mp->ninterface_elastic = *h_ninterface_elastic;
  copy_todevice_int((void**)&mp->d_inum_interfaces_elastic,h_inum_interfaces_elastic,mp->num_interfaces_ext_mesh);

  // JC JC here we will need to add GPU support for the new C-PML routines

  // debug
  //printf("prepare_fields_elastic_device: rank %d - done\n",mp->myrank);
  //synchronize_mpi();

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_elastic_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_elastic_adj_dev,
              PREPARE_FIELDS_ELASTIC_ADJ_DEV)(long* Mesh_pointer,
                                             int* size_f,
                                             int* APPROXIMATE_HESS_KL){

  TRACE("prepare_fields_elastic_adj_dev");

  Mesh* mp = (Mesh*)(*Mesh_pointer);
  int size;

  // checks if kernel simulation
  if( mp->simulation_type != 3 ) return;

  // kernel simulations
  // debug
  //printf("prepare_fields_elastic_adj_dev: rank %d - kernel setup\n",mp->myrank);
  //synchronize_mpi();

  // backward/reconstructed wavefields
  // allocates backward/reconstructed arrays on device (GPU)
  size = *size_f;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_displ),sizeof(realw)*size),5201);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_veloc),sizeof(realw)*size),5202);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_accel),sizeof(realw)*size),5203);
  // initializes values to zero
  //print_CUDA_error_if_any(hipMemset(mp->d_b_displ,0,sizeof(realw)*size),5207);
  //print_CUDA_error_if_any(hipMemset(mp->d_b_veloc,0,sizeof(realw)*size),5207);
  //print_CUDA_error_if_any(hipMemset(mp->d_b_accel,0,sizeof(realw)*size),5207);


  // anisotropic/isotropic kernels
  // debug
  //printf("prepare_fields_elastic_adj_dev: rank %d -  anisotropic/isotropic kernels\n",mp->myrank);
  //synchronize_mpi();

  // allocates kernels
  size = NGLL2 * mp->NSPEC_AB; // note: non-aligned; if align, check memcpy below and indexing
  // density kernel
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_kl),size*sizeof(realw)),5204);
  // initializes kernel values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_rho_kl,0,size*sizeof(realw)),5214);


    // isotropic kernels
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_mu_kl),size*sizeof(realw)),5206);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_kappa_kl),size*sizeof(realw)),5207);
    print_CUDA_error_if_any(hipMemset(mp->d_mu_kl,0,size*sizeof(realw)),5216);
    print_CUDA_error_if_any(hipMemset(mp->d_kappa_kl,0,size*sizeof(realw)),5217);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_dsxx),size*sizeof(realw)),5207);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_dsxz),size*sizeof(realw)),5207);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_dszz),size*sizeof(realw)),5207);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_dsxx),size*sizeof(realw)),5207);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_dsxz),size*sizeof(realw)),5207);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_dszz),size*sizeof(realw)),5207);

  // approximate hessian kernel
  if( *APPROXIMATE_HESS_KL ){
    // debug
    //printf("prepare_fields_elastic_adj_dev: rank %d - hessian kernel\n",mp->myrank);
    //synchronize_mpi();

    size = NGLL2 * mp->NSPEC_AB; // note: non-aligned; if align, check memcpy below and indexing
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_el_kl),size*sizeof(realw)),5450);
    print_CUDA_error_if_any(hipMemset(mp->d_hess_el_kl,0,size*sizeof(realw)),5451);
  }

  // debug
  //printf("prepare_fields_elastic_adj_dev: rank %d - done\n",mp->myrank);
  //synchronize_mpi();

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_elastic_adj_dev");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// purely adjoint & kernel simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_sim2_or_3_const_device,
              PREPARE_SIM2_OR_3_CONST_DEVICE)(long* Mesh_pointer,
                                              int* islice_selected_rec,
                                              int* nadj_rec_local,
                                              int* nrec,realw* h_source_adjointe,int* NSTEP) {

  TRACE("prepare_sim2_or_3_const_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer);

  // adjoint source arrays
  mp->nadj_rec_local = *nadj_rec_local;
  if( mp->nadj_rec_local > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_adj_sourcearrays,
                                       (mp->nadj_rec_local)*2*NGLL2*sizeof(realw)),6003);

    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_pre_computed_irec,
                                       (mp->nadj_rec_local)*sizeof(int)),6004);

    // prepares local irec array:
    // the irec_local variable needs to be precomputed (as
    // h_pre_comp..), because normally it is in the loop updating accel,
    // and due to how it's incremented, it cannot be parallelized
    int* h_pre_computed_irec = (int*) malloc( (mp->nadj_rec_local)*sizeof(int) );
    if( h_pre_computed_irec == NULL ) exit_on_error("prepare_sim2_or_3_const_device: h_pre_computed_irec not allocated\n");

    int irec_local = 0;
    for(int irec = 0; irec < *nrec; irec++) {
      if(mp->myrank == islice_selected_rec[irec]) {
        irec_local++;
        h_pre_computed_irec[irec_local-1] = irec;
      }
    }
    // checks if all local receivers have been found
    if( irec_local != mp->nadj_rec_local ) exit_on_error("prepare_sim2_or_3_const_device: irec_local not equal\n");

    // copies values onto GPU
    print_CUDA_error_if_any(hipMemcpy(mp->d_pre_computed_irec,h_pre_computed_irec,
                                       (mp->nadj_rec_local)*sizeof(int),hipMemcpyHostToDevice),6010);
    free(h_pre_computed_irec);

    copy_todevice_realw((void**)&mp->d_source_adjointe,h_source_adjointe,(*NSTEP)*(*nadj_rec_local)*NDIM);




  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_sim2_or_3_const_device");
#endif
}



/* ----------------------------------------------------------------------------------------------- */

// cleanup

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_cleanup_device,
              PREPARE_CLEANUP_DEVICE)(long* Mesh_pointer,
                                      int* ACOUSTIC_SIMULATION,
                                      int* ELASTIC_SIMULATION,
                                      int* ABSORBING_CONDITIONS,
                                      int* ANISOTROPY,
                                      int* APPROXIMATE_HESS_KL) {


TRACE("prepare_cleanup_device");

  // frees allocated memory arrays
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  // frees memory on GPU
  // mesh
  hipFree(mp->d_xix);
  hipFree(mp->d_xiz);
  hipFree(mp->d_gammax);
  hipFree(mp->d_gammaz);
  hipFree(mp->d_muv);

  // absorbing boundaries
  if( *ABSORBING_CONDITIONS && mp->d_num_abs_boundary_faces > 0 ){
    hipFree(mp->d_abs_boundary_ispec);
    hipFree(mp->d_abs_boundary_ijk);
    hipFree(mp->d_abs_boundary_normal);
    hipFree(mp->d_abs_boundary_jacobian2Dw);
    hipFree(mp->d_cote_abs);
    hipFree(mp->d_ib_left);
    hipFree(mp->d_ib_right);
    hipFree(mp->d_ib_top);
    hipFree(mp->d_ib_bottom);
  }

  // interfaces
  if( mp->num_interfaces_ext_mesh > 0 ){
    hipFree(mp->d_nibool_interfaces_ext_mesh);
    hipFree(mp->d_ibool_interfaces_ext_mesh);
  }

  // global indexing
  hipFree(mp->d_ispec_is_inner);
  hipFree(mp->d_ibool);

  // sources
  if( mp->nsources_local > 0){
    hipFree(mp->d_sourcearrays);
    hipFree(mp->d_source_time_function);
  hipFree(mp->d_ispec_selected_source);
  }



  // receivers
  if( mp->nrec_local > 0 ){
  hipFree(mp->d_number_receiver_global);hipFree(mp->d_seismograms);
  hipFree(mp->d_cosrot),hipFree(mp->d_sinrot);
  }

  hipFree(mp->d_ispec_selected_rec);

  hipFree(mp->d_gammar_store_loc);
  hipFree(mp->d_xir_store_loc);

  // ACOUSTIC arrays
  if( *ACOUSTIC_SIMULATION ){
    hipFree(mp->d_potential_acoustic);
    hipFree(mp->d_potential_dot_acoustic);
    hipFree(mp->d_potential_dot_dot_acoustic);
    hipFree(mp->d_send_potential_dot_dot_buffer);
    hipFree(mp->d_rmass_acoustic);
    hipFree(mp->d_rhostore);
    hipFree(mp->d_kappastore);
    hipFree(mp->d_phase_ispec_inner_acoustic);
    hipFree(mp->d_ispec_is_acoustic);
    hipFree(mp->d_inum_interfaces_acoustic);

    if( mp->simulation_type == 3 ) {
      hipFree(mp->d_b_potential_acoustic);
      hipFree(mp->d_b_potential_dot_acoustic);
      hipFree(mp->d_b_potential_dot_dot_acoustic);
      hipFree(mp->d_rho_ac_kl);
      hipFree(mp->d_kappa_ac_kl);
      if( *APPROXIMATE_HESS_KL) hipFree(mp->d_hess_ac_kl);
      if( mp->size_mpi_buffer_potential > 0 ) hipFree(mp->d_b_send_potential_dot_dot_buffer);
  }


 if( *ABSORBING_CONDITIONS && mp->d_num_abs_boundary_faces > 0){
 if(mp->simulation_type == 3 || ( mp->simulation_type == 1 && mp->save_forward )){
      hipFree(mp->d_b_absorb_potential_bottom);
      hipFree(mp->d_b_absorb_potential_left);
      hipFree(mp->d_b_absorb_potential_right);
      hipFree(mp->d_b_absorb_potential_top); }}

  } // ACOUSTIC_SIMULATION

  // ELASTIC arrays
  if( *ELASTIC_SIMULATION ){
    hipFree(mp->d_displ);
    hipFree(mp->d_veloc);
    hipFree(mp->d_accel);

    hipFree(mp->d_send_accel_buffer);
    if( mp->simulation_type == 3) hipFree(mp->d_b_send_accel_buffer);

    hipFree(mp->d_rmassx);
    hipFree(mp->d_rmassz);

    hipFree(mp->d_phase_ispec_inner_elastic);
    hipFree(mp->d_ispec_is_elastic);
    hipFree(mp->d_inum_interfaces_elastic);



    if( *ABSORBING_CONDITIONS && mp->d_num_abs_boundary_faces > 0){
      hipFree(mp->d_rho_vp);
      hipFree(mp->d_rho_vs);
      hipFree(mp->d_b_absorb_elastic_bottom);
      hipFree(mp->d_b_absorb_elastic_left);
      hipFree(mp->d_b_absorb_elastic_right);
      hipFree(mp->d_b_absorb_elastic_top);

    }

    if( mp->simulation_type == 3 ) {
      hipFree(mp->d_b_displ);
      hipFree(mp->d_b_veloc);
      hipFree(mp->d_b_accel);
      hipFree(mp->d_rho_kl);

        hipFree(mp->d_mu_kl);
        hipFree(mp->d_kappa_kl);
        hipFree(mp->d_b_dsxx);
        hipFree(mp->d_b_dsxz);
        hipFree(mp->d_b_dszz);
        hipFree(mp->d_dsxx);
        hipFree(mp->d_dsxz);
        hipFree(mp->d_dszz);

      if( *APPROXIMATE_HESS_KL ) hipFree(mp->d_hess_el_kl);
    }

    if( *ANISOTROPY ){
      hipFree(mp->d_c11store);
      hipFree(mp->d_c12store);
      hipFree(mp->d_c13store);
      hipFree(mp->d_c15store);
      hipFree(mp->d_c23store);
      hipFree(mp->d_c25store);
      hipFree(mp->d_c33store);
      hipFree(mp->d_c35store);
      hipFree(mp->d_c55store);

    }


  } // ELASTIC_SIMULATION

  // purely adjoint & kernel array
  if(mp->simulation_type == 3 ){
    if(mp->nadj_rec_local > 0 ){
      hipFree(mp->d_adj_sourcearrays);
      hipFree(mp->d_pre_computed_irec);
    }
  }


  // mesh pointer - not needed anymore
  free(mp);
}
