#include "hip/hip_runtime.h"
/*
Copyright 2009 NVIDIA Corporation. All rights reserved.

NOTICE TO LICENSEE:

This source code and/or documentation ("Licensed Deliverables") are subject
to NVIDIA intellectual property rights under U.S. and international Copyright
laws.

These Licensed Deliverables contained herein is PROPRIETARY and CONFIDENTIAL
to NVIDIA and is being provided under the terms and conditions of a form of
NVIDIA software license agreement by and between NVIDIA and Licensee ("License
Agreement") or electronically accepted by Licensee. Notwithstanding any terms
or conditions to the contrary in the License Agreement, reproduction or
disclosure of the Licensed Deliverables to any third party without the express
written consent of NVIDIA is prohibited.

NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE LICENSE AGREEMENT,
NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THESE LICENSED
DELIVERABLES FOR ANY PURPOSE. IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED
WARRANTY OF ANY KIND. NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE
LICENSED DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE. NOTWITHSTANDING ANY
TERMS OR CONDITIONS TO THE CONTRARY IN THE LICENSE AGREEMENT, IN NO EVENT SHALL
NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES,
OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS, WHETHER
IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS ACTION, ARISING OUT OF
OR IN CONNECTION WITH THE USE OR PERFORMANCE OF THESE LICENSED DELIVERABLES.

U.S. Government End Users. These Licensed Deliverables are a "commercial item"
as that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting of
"commercial computer software" and "commercial computer software documentation"
as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) and is provided to the
U.S. Government only as a commercial end item. Consistent with 48 C.F.R.12.212
and 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all U.S. Government
End Users acquire the Licensed Deliverables with only those rights set forth
herein.

Any use of the Licensed Deliverables in individual and commercial software must
include, in the user documentation and internal comments to the code, the above
Disclaimer and U.S. Government End Users Notice.
*/

/*
* cuPrintf.cu
*
* This is a printf command callable from within a kernel. It is set
* up so that output is sent to a memory buffer, which is emptied from
* the host side - but only after a hipDeviceSynchronize() on the host.
*
* Currently, there is a limitation of around 200 characters of output
* and no more than 10 arguments to a single cuPrintf() call. Issue
* multiple calls if longer format strings are required.
*
* It requires minimal setup, and is *NOT* optimised for performance.
* For example, writes are not coalesced - this is because there is an
* assumption that people will not want to printf from every single one
* of thousands of threads, but only from individual threads at a time.
*
* Using this is simple - it requires one host-side call to initialise
* everything, and then kernels can call cuPrintf at will. Sample code
* is the easiest way to demonstrate:
*
#include "cuPrintf.cu"

__global__ void testKernel(int val)
{
cuPrintf("Value is: %d\n", val);
}

int main()
{
cudaPrintfInit();
testKernel<<< 2, 3 >>>(10);
cudaPrintfDisplay(stdout, true);
cudaPrintfEnd();
return 0;
}
*
* See the header file, "cuPrintf.cuh" for more info, especially
* arguments to cudaPrintfInit() and cudaPrintfDisplay();
*/

#ifndef CUPRINTF_CU
#define CUPRINTF_CU

#include "cuPrintf.cuh"
#if __CUDA_ARCH__ > 100 // Atomics only used with > sm_10 architecture
#include <sm_11_atomic_functions.h>
#endif

// This is the smallest amount of memory, per-thread, which is allowed.
// It is also the largest amount of space a single printf() can take up
const static int CUPRINTF_MAX_LEN = 256;

// This structure is used internally to track block/thread output restrictions.
typedef struct __align__(8) {
int threadid; // CUPRINTF_UNRESTRICTED for unrestricted
int blockid; // CUPRINTF_UNRESTRICTED for unrestricted
} cuPrintfRestriction;

// The main storage is in a global print buffer, which has a known
// start/end/length. These are atomically updated so it works as a
// circular buffer.
// Since the only control primitive that can be used is atomicAdd(),
// we cannot wrap the pointer as such. The actual address must be
// calculated from printfBufferPtr by mod-ing with printfBufferLength.
// For sm_10 architecture, we must subdivide the buffer per-thread
// since we do not even have an atomic primitive.
__constant__ static char *globalPrintfBuffer = NULL; // Start of circular buffer (set up by host)
__constant__ static int printfBufferLength = 0; // Size of circular buffer (set up by host)
__device__ static cuPrintfRestriction restrictRules; // Output restrictions
__device__ volatile static char *printfBufferPtr = NULL; // Current atomically-incremented non-wrapped offset

// This is the header preceeding all printf entries.
// NOTE: It *must* be size-aligned to the maximum entity size (size_t)
typedef struct __align__(8) {
unsigned short magic; // Magic number says we're valid
unsigned short fmtoffset; // Offset of fmt string into buffer
unsigned short blockid; // Block ID of author
unsigned short threadid; // Thread ID of author
} cuPrintfHeader;

// Special header for sm_10 architecture
#define CUPRINTF_SM10_MAGIC 0xC810 // Not a valid ascii character
typedef struct __align__(16) {
unsigned short magic; // sm_10 specific magic number
unsigned short unused;
unsigned int thread_index; // thread ID for this buffer
unsigned int thread_buf_len; // per-thread buffer length
unsigned int offset; // most recent printf's offset
} cuPrintfHeaderSM10;


// Because we can't write an element which is not aligned to its bit-size,
// we have to align all sizes and variables on maximum-size boundaries.
// That means sizeof(double) in this case, but we'll use (long long) for
// better arch<1.3 support
#define CUPRINTF_ALIGN_SIZE sizeof(long long)

// All our headers are prefixed with a magic number so we know they're ready
#define CUPRINTF_SM11_MAGIC (unsigned short)0xC811 // Not a valid ascii character


//
// getNextPrintfBufPtr
//
// Grabs a block of space in the general circular buffer, using an
// atomic function to ensure that it's ours. We handle wrapping
// around the circular buffer and return a pointer to a place which
// can be written to.
//
// Important notes:
// 1. We always grab CUPRINTF_MAX_LEN bytes
// 2. Because of 1, we never worry about wrapping around the end
// 3. Because of 1, printfBufferLength *must* be a factor of CUPRINTF_MAX_LEN
//
// This returns a pointer to the place where we own.
//
__device__ static char *getNextPrintfBufPtr()
{
// Initialisation check
if(!printfBufferPtr)
return NULL;

// Thread/block restriction check
if((restrictRules.blockid != CUPRINTF_UNRESTRICTED) && (restrictRules.blockid != (blockIdx.x + gridDim.x*blockIdx.y)))
return NULL;
if((restrictRules.threadid != CUPRINTF_UNRESTRICTED) && (restrictRules.threadid != (threadIdx.x + blockDim.x*threadIdx.y + blockDim.x*blockDim.y*threadIdx.z)))
return NULL;

// Conditional section, dependent on architecture
#if __CUDA_ARCH__ == 100
// For sm_10 architectures, we have no atomic add - this means we must split the
// entire available buffer into per-thread blocks. Inefficient, but what can you do.
int thread_count = (gridDim.x * gridDim.y) * (blockDim.x * blockDim.y * blockDim.z);
int thread_index = threadIdx.x + blockDim.x*threadIdx.y + blockDim.x*blockDim.y*threadIdx.z +
(blockIdx.x + gridDim.x*blockIdx.y) * (blockDim.x * blockDim.y * blockDim.z);

// Find our own block of data and go to it. Make sure the per-thread length
// is a precise multiple of CUPRINTF_MAX_LEN, otherwise we risk size and
// alignment issues! We must round down, of course.
unsigned int thread_buf_len = printfBufferLength / thread_count;
thread_buf_len &= ~(CUPRINTF_MAX_LEN-1);

// We *must* have a thread buffer length able to fit at least two printfs (one header, one real)
if(thread_buf_len < (CUPRINTF_MAX_LEN * 2))
return NULL;

// Now address our section of the buffer. The first item is a header.
char *myPrintfBuffer = globalPrintfBuffer + (thread_buf_len * thread_index);
cuPrintfHeaderSM10 hdr = *(cuPrintfHeaderSM10 *)(void *)myPrintfBuffer;
if(hdr.magic != CUPRINTF_SM10_MAGIC)
{
// If our header is not set up, initialise it
hdr.magic = CUPRINTF_SM10_MAGIC;
hdr.thread_index = thread_index;
hdr.thread_buf_len = thread_buf_len;
hdr.offset = 0; // Note we start at 0! We pre-increment below.
*(cuPrintfHeaderSM10 *)(void *)myPrintfBuffer = hdr; // Write back the header

// For initial setup purposes, we might need to init thread0's header too
// (so that cudaPrintfDisplay() below will work). This is only run once.
cuPrintfHeaderSM10 *tophdr = (cuPrintfHeaderSM10 *)(void *)globalPrintfBuffer;
tophdr->thread_buf_len = thread_buf_len;
}

// Adjust the offset by the right amount, and wrap it if need be
unsigned int offset = hdr.offset + CUPRINTF_MAX_LEN;
if(offset >= hdr.thread_buf_len)
offset = CUPRINTF_MAX_LEN;

// Write back the new offset for next time and return a pointer to it
((cuPrintfHeaderSM10 *)(void *)myPrintfBuffer)->offset = offset;
return myPrintfBuffer + offset;
#else
// Much easier with an atomic operation!
size_t offset = atomicAdd((unsigned int *)&printfBufferPtr, CUPRINTF_MAX_LEN) - (size_t)globalPrintfBuffer;
offset %= printfBufferLength;
return globalPrintfBuffer + offset;
#endif
}


//
// writePrintfHeader
//
// Inserts the header for containing our UID, fmt position and
// block/thread number. We generate it dynamically to avoid
// issues arising from requiring pre-initialisation.
//
__device__ static void writePrintfHeader(char *ptr, char *fmtptr)
{
if(ptr)
{
cuPrintfHeader header;
header.magic = CUPRINTF_SM11_MAGIC;
header.fmtoffset = (unsigned short)(fmtptr - ptr);
header.blockid = blockIdx.x + gridDim.x*blockIdx.y;
header.threadid = threadIdx.x + blockDim.x*threadIdx.y + blockDim.x*blockDim.y*threadIdx.z;
*(cuPrintfHeader *)(void *)ptr = header;
}
}


//
// cuPrintfStrncpy
//
// This special strncpy outputs an aligned length value, followed by the
// string. It then zero-pads the rest of the string until a 64-aligned
// boundary. The length *includes* the padding. A pointer to the byte
// just after the \0 is returned.
//
// This function could overflow CUPRINTF_MAX_LEN characters in our buffer.
// To avoid it, we must count as we output and truncate where necessary.
//
__device__ static char *cuPrintfStrncpy(char *dest, const char *src, int n, char *end)
{
// Initialisation and overflow check
if(!dest || !src || (dest >= end))
return NULL;

// Prepare to write the length specifier. We're guaranteed to have
// at least "CUPRINTF_ALIGN_SIZE" bytes left because we only write out in
// chunks that size, and CUPRINTF_MAX_LEN is aligned with CUPRINTF_ALIGN_SIZE.
int *lenptr = (int *)(void *)dest;
int len = 0;
dest += CUPRINTF_ALIGN_SIZE;

// Now copy the string
while(n--)
{
if(dest >= end) // Overflow check
break;

len++;
*dest++ = *src;
if(*src++ == '\0')
break;
}

// Now write out the padding bytes, and we have our length.
while((dest < end) && (((long)dest & (CUPRINTF_ALIGN_SIZE-1)) != 0))
{
len++;
*dest++ = 0;
}
*lenptr = len;
return (dest < end) ? dest : NULL; // Overflow means return NULL
}


//
// copyArg
//
// This copies a length specifier and then the argument out to the
// data buffer. Templates let the compiler figure all this out at
// compile-time, making life much simpler from the programming
// point of view. I'm assuimg all (const char *) is a string, and
// everything else is the variable it points at. I'd love to see
// a better way of doing it, but aside from parsing the format
// string I can't think of one.
//
// The length of the data type is inserted at the beginning (so that
// the display can distinguish between float and double), and the
// pointer to the end of the entry is returned.
//
__device__ static char *copyArg(char *ptr, const char *arg, char *end)
{
// Initialisation check
if(!ptr || !arg)
return NULL;

// strncpy does all our work. We just terminate.
if((ptr = cuPrintfStrncpy(ptr, arg, CUPRINTF_MAX_LEN, end)) != NULL)
*ptr = 0;

return ptr;
}

template <typename T>
__device__ static char *copyArg(char *ptr, T &arg, char *end)
{
// Initisalisation and overflow check. Alignment rules mean that
// we're at least CUPRINTF_ALIGN_SIZE away from "end", so we only need
// to check that one offset.
if(!ptr || ((ptr+CUPRINTF_ALIGN_SIZE) >= end))
return NULL;

// Write the length and argument
*(int *)(void *)ptr = sizeof(arg);
ptr += CUPRINTF_ALIGN_SIZE;
*(T *)(void *)ptr = arg;
ptr += CUPRINTF_ALIGN_SIZE;
*ptr = 0;

return ptr;
}


//
// cuPrintf
//
// Templated printf functions to handle multiple arguments.
// Note we return the total amount of data copied, not the number
// of characters output. But then again, who ever looks at the
// return from printf() anyway?
//
// The format is to grab a block of circular buffer space, the
// start of which will hold a header and a pointer to the format
// string. We then write in all the arguments, and finally the
// format string itself. This is to make it easy to prevent
// overflow of our buffer (we support up to 10 arguments, each of
// which can be 12 bytes in length - that means that only the
// format string (or a %s) can actually overflow; so the overflow
// check need only be in the strcpy function.
//
// The header is written at the very last because that's what
// makes it look like we're done.
//
// Errors, which are basically lack-of-initialisation, are ignored
// in the called functions because NULL pointers are passed around
//

// All printf variants basically do the same thing, setting up the
// buffer, writing all arguments, then finalising the header. For
// clarity, we'll pack the code into some big macros.
#define CUPRINTF_PREAMBLE \
char *start, *end, *bufptr, *fmtstart; \
if((start = getNextPrintfBufPtr()) == NULL) return 0; \
end = start + CUPRINTF_MAX_LEN; \
bufptr = start + sizeof(cuPrintfHeader);

// Posting an argument is easy
#define CUPRINTF_ARG(argname) \
bufptr = copyArg(bufptr, argname, end);

// After args are done, record start-of-fmt and write the fmt and header
#define CUPRINTF_POSTAMBLE \
fmtstart = bufptr; \
end = cuPrintfStrncpy(bufptr, fmt, CUPRINTF_MAX_LEN, end); \
writePrintfHeader(start, end ? fmtstart : NULL); \
return end ? (int)(end - start) : 0;

__device__ int cuPrintf(const char *fmt)
{
CUPRINTF_PREAMBLE;

CUPRINTF_POSTAMBLE;
}
template <typename T1> __device__ int cuPrintf(const char *fmt, T1 arg1)
{
CUPRINTF_PREAMBLE;

CUPRINTF_ARG(arg1);

CUPRINTF_POSTAMBLE;
}
template <typename T1, typename T2> __device__ int cuPrintf(const char *fmt, T1 arg1, T2 arg2)
{
CUPRINTF_PREAMBLE;

CUPRINTF_ARG(arg1);
CUPRINTF_ARG(arg2);

CUPRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3> __device__ int cuPrintf(const char *fmt, T1 arg1, T2 arg2, T3 arg3)
{
CUPRINTF_PREAMBLE;

CUPRINTF_ARG(arg1);
CUPRINTF_ARG(arg2);
CUPRINTF_ARG(arg3);

CUPRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4> __device__ int cuPrintf(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4)
{
CUPRINTF_PREAMBLE;

CUPRINTF_ARG(arg1);
CUPRINTF_ARG(arg2);
CUPRINTF_ARG(arg3);
CUPRINTF_ARG(arg4);

CUPRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4, typename T5> __device__ int cuPrintf(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4, T5 arg5)
{
CUPRINTF_PREAMBLE;

CUPRINTF_ARG(arg1);
CUPRINTF_ARG(arg2);
CUPRINTF_ARG(arg3);
CUPRINTF_ARG(arg4);
CUPRINTF_ARG(arg5);

CUPRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4, typename T5, typename T6> __device__ int cuPrintf(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4, T5 arg5, T6 arg6)
{
CUPRINTF_PREAMBLE;

CUPRINTF_ARG(arg1);
CUPRINTF_ARG(arg2);
CUPRINTF_ARG(arg3);
CUPRINTF_ARG(arg4);
CUPRINTF_ARG(arg5);
CUPRINTF_ARG(arg6);
CUPRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4, typename T5, typename T6, typename T7> __device__ int cuPrintf(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4, T5 arg5, T6 arg6, T7 arg7)
{
CUPRINTF_PREAMBLE;

CUPRINTF_ARG(arg1);
CUPRINTF_ARG(arg2);
CUPRINTF_ARG(arg3);
CUPRINTF_ARG(arg4);
CUPRINTF_ARG(arg5);
CUPRINTF_ARG(arg6);
CUPRINTF_ARG(arg7);

CUPRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4, typename T5, typename T6, typename T7, typename T8> __device__ int cuPrintf(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4, T5 arg5, T6 arg6, T7 arg7, T8 arg8)
{
CUPRINTF_PREAMBLE;

CUPRINTF_ARG(arg1);
CUPRINTF_ARG(arg2);
CUPRINTF_ARG(arg3);
CUPRINTF_ARG(arg4);
CUPRINTF_ARG(arg5);
CUPRINTF_ARG(arg6);
CUPRINTF_ARG(arg7);
CUPRINTF_ARG(arg8);

CUPRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4, typename T5, typename T6, typename T7, typename T8, typename T9> __device__ int cuPrintf(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4, T5 arg5, T6 arg6, T7 arg7, T8 arg8, T9 arg9)
{
CUPRINTF_PREAMBLE;

CUPRINTF_ARG(arg1);
CUPRINTF_ARG(arg2);
CUPRINTF_ARG(arg3);
CUPRINTF_ARG(arg4);
CUPRINTF_ARG(arg5);
CUPRINTF_ARG(arg6);
CUPRINTF_ARG(arg7);
CUPRINTF_ARG(arg8);
CUPRINTF_ARG(arg9);

CUPRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4, typename T5, typename T6, typename T7, typename T8, typename T9, typename T10> __device__ int cuPrintf(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4, T5 arg5, T6 arg6, T7 arg7, T8 arg8, T9 arg9, T10 arg10)
{
CUPRINTF_PREAMBLE;

CUPRINTF_ARG(arg1);
CUPRINTF_ARG(arg2);
CUPRINTF_ARG(arg3);
CUPRINTF_ARG(arg4);
CUPRINTF_ARG(arg5);
CUPRINTF_ARG(arg6);
CUPRINTF_ARG(arg7);
CUPRINTF_ARG(arg8);
CUPRINTF_ARG(arg9);
CUPRINTF_ARG(arg10);

CUPRINTF_POSTAMBLE;
}
#undef CUPRINTF_PREAMBLE
#undef CUPRINTF_ARG
#undef CUPRINTF_POSTAMBLE


//
// cuPrintfRestrict
//
// Called to restrict output to a given thread/block.
// We store the info in "restrictRules", which is set up at
// init time by the host. It's not the cleanest way to do this
// because it means restrictions will last between
// invocations, but given the output-pointer continuity,
// I feel this is reasonable.
//
__device__ void cuPrintfRestrict(int threadid, int blockid)
{
int thread_count = blockDim.x * blockDim.y * blockDim.z;
if(((threadid < thread_count) && (threadid >= 0)) || (threadid == CUPRINTF_UNRESTRICTED))
restrictRules.threadid = threadid;

int block_count = gridDim.x * gridDim.y;
if(((blockid < block_count) && (blockid >= 0)) || (blockid == CUPRINTF_UNRESTRICTED))
restrictRules.blockid = blockid;
}


///////////////////////////////////////////////////////////////////////////////
// HOST SIDE

#include <stdio.h>
static FILE *printf_fp;

static char *printfbuf_start=NULL;
static char *printfbuf_device=NULL;
static int printfbuf_len=0;


//
// outputPrintfData
//
// Our own internal function, which takes a pointer to a data buffer
// and passes it through libc's printf for output.
//
// We receive the formate string and a pointer to where the data is
// held. We then run through and print it out.
//
// Returns 0 on failure, 1 on success
//
static int outputPrintfData(char *fmt, char *data)
{
// Format string is prefixed by a length that we don't need
fmt += CUPRINTF_ALIGN_SIZE;

// Now run through it, printing everything we can. We must
// run to every % character, extract only that, and use printf
// to format it.
char *p = strchr(fmt, '%');
while(p != NULL)
{
// Print up to the % character
*p = '\0';
fputs(fmt, printf_fp);
*p = '%'; // Put back the %

// Now handle the format specifier
char *format = p++; // Points to the '%'
p += strcspn(p, "%cdiouxXeEfgGaAnps");
if(*p == '\0') // If no format specifier, print the whole thing
{
fmt = format;
break;
}

// Cut out the format bit and use printf to print it. It's prefixed
// by its length.
int arglen = *(int *)data;
if(arglen > CUPRINTF_MAX_LEN)
{
fputs("Corrupt printf buffer data - aborting\n", printf_fp);
return 0;
}

data += CUPRINTF_ALIGN_SIZE;

char specifier = *p++;
char c = *p; // Store for later
*p = '\0';
switch(specifier)
{
// These all take integer arguments
case 'c':
case 'd':
case 'i':
case 'o':
case 'u':
case 'x':
case 'X':
case 'p':
fprintf(printf_fp, format, *((int *)data));
break;

// These all take double arguments
case 'e':
case 'E':
case 'f':
case 'g':
case 'G':
case 'a':
case 'A':
if(arglen == 4) // Float vs. Double thing
fprintf(printf_fp, format, *((float *)data));
else
fprintf(printf_fp, format, *((double *)data));
break;

// Strings are handled in a special way
case 's':
fprintf(printf_fp, format, (char *)data);
break;

// % is special
case '%':
fprintf(printf_fp, "%%");
break;

// Everything else is just printed out as-is
default:
fprintf(printf_fp, format);
break;
}
data += CUPRINTF_ALIGN_SIZE; // Move on to next argument
*p = c; // Restore what we removed
fmt = p; // Adjust fmt string to be past the specifier
p = strchr(fmt, '%'); // and get the next specifier
}

// Print out the last of the string
fputs(fmt, printf_fp);
return 1;
}


//
// doPrintfDisplay
//
// This runs through the blocks of CUPRINTF_MAX_LEN-sized data, calling the
// print function above to display them. We've got this separate from
// cudaPrintfDisplay() below so we can handle the SM_10 architecture
// partitioning.
//
static int doPrintfDisplay(int headings, int clear, char *bufstart, char *bufend, char *bufptr, char *endptr)
{
// Grab, piece-by-piece, each output element until we catch
// up with the circular buffer end pointer
int printf_count=0;
char printfbuf_local[CUPRINTF_MAX_LEN+1];
printfbuf_local[CUPRINTF_MAX_LEN] = '\0';

while(bufptr != endptr)
{
// Wrap ourselves at the end-of-buffer
if(bufptr == bufend)
bufptr = bufstart;

// Adjust our start pointer to within the circular buffer and copy a block.
hipMemcpy(printfbuf_local, bufptr, CUPRINTF_MAX_LEN, hipMemcpyDeviceToHost);

// If the magic number isn't valid, then this write hasn't gone through
// yet and we'll wait until it does (or we're past the end for non-async printfs).
cuPrintfHeader *hdr = (cuPrintfHeader *)printfbuf_local;
if((hdr->magic != CUPRINTF_SM11_MAGIC) || (hdr->fmtoffset >= CUPRINTF_MAX_LEN))
{
//fprintf(printf_fp, "Bad magic number in printf header\n");
break;
}

// Extract all the info and get this printf done
if(headings)
fprintf(printf_fp, "[%d, %d]: ", hdr->blockid, hdr->threadid);
if(hdr->fmtoffset == 0)
fprintf(printf_fp, "printf buffer overflow\n");
else if(!outputPrintfData(printfbuf_local+hdr->fmtoffset, printfbuf_local+sizeof(cuPrintfHeader)))
break;
printf_count++;

// Clear if asked
if(clear)
hipMemset(bufptr, 0, CUPRINTF_MAX_LEN);

// Now advance our start location, because we're done, and keep copying
bufptr += CUPRINTF_MAX_LEN;
}

return printf_count;
}


//
// cudaPrintfInit
//
// Takes a buffer length to allocate, creates the memory on the device and
// returns a pointer to it for when a kernel is called. It's up to the caller
// to free it.
//
extern "C" hipError_t cudaPrintfInit(size_t bufferLen)
{
// Fix up bufferlen to be a multiple of CUPRINTF_MAX_LEN
bufferLen = (bufferLen < CUPRINTF_MAX_LEN) ? CUPRINTF_MAX_LEN : bufferLen;
if((bufferLen % CUPRINTF_MAX_LEN) > 0)
bufferLen += (CUPRINTF_MAX_LEN - (bufferLen % CUPRINTF_MAX_LEN));
printfbuf_len = (int)bufferLen;

// Allocate a print buffer on the device and zero it
if(hipMalloc((void **)&printfbuf_device, printfbuf_len) != hipSuccess)
return hipErrorNotInitialized;
hipMemset(printfbuf_device, 0, printfbuf_len);
printfbuf_start = printfbuf_device; // Where we start reading from

// No restrictions to begin with
cuPrintfRestriction restrict;
restrict.threadid = restrict.blockid = CUPRINTF_UNRESTRICTED;
hipMemcpyToSymbol(HIP_SYMBOL(restrictRules), &restrict, sizeof(restrict));

// Initialise the buffer and the respective lengths/pointers.
hipMemcpyToSymbol(HIP_SYMBOL(globalPrintfBuffer), &printfbuf_device, sizeof(char *));
hipMemcpyToSymbol(HIP_SYMBOL(printfBufferPtr), &printfbuf_device, sizeof(char *));
hipMemcpyToSymbol(HIP_SYMBOL(printfBufferLength), &printfbuf_len, sizeof(printfbuf_len));

return hipSuccess;
}


//
// cudaPrintfEnd
//
// Frees up the memory which we allocated
//
extern "C" void cudaPrintfEnd()
{
if(!printfbuf_start || !printfbuf_device)
return;

hipFree(printfbuf_device);
printfbuf_start = printfbuf_device = NULL;
}


//
// cudaPrintfDisplay
//
// Each call to this function dumps the entire current contents
// of the printf buffer to the pre-specified FILE pointer. The
// circular "start" pointer is advanced so that subsequent calls
// dumps only new stuff.
//
// In the case of async memory access (via streams), call this
// repeatedly to keep trying to empty the buffer. If it's a sync
// access, then the whole buffer should empty in one go.
//
// Arguments:
// outputFP - File descriptor to output to (NULL => stdout)
// showThreadID - If true, prints [block,thread] before each line
//
extern "C" hipError_t cudaPrintfDisplay(void *outputFP, bool showThreadID)
{
printf_fp = (FILE *)((outputFP == NULL) ? stdout : outputFP);

// For now, we force "synchronous" mode which means we're not concurrent
// with kernel execution. This also means we don't need clearOnPrint.
// If you're patching it for async operation, here's where you want it.
bool sync_printfs = true;
bool clearOnPrint = false;

// Initialisation check
if(!printfbuf_start || !printfbuf_device || !printf_fp)
return hipErrorMissingConfiguration;

// To determine which architecture we're using, we read the
// first short from the buffer - it'll be the magic number
// relating to the version.
unsigned short magic;
hipMemcpy(&magic, printfbuf_device, sizeof(unsigned short), hipMemcpyDeviceToHost);

// For SM_10 architecture, we've split our buffer into one-per-thread.
// That means we must do each thread block separately. It'll require
// extra reading. We also, for now, don't support async printfs because
// that requires tracking one start pointer per thread.
if(magic == CUPRINTF_SM10_MAGIC)
{
sync_printfs = true;
clearOnPrint = false;
int blocklen = 0;
char *blockptr = printfbuf_device;
while(blockptr < (printfbuf_device + printfbuf_len))
{
cuPrintfHeaderSM10 hdr;
hipMemcpy(&hdr, blockptr, sizeof(hdr), hipMemcpyDeviceToHost);

// We get our block-size-step from the very first header
if(hdr.thread_buf_len != 0)
blocklen = hdr.thread_buf_len;

// No magic number means no printfs from this thread
if(hdr.magic != CUPRINTF_SM10_MAGIC)
{
if(blocklen == 0)
{
fprintf(printf_fp, "No printf headers found at all!\n");
break; // No valid headers!
}
blockptr += blocklen;
continue;
}

// "offset" is non-zero then we can print the block contents
if(hdr.offset > 0)
{
// For synchronous printfs, we must print from endptr->bufend, then from start->end
if(sync_printfs)
doPrintfDisplay(showThreadID, clearOnPrint, blockptr+CUPRINTF_MAX_LEN, blockptr+hdr.thread_buf_len, blockptr+hdr.offset+CUPRINTF_MAX_LEN, blockptr+hdr.thread_buf_len);
doPrintfDisplay(showThreadID, clearOnPrint, blockptr+CUPRINTF_MAX_LEN, blockptr+hdr.thread_buf_len, blockptr+CUPRINTF_MAX_LEN, blockptr+hdr.offset+CUPRINTF_MAX_LEN);
}

// Move on to the next block and loop again
blockptr += hdr.thread_buf_len;
}
}
// For SM_11 and up, everything is a single buffer and it's simple
else if(magic == CUPRINTF_SM11_MAGIC)
{
// Grab the current "end of circular buffer" pointer.
char *printfbuf_end = NULL;
hipMemcpyFromSymbol(&printfbuf_end, HIP_SYMBOL(printfBufferPtr), sizeof(char *));

// Adjust our starting and ending pointers to within the block
char *bufptr = ((printfbuf_start - printfbuf_device) % printfbuf_len) + printfbuf_device;
char *endptr = ((printfbuf_end - printfbuf_device) % printfbuf_len) + printfbuf_device;

// For synchronous (i.e. after-kernel-exit) printf display, we have to handle circular
// buffer wrap carefully because we could miss those past "end".
if(sync_printfs)
doPrintfDisplay(showThreadID, clearOnPrint, printfbuf_device, printfbuf_device+printfbuf_len, endptr, printfbuf_device+printfbuf_len);
doPrintfDisplay(showThreadID, clearOnPrint, printfbuf_device, printfbuf_device+printfbuf_len, bufptr, endptr);

printfbuf_start = printfbuf_end;
}
else
;//printf("Bad magic number in cuPrintf buffer header\n");

// If we were synchronous, then we must ensure that the memory is cleared on exit
// otherwise another kernel launch with a different grid size could conflict.
if(sync_printfs)
hipMemset(printfbuf_device, 0, printfbuf_len);

return hipSuccess;
}

// Cleanup
#undef CUPRINTF_MAX_LEN
#undef CUPRINTF_ALIGN_SIZE
#undef CUPRINTF_SM10_MAGIC
#undef CUPRINTF_SM11_MAGIC

#endif


