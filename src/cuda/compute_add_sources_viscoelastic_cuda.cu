#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 1
 !               ---------------------------------------
 !
 !     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
 !                        Princeton University, USA
 !                and CNRS / University of Marseille, France
 !                 (there are currently many more authors!)
 ! (c) Princeton University and CNRS / University of Marseille, July 2012
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/types.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

// elastic domain sources

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_add_sources_kernel(realw* accel,
                                           int* d_ibool,
                                           int* ispec_is_inner,
                                           int phase_is_inner,
                                           realw* sourcearrays,
                                           realw* d_source_time_function,
                                           int myrank,
                                           int* islice_selected_source,
                                           int* ispec_selected_source,
                                           int* ispec_is_elastic,
                                           int NSOURCES, int it,int* d_num_src_loc,int nsources_local) {
  int i = threadIdx.x;
  int j = threadIdx.y;

  int isource  = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int ispec,iglob,num_source_locale;
  realw stf;

  if(isource < NSOURCES) { // when NSOURCES > 65535, but mod(nspec_top,2) > 0, we end up with an extra block.

    if(myrank == islice_selected_source[isource]) {

      ispec = ispec_selected_source[isource]-1;

      if(ispec_is_inner[ispec] == phase_is_inner && ispec_is_elastic[ispec] ) {

        num_source_locale = d_num_src_loc[isource]-1;

        stf = d_source_time_function[INDEX2(nsources_local,num_source_locale,it)];
        iglob = d_ibool[INDEX3_PADDED(NGLLX,NGLLX,i,j,ispec)] - 1;


        atomicAdd(&accel[iglob*2],sourcearrays[INDEX4(NSOURCES,NDIM,NGLLX,isource, 0,i,j)]*stf);
        atomicAdd(&accel[iglob*2+1],sourcearrays[INDEX4(NSOURCES,NDIM,NGLLX,isource, 1,i,j)]*stf);


      }
    }
  }

}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_add_sources_el_cuda,
              COMPUTE_ADD_SOURCES_EL_CUDA)(long* Mesh_pointer, 
                                           int* h_phase_is_inner,
                                           int* h_NSOURCES,
                                           int* itf) {

  TRACE("\tcompute_add_sources_el_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // check if anything to do
  if( mp->nsources_local == 0 ) return;

  int NSOURCES = *h_NSOURCES;
  int phase_is_inner = *h_phase_is_inner;
  int it = *itf -1;



  int num_blocks_x, num_blocks_y;
  get_blocks_xy(NSOURCES,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(5,5,1);

  compute_add_sources_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_accel,mp->d_ibool,
                                                                    mp->d_ispec_is_inner,phase_is_inner,
                                                                    mp->d_sourcearrays,
                                                                    mp->d_source_time_function,
                                                                    mp->myrank,
                                                                    mp->d_islice_selected_source,mp->d_ispec_selected_source,
                                                                    mp->d_ispec_is_elastic,
                                                                    NSOURCES,it,mp->d_num_src_loc,mp->nsources_local);


#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_add_sources_el_cuda");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_add_sources_el_s3_cuda,
              COMPUTE_ADD_SOURCES_EL_S3_CUDA)(long* Mesh_pointer,
                                              int* phase_is_innerf,
                                              int* NSOURCESf,
                                              int* itf) {

  TRACE("\tcompute_add_sources_el_s3_cuda");
  // EPIK_TRACER("compute_add_sources_el_s3_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // check if anything to do
  if( mp->nsources_local == 0 ) return;


  int NSOURCES = *NSOURCESf;
  int phase_is_inner = *phase_is_innerf;
  int num_blocks_x, num_blocks_y;
  get_blocks_xy(NSOURCES,&num_blocks_x,&num_blocks_y);
  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(5,5,1);
  int it = *itf -1;

  compute_add_sources_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_accel,mp->d_ibool,
                                                                    mp->d_ispec_is_inner, phase_is_inner,
                                                                    mp->d_sourcearrays,
                                                                    mp->d_source_time_function,
                                                                    mp->myrank,
                                                                    mp->d_islice_selected_source,mp->d_ispec_selected_source,
                                                                    mp->d_ispec_is_elastic,
                                                                    NSOURCES,it,mp->d_num_src_loc,mp->nsources_local);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_add_sources_el_s3_cuda");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// ADJOINT sources

/* ----------------------------------------------------------------------------------------------- */

__global__ void add_sources_el_SIM_TYPE_2_OR_3_kernel(realw* accel,
                                                      realw* source_adjointe,
                                                      realw* xir_store,
                                                      realw* gammar_store,
                                                      int* d_ibool,
                                                      int* ispec_is_inner,
                                                      int* ispec_is_elastic,
                                                      int* ispec_selected_rec,
                                                      int phase_is_inner,
                                                      int it,
                                                      int* pre_computed_irec,
                                                      int nadj_rec_local,
                                                      int NSTEP ) {

  int irec_local = blockIdx.x + gridDim.x*blockIdx.y;

  if(irec_local < nadj_rec_local) { // when nrec > 65535, but mod(nspec_top,2) > 0, we end up with an extra block.

    int irec = pre_computed_irec[irec_local];

    int ispec = ispec_selected_rec[irec]-1;
    if( ispec_is_elastic[ispec] ){

      if(ispec_is_inner[ispec] == phase_is_inner) {
        int i = threadIdx.x;
        int j = threadIdx.y;
        int iglob = d_ibool[INDEX3_PADDED(NGLLX,NGLLX,i,j,ispec)]-1;

        realw  xir = xir_store[INDEX2(nadj_rec_local,irec_local,i)];

        realw  gammar = gammar_store[INDEX2(nadj_rec_local,irec_local,j)];

        realw  source_adjx = source_adjointe[INDEX3(nadj_rec_local,NSTEP,irec_local,it,0)];

        realw  source_adjz = source_adjointe[INDEX3(nadj_rec_local,NSTEP,irec_local,it,1)];


        // atomic operations are absolutely necessary for correctness!
        atomicAdd(&accel[2*iglob],source_adjx * gammar * xir);
        atomicAdd(&accel[1+2*iglob], source_adjz * gammar * xir);

      }
    } // ispec_is_elastic
  }

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(add_sources_el_sim_type_2_or_3,
              ADD_SOURCES_EL_SIM_TYPE_2_OR_3)(long* Mesh_pointer,
                                               int* phase_is_inner,
                                               int* it,
                                               int* nadj_rec_local,
                                               int* NSTEP) {

  TRACE("\tadd_sources_el_sim_type_2_or_3");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // checks
  if( *nadj_rec_local != mp->nadj_rec_local) exit_on_error("add_sources_el_sim_type_2_or_3: nadj_rec_local not equal\n");

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->nadj_rec_local,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y,1);
  dim3 threads(5,5,1);

  int it_index = (*it) - 1;


  // the irec_local variable needs to be precomputed (as
  // h_pre_comp..), because normally it is in the loop updating accel,
  // and due to how it's incremented, it cannot be parallelized

  add_sources_el_SIM_TYPE_2_OR_3_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_accel,
                                                                               mp->d_source_adjointe,
                                                                               mp->d_xir_store_loc,
                                                                               mp->d_gammar_store_loc,
                                                                               mp->d_ibool,
                                                                               mp->d_ispec_is_inner,
                                                                               mp->d_ispec_is_elastic,
                                                                               mp->d_ispec_selected_rec,
                                                                               *phase_is_inner,
                                                                               it_index,
                                                                               mp->d_pre_computed_irec,
                                                                               mp->nadj_rec_local,
                                                                               *NSTEP);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("add_sources_SIM_TYPE_2_OR_3_kernel");
#endif
}

