#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This software is governed by the CeCILL license under French law and
! abiding by the rules of distribution of free software. You can use,
! modify and/or redistribute the software under the terms of the CeCILL
! license as circulated by CEA, CNRS and Inria at the following URL
! "http://www.cecill.info".
!
! As a counterpart to the access to the source code and rights to copy,
! modify and redistribute granted by the license, users are provided only
! with a limited warranty and the software's author, the holder of the
! economic rights, and the successive licensors have only limited
! liability.
!
! In this respect, the user's attention is drawn to the risks associated
! with loading, using, modifying and/or developing or reproducing the
! software by the user in light of its specific status of free software,
! that may mean that it is complicated to manipulate, and that also
! therefore means that it is reserved for developers and experienced
! professionals having in-depth computer knowledge. Users are therefore
! encouraged to load and test the software's suitability as regards their
! requirements in conditions enabling the security of their systems and/or
! data to be ensured and, more generally, to use and operate it in the
! same conditions as regards security.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================

*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/types.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

// acoustic sources

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_add_sources_acoustic_kernel(realw* potential_dot_dot_acoustic,
                                                    int* d_ibool,
                                                    int* ispec_is_inner,
                                                    int phase_is_inner,
                                                    realw* sourcearrays,
                                                    realw* source_time_function,
                                                    int myrank,
                                                    int* ispec_selected_source,
                                                    int* ispec_is_acoustic,
                                                    realw* kappastore,
                                                    int it,int nsources_local) {
  int i = threadIdx.x;
  int j = threadIdx.y;

  int isource  = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int ispec,iglob;
  realw stf,kappal;

  if( isource < nsources_local ){

      ispec = ispec_selected_source[isource]-1;

      if(ispec_is_inner[ispec] == phase_is_inner && ispec_is_acoustic[ispec] ) {

        iglob = d_ibool[INDEX3_PADDED(NGLLX,NGLLX,i,j,ispec)] - 1;

        kappal = kappastore[INDEX3(NGLLX,NGLLX,i,j,ispec)];

        stf = source_time_function[INDEX2(nsources_local,isource,it)]/kappal;
        atomicAdd(&potential_dot_dot_acoustic[iglob],
                  -sourcearrays[INDEX4(nsources_local,NDIM,NGLLX,isource, 0,i,j)]*stf);



    }
  }
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_add_sources_ac_cuda,
              COMPUTE_ADD_SOURCES_AC_CUDA)(long* Mesh_pointer,
                                           int* phase_is_innerf,
                                           int * itf) {

  TRACE("compute_add_sources_ac_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // check if anything to do
  if( mp->nsources_local == 0 ) return;

  int phase_is_inner = *phase_is_innerf;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->nsources_local,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(5,5,1);

  int it = *itf -1;



  compute_add_sources_acoustic_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_potential_dot_dot_acoustic,
                                                                              mp->d_ibool,
                                                                              mp->d_ispec_is_inner,
                                                                              phase_is_inner,
                                                                              mp->d_sourcearrays,
                                                                              mp->d_source_time_function,
                                                                              mp->myrank,
                                                                              mp->d_ispec_selected_source,
                                                                              mp->d_ispec_is_acoustic,
                                                                              mp->d_kappastore,
                                                                              it,mp->nsources_local);

  print_CUDA_error_if_any(hipStreamSynchronize(mp->compute_stream),mp->myrank);


#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_add_sources_ac_cuda");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_add_sources_ac_s3_cuda,
              COMPUTE_ADD_SOURCES_AC_s3_CUDA)(long* Mesh_pointer,
                                              int* phase_is_innerf,
                                              int* itf) {

  TRACE("compute_add_sources_ac_s3_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // check if anything to do
  if( mp->nsources_local == 0 ) return;

  int phase_is_inner = *phase_is_innerf;
  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->nsources_local,&num_blocks_x,&num_blocks_y);
  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(5,5,1);
  int it = *itf -1;

  compute_add_sources_acoustic_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_potential_dot_dot_acoustic,
                                                                              mp->d_ibool,
                                                                              mp->d_ispec_is_inner,
                                                                              phase_is_inner,
                                                                              mp->d_sourcearrays,
                                                                              mp->d_source_time_function,
                                                                              mp->myrank,
                                                                              mp->d_ispec_selected_source,
                                                                              mp->d_ispec_is_acoustic,
                                                                              mp->d_kappastore,
                                                                              it,mp->nsources_local);


      print_CUDA_error_if_any(hipStreamSynchronize(mp->compute_stream),38);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_add_sources_ac_s3_cuda");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// acoustic adjoint sources

/* ----------------------------------------------------------------------------------------------- */

__global__ void add_sources_ac_SIM_TYPE_2_OR_3_kernel(realw* potential_dot_dot_acoustic,
                                                      realw* source_adjointe,
                                                      realw* xir_store,
                                                      realw* gammar_store,
                                                      int* d_ibool,
                                                      int* ispec_is_inner,
                                                      int* ispec_is_acoustic,
                                                      int* ispec_selected_rec,
                                                      int phase_is_inner,
                                                      int it,
                                                      int* pre_computed_irec,
                                                      int nadj_rec_local,
                                                      realw* kappastore,
                                                      int NSTEP  ) {

  int irec_local = blockIdx.x + gridDim.x*blockIdx.y;

  // because of grid shape, irec_local can be too big
  if(irec_local < nadj_rec_local) {

    int irec = pre_computed_irec[irec_local];

    int ispec = ispec_selected_rec[irec]-1;
    if( ispec_is_acoustic[ispec] ){

      // checks if element is in phase_is_inner run
      if(ispec_is_inner[ispec] == phase_is_inner) {
        int i = threadIdx.x;
        int j = threadIdx.y;


        int iglob = d_ibool[INDEX3_PADDED(NGLLX,NGLLX,i,j,ispec)]-1;

        realw  kappal = kappastore[INDEX3(NGLLX,NGLLX,i,j,ispec)];

        realw  xir = xir_store[INDEX2(nadj_rec_local,irec_local,i)];

        realw  gammar = gammar_store[INDEX2(nadj_rec_local,irec_local,j)];

        realw  source_adj = source_adjointe[INDEX3(nadj_rec_local,NSTEP,irec_local,it,0)];


        // beware, for acoustic medium, a pressure source would be taking the negative
        // and divide by Kappa of the fluid;
        // this would have to be done when constructing the adjoint source.
        //
        //          the idea is to have e.g. a pressure source, where all 3 components would be the same


        realw stf = source_adj * gammar * xir / kappal ;

        atomicAdd(&potential_dot_dot_acoustic[iglob],stf);


      }
    }
  }
}

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(add_sources_ac_sim_2_or_3_cuda,
              ADD_SOURCES_AC_SIM_2_OR_3_CUDA)(long* Mesh_pointer,
                                               int* phase_is_inner,
                                               int* it,
                                               int* nadj_rec_local,
                                               int* NSTEP) {

  TRACE("add_sources_ac_sim_2_or_3_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // checks
  if( *nadj_rec_local != mp->nadj_rec_local) exit_on_cuda_error("add_sources_ac_sim_type_2_or_3: nadj_rec_local not equal\n");

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->nadj_rec_local,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y,1);
  dim3 threads(5,5,1);

  int it_index = (*it) - 1;


  // launches cuda kernel for acoustic adjoint sources
  add_sources_ac_SIM_TYPE_2_OR_3_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_potential_dot_dot_acoustic,
                                                                                mp->d_source_adjointe,
                                                                                mp->d_xir_store_loc,
                                                                                mp->d_gammar_store_loc,
                                                                                mp->d_ibool,
                                                                                mp->d_ispec_is_inner,
                                                                                mp->d_ispec_is_acoustic,
                                                                                mp->d_ispec_selected_rec,
                                                                                *phase_is_inner,
                                                                                it_index,
                                                                                mp->d_pre_computed_irec,
                                                                                mp->nadj_rec_local,
                                                                                mp->d_kappastore,
                                                                                *NSTEP);


#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("add_sources_acoustic_SIM_TYPE_2_OR_3_kernel");
#endif
}
