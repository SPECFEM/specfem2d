#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This software is governed by the CeCILL license under French law and
! abiding by the rules of distribution of free software. You can use,
! modify and/or redistribute the software under the terms of the CeCILL
! license as circulated by CEA, CNRS and Inria at the following URL
! "http://www.cecill.info".
!
! As a counterpart to the access to the source code and rights to copy,
! modify and redistribute granted by the license, users are provided only
! with a limited warranty and the software's author, the holder of the
! economic rights, and the successive licensors have only limited
! liability.
!
! In this respect, the user's attention is drawn to the risks associated
! with loading, using, modifying and/or developing or reproducing the
! software by the user in light of its specific status of free software,
! that may mean that it is complicated to manipulate, and that also
! therefore means that it is reserved for developers and experienced
! professionals having in-depth computer knowledge. Users are therefore
! encouraged to load and test the software's suitability as regards their
! requirements in conditions enabling the security of their systems and/or
! data to be ensured and, more generally, to use and operate it in the
! same conditions as regards security.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================

*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_stacey_elastic_kernel(realw* veloc,
                                              realw* accel,
                                              int* abs_boundary_ispec,
                                              int* abs_boundary_ij,
                                              realw* abs_boundary_normal,
                                              realw* abs_boundary_jacobian1Dw,
                                              int* d_ibool,
                                              realw* rho_vp,
                                              realw* rho_vs,
                                              int* ispec_is_inner,
                                              int* ispec_is_elastic,
                                              int phase_is_inner,
                                              int SIMULATION_TYPE,
                                              int SAVE_FORWARD,
                                              int num_abs_boundary_faces,
                                              realw* b_absorb_elastic_left,
                                              realw* b_absorb_elastic_right,
                                              realw* b_absorb_elastic_top,
                                              realw* b_absorb_elastic_bottom,
                                              int* ib_left,
                                              int* ib_right,
                                              int* ib_top,
                                              int* ib_bottom,
                                              int* cote_abs) {

  int igll = threadIdx.x; // tx
  int iface = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int i,j,iglob,ispec,num_local;
  realw vx,vz,vn;
  realw nx,nz;
  realw rho_vp_temp,rho_vs_temp;
  realw tx,tz;
  realw jacobianw;

  // don't compute points outside NGLLSQUARE==NGLL2==25
  // way 2: no further check needed since blocksize = 25
  if( iface < num_abs_boundary_faces){

  //if(igll < NGLL2 && iface < num_abs_boundary_faces) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = abs_boundary_ispec[iface]-1;

    if(ispec_is_inner[ispec] == phase_is_inner && ispec_is_elastic[ispec] ) {

      i = abs_boundary_ij[INDEX3(NDIM,NGLLX,0,igll,iface)]-1;
      j = abs_boundary_ij[INDEX3(NDIM,NGLLX,1,igll,iface)]-1;

      iglob = d_ibool[INDEX3_PADDED(NGLLX,NGLLX,i,j,ispec)]-1;

      // gets associated velocity

      vx = veloc[iglob*2];
      vz = veloc[iglob*2+1];

      // gets associated normal
      nx = abs_boundary_normal[INDEX3(NDIM,NGLLX,0,igll,iface)];
      nz = abs_boundary_normal[INDEX3(NDIM,NGLLX,1,igll,iface)];

      // // velocity component in normal direction (normal points out of element)
      vn = vx*nx + vz*nz;

      rho_vp_temp = rho_vp[INDEX3(NGLLX,NGLLX,i,j,ispec)];
      rho_vs_temp = rho_vs[INDEX3(NGLLX,NGLLX,i,j,ispec)];

      tx = rho_vp_temp*vn*nx + rho_vs_temp*(vx-vn*nx);
      tz = rho_vp_temp*vn*nz + rho_vs_temp*(vz-vn*nz);

      jacobianw = abs_boundary_jacobian1Dw[INDEX2(NGLLX,igll,iface)];

      atomicAdd(&accel[iglob*2],-tx*jacobianw);
      atomicAdd(&accel[iglob*2+1],-tz*jacobianw);

      if(SAVE_FORWARD && SIMULATION_TYPE == 1) {

      if (cote_abs[iface] == 1) {num_local = ib_bottom[iface]-1;
                                b_absorb_elastic_bottom[INDEX3(NDIM,NGLLX,0,igll,num_local)] = tx*jacobianw;
                                b_absorb_elastic_bottom[INDEX3(NDIM,NGLLX,1,igll,num_local)] = tz*jacobianw;}
      else if (cote_abs[iface] == 2)   {num_local = ib_right[iface]-1;
                                 b_absorb_elastic_right[INDEX3(NDIM,NGLLX,0,igll,num_local)] = tx*jacobianw;
                                b_absorb_elastic_right[INDEX3(NDIM,NGLLX,1,igll,num_local)] = tz*jacobianw;}
      else if (cote_abs[iface] == 3)  {num_local = ib_top[iface]-1;
                                 b_absorb_elastic_top[INDEX3(NDIM,NGLLX,0,igll,num_local)] = tx*jacobianw;
                                b_absorb_elastic_top[INDEX3(NDIM,NGLLX,1,igll,num_local)] = tz*jacobianw;}
      else if (cote_abs[iface] == 4){num_local = ib_left[iface]-1;
                                b_absorb_elastic_left[INDEX3(NDIM,NGLLX,0,igll,num_local)] = tx*jacobianw;
                                b_absorb_elastic_left[INDEX3(NDIM,NGLLX,1,igll,num_local)] = tz*jacobianw;}

      } // SIMULATION_TYPE
    }
  } // num_abs_boundary_faces

}

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_stacey_elastic_sim3_kernel(int* abs_boundary_ispec,
                                                   int* abs_boundary_ijk,
                                                   int* d_ibool,
                                                   int* ispec_is_inner,
                                                   int* ispec_is_elastic,
                                                   int phase_is_inner,
                                                   int num_abs_boundary_faces,
                                                   realw* b_accel,
                                                   realw* b_absorb_elastic_left,
                                                   realw* b_absorb_elastic_right,
                                                   realw* b_absorb_elastic_top,
                                                   realw* b_absorb_elastic_bottom,
                                                   int* ib_left,
                                                   int* ib_right,
                                                   int* ib_top,
                                                   int* ib_bottom,
                                                   int* d_cote_abs) {

  int igll = threadIdx.x; // tx
  int iface = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int i,j,iglob,ispec,num_local;

  if( iface < num_abs_boundary_faces){



    // "-1" from index values to convert from Fortran-> C indexing
    ispec = abs_boundary_ispec[iface]-1;

    if(ispec_is_inner[ispec] == phase_is_inner && ispec_is_elastic[ispec] ) {

      i = abs_boundary_ijk[INDEX3(NDIM,NGLLX,0,igll,iface)]-1;
      j = abs_boundary_ijk[INDEX3(NDIM,NGLLX,1,igll,iface)]-1;

      iglob = d_ibool[INDEX3_PADDED(NGLLX,NGLLX,i,j,ispec)]-1;



if (d_cote_abs[iface] == 1){num_local= ib_bottom[iface]-1;
                               atomicAdd(&b_accel[iglob*2 ],
                                     -b_absorb_elastic_bottom[INDEX3(NDIM,NGLLX,0,igll,num_local)]);
                           atomicAdd(&b_accel[iglob*2+1 ],
                                     -b_absorb_elastic_bottom[INDEX3(NDIM,NGLLX,1,igll,num_local)]);}
else if (d_cote_abs[iface] == 2){num_local= ib_right[iface]-1;
                                  atomicAdd(&b_accel[iglob*2 ],
                                     -b_absorb_elastic_right[INDEX3(NDIM,NGLLX,0,igll,num_local)]);
                           atomicAdd(&b_accel[iglob*2+1 ],
                                     -b_absorb_elastic_right[INDEX3(NDIM,NGLLX,1,igll,num_local)]);}
else if (d_cote_abs[iface] == 3){num_local= ib_top[iface]-1;
                                  atomicAdd(&b_accel[iglob*2 ],
                                     -b_absorb_elastic_top[INDEX3(NDIM,NGLLX,0,igll,num_local)]);
                           atomicAdd(&b_accel[iglob*2+1 ],
                                     -b_absorb_elastic_top[INDEX3(NDIM,NGLLX,1,igll,num_local)]);}
else if (d_cote_abs[iface] == 4){num_local= ib_left[iface]-1;
                                  atomicAdd(&b_accel[iglob*2 ],
                                     -b_absorb_elastic_left[INDEX3(NDIM,NGLLX,0,igll,num_local)]);
                           atomicAdd(&b_accel[iglob*2+1 ],
                                     -b_absorb_elastic_left[INDEX3(NDIM,NGLLX,1,igll,num_local)]);}


    }
  } // num_abs_boundary_faces

}

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(compute_stacey_viscoelastic_cuda,
              COMPUTE_STACEY_VISCOELASTIC_CUDA)(long* Mesh_pointer,
                                           int* phase_is_innerf,
                                           realw* h_b_absorb_elastic_left,
                                           realw* h_b_absorb_elastic_right,
                                           realw* h_b_absorb_elastic_top,
                                           realw* h_b_absorb_elastic_bottom) {

  TRACE("\tcompute_stacey_viscoelastic_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // checks if anything to do
  if( mp->d_num_abs_boundary_faces == 0 ) return;

  int phase_is_inner    = *phase_is_innerf;


  int blocksize = NGLLX;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->d_num_abs_boundary_faces,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  if(mp->simulation_type == 3 && phase_is_inner == 0 ) {
    // reading is done in fortran routine

    print_CUDA_error_if_any(hipMemcpy(mp->d_b_absorb_elastic_left,h_b_absorb_elastic_left,
                                       2*mp->d_nspec_left*sizeof(realw)*NGLLX,hipMemcpyHostToDevice),7700);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_absorb_elastic_right,h_b_absorb_elastic_right,
                                       2*mp->d_nspec_right*sizeof(realw)*NGLLX,hipMemcpyHostToDevice),7700);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_absorb_elastic_top,h_b_absorb_elastic_top,
                                       2*mp->d_nspec_top*sizeof(realw)*NGLLX,hipMemcpyHostToDevice),7700);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_absorb_elastic_bottom,h_b_absorb_elastic_bottom,
                                       2*mp->d_nspec_bottom*sizeof(realw)*NGLLX,hipMemcpyHostToDevice),7700);


  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("between cudamemcpy and compute_stacey_elastic_kernel");
#endif

  compute_stacey_elastic_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_veloc,
                                                  mp->d_accel,
                                                  mp->d_abs_boundary_ispec,
                                                  mp->d_abs_boundary_ijk,
                                                  mp->d_abs_boundary_normal,
                                                  mp->d_abs_boundary_jacobian2Dw,
                                                  mp->d_ibool,
                                                  mp->d_rho_vp,
                                                  mp->d_rho_vs,
                                                  mp->d_ispec_is_inner,
                                                  mp->d_ispec_is_elastic,
                                                  phase_is_inner,
                                                  mp->simulation_type,
                                                  mp->save_forward,
                                                  mp->d_num_abs_boundary_faces,
                                                  mp->d_b_absorb_elastic_left,
                                                   mp->d_b_absorb_elastic_right,
                                                   mp->d_b_absorb_elastic_top,
                                                   mp->d_b_absorb_elastic_bottom,
                                                   mp->d_ib_left,
                                                   mp->d_ib_right,
                                                   mp->d_ib_top,
                                                   mp->d_ib_bottom,
                                                   mp->d_cote_abs);

  // adjoint simulations
  if(mp->simulation_type == 3 ){
    compute_stacey_elastic_sim3_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_abs_boundary_ispec,
                                                         mp->d_abs_boundary_ijk,
                                                         mp->d_ibool,
                                                         mp->d_ispec_is_inner,
                                                         mp->d_ispec_is_elastic,
                                                         phase_is_inner,
                                                         mp->d_num_abs_boundary_faces,
                                                         mp->d_b_accel,
                                                        mp->d_b_absorb_elastic_left,
                                                   mp->d_b_absorb_elastic_right,
                                                   mp->d_b_absorb_elastic_top,
                                                   mp->d_b_absorb_elastic_bottom,
                                                   mp->d_ib_left,
                                                   mp->d_ib_right,
                                                   mp->d_ib_top,
                                                   mp->d_ib_bottom,
                                                   mp->d_cote_abs);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_stacey_elastic_kernel");
#endif

  if(mp->simulation_type == 1 && mp->save_forward && phase_is_inner == 1) {
    // explicitly wait until compute stream is done
    // (hipMemcpy implicitly synchronizes all other cuda operations)
    hipStreamSynchronize(mp->compute_stream);

    // writing is done in fortran routine

    print_CUDA_error_if_any(hipMemcpy(h_b_absorb_elastic_left,mp->d_b_absorb_elastic_left,
                                       2*mp->d_nspec_left*sizeof(realw)*NGLLX,hipMemcpyDeviceToHost),7701);
    print_CUDA_error_if_any(hipMemcpy(h_b_absorb_elastic_right,mp->d_b_absorb_elastic_right,
                                       2*mp->d_nspec_right*sizeof(realw)*NGLLX,hipMemcpyDeviceToHost),7702);
    print_CUDA_error_if_any(hipMemcpy(h_b_absorb_elastic_top,mp->d_b_absorb_elastic_top,
                                       2*mp->d_nspec_top*sizeof(realw)*NGLLX,hipMemcpyDeviceToHost),7703);
    print_CUDA_error_if_any(hipMemcpy(h_b_absorb_elastic_bottom,mp->d_b_absorb_elastic_bottom,
                                       2*mp->d_nspec_bottom*sizeof(realw)*NGLLX,hipMemcpyDeviceToHost),7704);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after compute_stacey_elastic after cudamemcpy");
#endif
}

