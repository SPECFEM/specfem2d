#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This software is governed by the CeCILL license under French law and
! abiding by the rules of distribution of free software. You can use,
! modify and/or redistribute the software under the terms of the CeCILL
! license as circulated by CEA, CNRS and Inria at the following URL
! "http://www.cecill.info".
!
! As a counterpart to the access to the source code and rights to copy,
! modify and redistribute granted by the license, users are provided only
! with a limited warranty and the software's author, the holder of the
! economic rights, and the successive licensors have only limited
! liability.
!
! In this respect, the user's attention is drawn to the risks associated
! with loading, using, modifying and/or developing or reproducing the
! software by the user in light of its specific status of free software,
! that may mean that it is complicated to manipulate, and that also
! therefore means that it is reserved for developers and experienced
! professionals having in-depth computer knowledge. Users are therefore
! encouraged to load and test the software's suitability as regards their
! requirements in conditions enabling the security of their systems and/or
! data to be ensured and, more generally, to use and operate it in the
! same conditions as regards security.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/


#include "smooth_cuda.h"
#include "config.h"
#include <stdio.h>
// copies integer array from CPU host to GPU device
void copy_todevice_int(void** d_array_addr_ptr,int* h_array,int size){
   hipMalloc((void**)d_array_addr_ptr,size*sizeof(int));
   hipMemcpy((int*) *d_array_addr_ptr,h_array,size*sizeof(int),hipMemcpyHostToDevice);
}

void copy_todevice_realw(void** d_array_addr_ptr,realw* h_array,int size){
   hipMalloc((void**)d_array_addr_ptr,size*sizeof(realw));
   hipMemcpy((realw*) *d_array_addr_ptr,h_array,size*sizeof(realw),hipMemcpyHostToDevice);
}

__global__ void process_smooth(realw_const_p xstore_me,realw_const_p zstore_me,realw_const_p xstore_other,realw_const_p zstore_other, realw_const_p data_other, const realw sigma_h2_inv, const realw sigma_v2_inv, const int iker, const int nspec_me, const int nspec_other, const realw v_criterion, const realw h_criterion, realw_const_p jacobian, realw_p sum_data_smooth, realw_p normalisation,realw_const_p wgll_sq){

int ispec = blockIdx.x + gridDim.x*blockIdx.y;
int igll = threadIdx.x;
int gll_other;
realw x_me, z_me, x_other, z_other, coef, normalisation_slice;
realw dat;
__shared__ int sh_test[NGLL2];
__shared__ realw sh_x_other[NGLL2];
__shared__ realw sh_z_other[NGLL2];
__shared__ realw sh_jacobian[NGLL2];
__shared__ realw sh_wgll_sq[NGLL2];
__shared__ realw sh_data[NGLL2];

int n_loop = nspec_other/NGLL2 + 1;
x_me = xstore_me[NGLL2*ispec + igll ];
z_me = zstore_me[NGLL2*ispec + igll ];
sh_wgll_sq[igll]=wgll_sq[igll];
__syncthreads();

dat=0;
normalisation_slice=0;
//We test 32 spectral elements at a time
for (int i=0;i<n_loop;i++)
{
if (NGLL2*i + threadIdx.x < nspec_other){
x_other = xstore_other[i*NGLL2*NGLL2 + threadIdx.x*NGLL2 ];
z_other = zstore_other[i*NGLL2*NGLL2 + threadIdx.x*NGLL2 ];
}
sh_test[threadIdx.x] = ( NGLL2*i + threadIdx.x >= nspec_other || (x_me-x_other)*(x_me-x_other) > h_criterion || (z_me-z_other)*(z_me-z_other) > v_criterion ) ? 1 : 0 ;
__syncthreads();

//loop over each spectral element tested
for (int k=0;k<NGLL2;k++)
{
if (sh_test[k]) continue ;

//Load data from other slice to shared memory
sh_x_other[igll] = xstore_other[i*NGLL2*NGLL2 + k*NGLL2 + igll ];
sh_z_other[igll] = zstore_other[i*NGLL2*NGLL2 + k*NGLL2 + igll ];
sh_data[igll] = data_other[i*NGLL2*NGLL2 + k*NGLL2 + igll ];
sh_jacobian[igll] = jacobian[i*NGLL2*NGLL2 + k*NGLL2 + igll ];
__syncthreads();

for (int j=0;j<NGLL2;j++){

gll_other = (igll + j) % NGLL2;

x_other = sh_x_other[gll_other];
z_other = sh_z_other[gll_other];
coef = expf(- sigma_h2_inv*(x_me-x_other)*(x_me-x_other) - sigma_v2_inv*(z_me-z_other)*(z_me-z_other))*sh_jacobian[gll_other]*sh_wgll_sq[gll_other];
normalisation_slice = normalisation_slice + coef;
dat += sh_data[gll_other]*coef;
} //loop on each gll_other
} //loop on each spec_other tested
} //loop on each serie of 32 spec_other

sum_data_smooth[NGLL2*nspec_me*iker+NGLL2*ispec + igll] += dat;
normalisation[NGLL2*ispec + igll] += normalisation_slice;
}

__global__ void normalize_data(realw_p data_smooth, realw_const_p normalisation,int nker, int nspec_me){
int ispec = blockIdx.x + gridDim.x*blockIdx.y;
realw norm = normalisation[NGLL2*ispec + threadIdx.x];
for (int j=0;j<nker;j++) data_smooth[NGLL2*nspec_me*j + NGLL2*ispec + threadIdx.x] /= norm/nker;
}

extern "C"
void FC_FUNC_(prepare_gpu,
              PREPARE_GPU)(long * Container,
                          realw * xstore_me,
                          realw * zstore_me,
                          realw * sigma_h2_inv,
                          realw * sigma_v2_inv,
                          realw * h_criterion,
                          realw * v_criterion,
                          int * nspec_me,
                          int * nker,
                          realw * wgll_sq){

  Smooth_data* sp = (Smooth_data*)malloc( sizeof(Smooth_data) );
  *Container = (long)sp;

  copy_todevice_realw((void**)&sp->x_me,xstore_me, NGLL2*(*nspec_me));
  copy_todevice_realw((void**)&sp->z_me,zstore_me, NGLL2*(*nspec_me));
  copy_todevice_realw((void**)&sp->wgll_sq,wgll_sq, NGLL2);

  sp->sigma_h2_inv= *sigma_h2_inv;
  sp->sigma_v2_inv= *sigma_v2_inv;
  sp->h_criterion = *h_criterion;
  sp->v_criterion = *v_criterion;
  sp->nspec_me =  *nspec_me;
  sp->nker = *nker;

  print_CUDA_error_if_any(hipMalloc((void**)&sp->data_smooth,NGLL2*(*nspec_me)*(*nker)*sizeof(realw)),2000);
  print_CUDA_error_if_any(hipMemset(sp->data_smooth,0,NGLL2*(*nspec_me)*(*nker)*sizeof(realw)),2001);

  print_CUDA_error_if_any(hipMalloc((void**)&sp->normalisation,NGLL2*(*nspec_me)*sizeof(realw)),2002);
  print_CUDA_error_if_any(hipMemset(sp->normalisation,0,NGLL2*(*nspec_me)*sizeof(realw)),2003);
}

extern "C"
void FC_FUNC_(compute_smooth,
              COMPUTE_SMOOTH)(long * smooth_pointer,
                              realw * jacobian,
                              realw * xstore_other,
                              realw * zstore_other,
                              realw * data_other,
                              const int * nspec_other){
realw * x_other;
realw * z_other;
realw * d_data_other;
realw * d_jacobian;

Smooth_data * sp = (Smooth_data*)*smooth_pointer;

copy_todevice_realw((void**)&x_other,xstore_other,NGLL2*(*nspec_other));
copy_todevice_realw((void**)&z_other,zstore_other,NGLL2*(*nspec_other));
copy_todevice_realw((void**)&d_jacobian,jacobian,NGLL2*(*nspec_other));

dim3 grid(sp->nspec_me,1);
dim3 threads(NGLL2,1,1);

for (int i=0;i<sp->nker;i++)
{
copy_todevice_realw((void**)&d_data_other,&data_other[NGLL2*(*nspec_other)*i],NGLL2*(*nspec_other));
process_smooth<<<grid,threads>>>(sp->x_me,
                                 sp->z_me,
                                 x_other,
                                 z_other,
                                 d_data_other,
                                 sp->sigma_h2_inv,
                                 sp->sigma_v2_inv,
                                 i,
                                 sp->nspec_me,
                                 *nspec_other,
                                 sp->v_criterion,
                                 sp->h_criterion,
                                 d_jacobian,
                                 sp->data_smooth,
                                 sp->normalisation,
                                 sp->wgll_sq);
hipFree(d_data_other);
}

synchronize_cuda();
hipFree(x_other);
hipFree(z_other);
hipFree(d_jacobian);
}

extern "C"
void FC_FUNC_(get_smooth,
              GET_SMOOTH)(long * smooth_pointer,realw * data_smooth){

Smooth_data * sp = (Smooth_data*)*smooth_pointer;
dim3 grid(sp->nspec_me,1);
dim3 threads(NGLL2,1,1);

normalize_data<<<grid,threads>>>(sp->data_smooth,sp->normalisation,sp->nker,sp->nspec_me);
print_CUDA_error_if_any(hipMemcpy(data_smooth, sp->data_smooth,
                                     NGLL2*sp->nspec_me*sizeof(int)*sp->nker, hipMemcpyDeviceToHost),98012);

hipFree(sp->x_me);
hipFree(sp->z_me);
hipFree(sp->data_smooth);
hipFree(sp->wgll_sq);
hipFree(sp->normalisation);
free(sp);
}

