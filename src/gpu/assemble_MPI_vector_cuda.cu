#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                              CNRS, France
!                       and Princeton University, USA
!                 (there are currently many more authors!)
!                           (c) October 2017
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/

#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

// ASSEMBLY - mpi data transfer between CPU-GPU

/* ----------------------------------------------------------------------------------------------- */

// prepares and transfers the inter-element edge-nodes to the host to be MPI'd
// (elements on boundary)

extern "C"
void FC_FUNC_(transfer_boun_accel_from_device,
              TRANSFER_BOUN_ACCEL_FROM_DEVICE)(long* Mesh_pointer,
                                               realw* h_send_accel_buffer,
                                               const int* FORWARD_OR_ADJOINT){
TRACE("\ttransfer_boun_accel_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // checks if anything to do
  if (mp->size_mpi_buffer == 0) return;
  if (*FORWARD_OR_ADJOINT != 1 && *FORWARD_OR_ADJOINT != 3) return;

  int blocksize = BLOCKSIZE_TRANSFER;
  int size_padded = ((int)ceil(((double)mp->max_nibool_interfaces_ext_mesh)/((double)blocksize)))*blocksize;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // sets gpu arrays
  realw* accel;
  realw* send_buffer;
  if (*FORWARD_OR_ADJOINT == 1) {
    // forward
    accel = mp->d_accel;
    send_buffer = mp->d_send_accel_buffer;
  }else if (*FORWARD_OR_ADJOINT == 3) {
    // backward/reconstructed
    accel = mp->d_b_accel;
    send_buffer = mp->d_b_send_accel_buffer;
  }

  // Cuda timing
  //hipEvent_t start, stop;
  //start_timing_cuda(&start,&stop);

  prepare_boundary_accel_on_device<<<grid,threads,0,mp->compute_stream>>>(accel,
                                                                          send_buffer,
                                                                          mp->ninterface_elastic,
                                                                          mp->max_nibool_interfaces_ext_mesh,
                                                                          mp->d_nibool_interfaces_ext_mesh,
                                                                          mp->d_ibool_interfaces_ext_mesh,
                                                                          mp->d_inum_interfaces_elastic);


  // synchronizes
  // explicitly waits until previous compute stream finishes
  // (hipMemcpy implicitly synchronizes all other cuda operations)
  hipStreamSynchronize(mp->compute_stream);

  // copies buffer from GPU to CPU host
  print_CUDA_error_if_any(hipMemcpy(h_send_accel_buffer,send_buffer,
                          mp->size_mpi_buffer*sizeof(realw),hipMemcpyDeviceToHost),97001);

  // Cuda timing
  // finish timing of kernel+memcpy
  //stop_timing_cuda(&start,&stop,"prepare_boundary_accel_on_device");

  GPU_ERROR_CHECKING ("transfer_boun_accel_from_device");
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_boundary_from_device_a,
              TRANSFER_BOUNDARY_FROM_DEVICE_A)(long* Mesh_pointer) {

// asynchronous transfer from device to host

  TRACE("\ttransfer_boundary_from_device_a");

  Mesh* mp = (Mesh*)(*Mesh_pointer); // get Mesh from fortran integer wrapper

  // checks if anything to do
  if (mp->size_mpi_buffer == 0) return;

  int blocksize = BLOCKSIZE_TRANSFER;
  int size_padded = ((int)ceil(((double)mp->max_nibool_interfaces_ext_mesh)/((double)blocksize)))*blocksize;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  prepare_boundary_accel_on_device<<<grid,threads,0,mp->compute_stream>>>(mp->d_accel,mp->d_send_accel_buffer,
                                                                          mp->ninterface_elastic,
                                                                          mp->max_nibool_interfaces_ext_mesh,
                                                                          mp->d_nibool_interfaces_ext_mesh,
                                                                          mp->d_ibool_interfaces_ext_mesh,
                                                                          mp->d_inum_interfaces_elastic);
  // waits until kernel is finished before starting async memcpy
  //synchronize_cuda();
  // waits until previous compute stream finishes
  hipStreamSynchronize(mp->compute_stream);

  hipMemcpyAsync(mp->h_send_accel_buffer,mp->d_send_accel_buffer,
                  mp->size_mpi_buffer*sizeof(realw),hipMemcpyDeviceToHost,mp->copy_stream);

  GPU_ERROR_CHECKING ("transfer_boundary_from_device_a");
}

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(prepare_boundary_on_device,
              PREPARE_BOUNDARY_ON_DEVICE)(long* Mesh_pointer) {

// asynchronous transfer from device to host

  TRACE("\ttransfer_boundary_from_device_a");

  Mesh* mp = (Mesh*)(*Mesh_pointer); // get Mesh from fortran integer wrapper

  // checks if anything to do
  if (mp->size_mpi_buffer == 0) return;

  int blocksize = BLOCKSIZE_TRANSFER;
  int size_padded = ((int)ceil(((double)mp->max_nibool_interfaces_ext_mesh)/((double)blocksize)))*blocksize;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  prepare_boundary_accel_on_device<<<grid,threads,0,mp->compute_stream>>>(mp->d_accel,mp->d_send_accel_buffer,
                                                                          mp->ninterface_elastic,
                                                                          mp->max_nibool_interfaces_ext_mesh,
                                                                          mp->d_nibool_interfaces_ext_mesh,
                                                                          mp->d_ibool_interfaces_ext_mesh,
                                                                          mp->d_inum_interfaces_elastic);
  // waits until kernel is finished before starting async memcpy
  //synchronize_cuda();
  // waits until previous compute stream finishes
  hipStreamSynchronize(mp->compute_stream);

  GPU_ERROR_CHECKING ("prepare_boundary_on_device");
}



/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_boundary_to_device_a,
              TRANSFER_BOUNDARY_TO_DEVICE_A)(long* Mesh_pointer,
                                             realw* buffer_recv_vector_gpu,
                                             const int* max_nibool_interfaces_ext_mesh) {

// asynchronous transfer from host to device

  TRACE("transfer_boundary_to_device_a");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  if (mp->size_mpi_buffer > 0) {
    // copy on host memory
    memcpy(mp->h_recv_accel_buffer,buffer_recv_vector_gpu,mp->size_mpi_buffer*sizeof(realw));

    // asynchronous copy to GPU using copy_stream
    hipMemcpyAsync(mp->d_send_accel_buffer,mp->h_recv_accel_buffer,
                    mp->size_mpi_buffer*sizeof(realw),hipMemcpyHostToDevice,mp->copy_stream);
  }

  GPU_ERROR_CHECKING ("transfer_boundary_to_device_a");
}


/* ----------------------------------------------------------------------------------------------- */

// Assembly

/* ----------------------------------------------------------------------------------------------- */


// FORWARD_OR_ADJOINT == 1 for accel, and == 3 for b_accel
extern "C"
void FC_FUNC_(transfer_asmbl_accel_to_device,
              TRANSFER_ASMBL_ACCEL_TO_DEVICE)(long* Mesh_pointer,
                                              realw* buffer_recv_vector_gpu,
                                              const int* max_nibool_interfaces_ext_mesh,
                                              const int* nibool_interfaces_ext_mesh,
                                              const int* ibool_interfaces_ext_mesh,
                                              const int* FORWARD_OR_ADJOINT) {
  TRACE("\ttransfer_asmbl_accel_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // checks if anything to do
  if (mp->size_mpi_buffer == 0) return;
  if (*FORWARD_OR_ADJOINT != 1 && *FORWARD_OR_ADJOINT != 3) return;

  //daniel: todo - check if this copy is only needed for adjoint simulation, otherwise it is called asynchronously?
  if (*FORWARD_OR_ADJOINT == 1) {
    // Wait until previous copy stream finishes. We assemble while other compute kernels execute.
    hipStreamSynchronize(mp->copy_stream);
  }
  else if (*FORWARD_OR_ADJOINT == 3) {
    // explicitly synchronizes
    // (hipMemcpy implicitly synchronizes all other cuda operations)
    synchronize_cuda();

    print_CUDA_error_if_any(hipMemcpy(mp->d_b_send_accel_buffer, buffer_recv_vector_gpu,
                            mp->size_mpi_buffer*sizeof(realw),hipMemcpyHostToDevice),97001);
  }

  int blocksize = BLOCKSIZE_TRANSFER;
  int size_padded = ((int)ceil(((double)mp->max_nibool_interfaces_ext_mesh)/((double)blocksize)))*blocksize;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // sets gpu arrays
  realw* accel;
  realw* send_buffer;
  if (*FORWARD_OR_ADJOINT == 1) {
    // forward
    accel = mp->d_accel;
    send_buffer = mp->d_send_accel_buffer;
  }else if (*FORWARD_OR_ADJOINT == 3) {
    // backward/reconstructed
    accel = mp->d_b_accel;
    send_buffer = mp->d_b_send_accel_buffer;
  }

  //double start_time = get_time();
  // hipEvent_t start, stop;
  // realw time;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord( start, 0 );

  //assembles
  assemble_boundary_accel_on_device<<<grid,threads,0,mp->compute_stream>>>(accel,
                                                                           send_buffer,
                                                                           mp->ninterface_elastic,
                                                                           mp->max_nibool_interfaces_ext_mesh,
                                                                           mp->d_nibool_interfaces_ext_mesh,
                                                                           mp->d_ibool_interfaces_ext_mesh,
                                                                           mp->d_inum_interfaces_elastic);

  // hipEventRecord( stop, 0 );
  // hipEventSynchronize( stop );
  // hipEventElapsedTime( &time, start, stop );
  // hipEventDestroy( start );
  // hipEventDestroy( stop );
  // printf("Boundary Assemble Kernel Execution Time: %f ms\n",time);

  //double end_time = get_time();
  //printf("Elapsed time: %e\n",end_time-start_time);

  GPU_ERROR_CHECKING ("transfer_asmbl_accel_to_device");
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(sync_copy_from_device,
              SYNC_copy_FROM_DEVICE)(long* Mesh_pointer,
                                     int* iphase,
                                     realw* send_buffer) {

  TRACE("sync_copy_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); // get Mesh from fortran integer wrapper

  // Wait until async-memcpy of outer elements is finished and start MPI.
  if (*iphase != 2) { exit_on_cuda_error("sync_copy_from_device must be called for iphase == 2"); }

  if (mp->size_mpi_buffer > 0) {
    // waits for asynchronous copy to finish
    hipStreamSynchronize(mp->copy_stream);

    // There have been problems using the pinned-memory with MPI, so
    // we copy the buffer into a non-pinned region.
    memcpy(send_buffer,mp->h_send_accel_buffer,mp->size_mpi_buffer*sizeof(float));
  }
  // memory copy is now finished, so non-blocking MPI send can proceed

  GPU_ERROR_CHECKING ("sync_copy_from_device");
}

