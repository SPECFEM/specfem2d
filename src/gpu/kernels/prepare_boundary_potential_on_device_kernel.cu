#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/


// prepares a device array with with all inter-element edge-nodes -- this
// is followed by a memcpy and MPI operations

__global__ void prepare_boundary_potential_on_device(realw* d_potential_dot_dot_acoustic,
                                                     realw* d_send_potential_dot_dot_buffer,
                                                     const int ninterface_ac,
                                                     const int max_nibool_interfaces_ext_mesh,
                                                     const int* d_nibool_interfaces_ext_mesh,
                                                     const int* d_ibool_interfaces_ext_mesh,
                                                     const int* inum_inter_acoustic) {

  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;
  int ientry,iglob,num_int;

  for(int iinterface=0; iinterface < ninterface_ac; iinterface++) {

   num_int=inum_inter_acoustic[iinterface]-1;

    if (id<d_nibool_interfaces_ext_mesh[num_int]) {

      // entry in interface array
      ientry = id + max_nibool_interfaces_ext_mesh*num_int;
      // global index in wavefield
      iglob = d_ibool_interfaces_ext_mesh[ientry] - 1;

      d_send_potential_dot_dot_buffer[ientry] = d_potential_dot_dot_acoustic[iglob];
    }
  }

}

