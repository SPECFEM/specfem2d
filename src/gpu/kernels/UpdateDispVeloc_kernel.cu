#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/


__global__ void UpdateDispVeloc_kernel(realw* displ,
                                       realw* veloc,
                                       realw* accel,
                                       int size,
                                       realw deltat,
                                       realw deltatsqover2,
                                       realw deltatover2) {

  // two dimensional array of blocks on grid where each block has one dimensional array of threads
  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;

  // because of block and grid sizing problems, there is a small
  // amount of buffer at the end of the calculation
  if (id < size) {
    displ[id] = displ[id] + deltat*veloc[id] + deltatsqover2*accel[id];
    veloc[id] = veloc[id] + deltatover2*accel[id];
    accel[id] = 0.0f; // can do this using memset...not sure if faster,probably not
  }

// -----------------
// total of: 6 FLOP per thread (without int id calculation at beginning)
//
//           8 * 4 BYTE = 32 DRAM accesses per thread
//
// arithmetic intensity: 6 FLOP / 32 BYTES ~ 0.19 FLOP/BYTE
// -----------------
// nvprof: 24599250 flops for 4099875 threads -> 6 FLOP per thread
}


