#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/


// prepares a device array with with all inter-element edge-nodes -- this
// is followed by a memcpy and MPI operations

__global__ void prepare_boundary_accel_on_device(realw* d_accel, realw* d_send_accel_buffer,
                                                 const int ninterface_el,
                                                 const int max_nibool_interfaces_ext_mesh,
                                                 const int* d_nibool_interfaces_ext_mesh,
                                                 const int* d_ibool_interfaces_ext_mesh,
                                                 const int* inum_inter_elastic) {

  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;
  int ientry,iglob,num_int;

  for( int iinterface=0; iinterface < ninterface_el; iinterface++) {

     num_int=inum_inter_elastic[iinterface]-1;

      if (id < d_nibool_interfaces_ext_mesh[num_int]) {


      // entry in interface array
      ientry = id + max_nibool_interfaces_ext_mesh*num_int;
      // global index in wavefield
      iglob = d_ibool_interfaces_ext_mesh[ientry] - 1;

      d_send_accel_buffer[2*ientry] = d_accel[2*iglob];
      d_send_accel_buffer[2*ientry + 1 ] = d_accel[2*iglob + 1];

    }
  }

}

