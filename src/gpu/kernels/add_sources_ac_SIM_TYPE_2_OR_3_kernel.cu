#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                            S P E C F E M 2 D
!                            -----------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/


__global__ void add_sources_ac_SIM_TYPE_2_OR_3_kernel(realw* potential_dot_dot_acoustic,
                                                      realw* source_adjoint,
                                                      realw* xir_store,
                                                      realw* gammar_store,
                                                      int* d_ibool,
                                                      int* ispec_is_acoustic,
                                                      int* ispec_selected_rec_loc,
                                                      int it,
                                                      int nadj_rec_local,
                                                      //realw* kappastore,
                                                      int NSTEP ) {

  int irec_local = blockIdx.x + gridDim.x*blockIdx.y;

  // because of grid shape, irec_local can be too big
  if (irec_local < nadj_rec_local) {

    int ispec = ispec_selected_rec_loc[irec_local] - 1;

    if (ispec_is_acoustic[ispec]) {
      int i = threadIdx.x;
      int j = threadIdx.y;

      int iglob = d_ibool[INDEX3_PADDED(NGLLX,NGLLX,i,j,ispec)] - 1;

      //realw  kappal = kappastore[INDEX3(NGLLX,NGLLX,i,j,ispec)];
      realw  xir = xir_store[INDEX2(nadj_rec_local,irec_local,i)];
      realw  gammar = gammar_store[INDEX2(nadj_rec_local,irec_local,j)];
      realw  source_adj = source_adjoint[INDEX3(nadj_rec_local,NSTEP,irec_local,it,0)];

      // adjoint source of Peter et al. (A8):
      //   f^adj = - sum_i \partial_t^2 (p^syn - p^obs)(T-t) \delta(x - x_i)
      // note that using the adjoint source derived from the optimization problem, there is no 1/kappa term applied
      // to the adjoint source. the negative sign also is part of the construction of the adjoint source.
      //
      // since we don't know which formulation of adjoint source is used for the input, we add the adjoint source as is,
      // without 1/kappa factor, and with a positive sign.
      realw stf = source_adj * gammar * xir;
      atomicAdd(&potential_dot_dot_acoustic[iglob],stf);
    }
  }
}

