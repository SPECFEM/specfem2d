#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/


__global__ void enforce_free_surface_cuda_kernel(realw_p potential_acoustic,
                                                 realw_p potential_dot_acoustic,
                                                 realw_p potential_dot_dot_acoustic,
                                                 const int num_free_surface_faces,
                                                 const int* free_surface_ispec,
                                                 const int* free_surface_ijk,
                                                 const int* d_ibool,
                                                 const int* ispec_is_acoustic) {
  // gets spectral element face id
  int iface = blockIdx.x + gridDim.x*blockIdx.y;

  // for all faces on free surface
  if (iface < num_free_surface_faces) {

    int ispec = free_surface_ispec[iface]-1;

    // checks if element is in acoustic domain
    if (ispec_is_acoustic[ispec]) {

      // gets global point index
      int igll = threadIdx.x + threadIdx.y*blockDim.x;

      int i = free_surface_ijk[INDEX3(NDIM,NGLLX,0,igll,iface)] - 1; // (1,igll,iface)
      int j = free_surface_ijk[INDEX3(NDIM,NGLLX,1,igll,iface)] - 1;

      int iglob = d_ibool[INDEX3_PADDED(NGLLX,NGLLX,i,j,ispec)] - 1;

      // sets potentials to zero at free surface
      potential_acoustic[iglob] = 0.f;
      potential_dot_acoustic[iglob] = 0.f;
      potential_dot_dot_acoustic[iglob] = 0.f;
    }
  }
}

