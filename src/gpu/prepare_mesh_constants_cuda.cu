#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                            S P E C F E M 2 D
!                            -----------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                              CNRS, France
!                       and Princeton University, USA
!                 (there are currently many more authors!)
!                           (c) October 2017
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/

#include "mesh_constants_cuda.h"
#include "prepare_constants_cuda.h"

// additional pragma messages for compilation info
#ifdef USE_TEXTURES_FIELDS
#pragma message ("Compiling with: USE_TEXTURES_FIELDS enabled\n")
#endif
#ifdef USE_TEXTURES_CONSTANTS
#pragma message ("Compiling with: USE_TEXTURES_CONSTANTS enabled\n")
#endif
#ifdef USE_LAUNCH_BOUNDS
#pragma message ("\nCompiling with: USE_LAUNCH_BOUNDS enabled\n")
#endif

// texture arrays
#ifdef USE_OLDER_CUDA4_GPU
#else
  #ifdef USE_TEXTURES_FIELDS
    // elastic
    extern realw_texture d_displ_tex;
    extern realw_texture d_accel_tex;
    // backward/reconstructed
    extern realw_texture d_b_displ_tex;
    extern realw_texture d_b_accel_tex;
    // acoustic
    extern realw_texture d_potential_tex;
    extern realw_texture d_potential_dot_dot_tex;
    // backward/reconstructed
    extern realw_texture d_b_potential_tex;
    extern realw_texture d_b_potential_dot_dot_tex;
  #endif
  #ifdef USE_TEXTURES_CONSTANTS
    extern realw_texture d_hprime_xx_tex;
    extern size_t d_hprime_xx_tex_offset;
    extern realw_texture d_wxgll_xx_tex;
    extern size_t d_wxgll_xx_tex_offset;
  #endif
#endif


/* ----------------------------------------------------------------------------------------------- */

// GPU preparation

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_constants_device,
              PREPARE_CONSTANTS_DEVICE)(long* Mesh_pointer,
                                        int* h_NGLLX, int* NSPEC_AB, int* NGLOB_AB,
                                        realw* h_xix, realw* h_xiz,
                                        realw* h_gammax, realw* h_gammaz,
                                        realw* h_kappav, realw* h_muv,
                                        int* h_ibool,
                                        int* num_interfaces_ext_mesh, int* max_nibool_interfaces_ext_mesh,
                                        int* h_nibool_interfaces_ext_mesh, int* h_ibool_interfaces_ext_mesh,
                                        realw* h_hprime_xx, realw* h_hprimewgll_xx,
                                        realw* h_wxgll,
                                        int* STACEY_BOUNDARY_CONDITIONS,
                                        int* PML_BOUNDARY_CONDITIONS,
                                        int* h_ispec_is_inner,
                                        int* nsources_local_f,
                                        realw* h_sourcearrays, realw * h_source_time_function,
                                        int* NSTEP,
                                        int* h_ispec_selected_source,
                                        int* h_ispec_selected_rec_loc,
                                        int* nrec_local,
                                        realw * h_cosrot,realw * h_sinrot,
                                        int* SIMULATION_TYPE,
                                        int* P_SV,
                                        int* nspec_acoustic,int* nspec_elastic,
                                        int* ispec_is_acoustic, int* ispec_is_elastic,
                                        int* h_myrank,
                                        int* SAVE_FORWARD,
                                        realw* h_xir_store, realw* h_gammar_store,
                                        int* h_NSIGTYPE, int* h_seismotypeVec,
                                        int* nlength_seismogram) {

  TRACE("prepare_constants_device");

  // allocates mesh parameter structure
  Mesh* mp = (Mesh*) malloc( sizeof(Mesh) );
  if (mp == NULL) exit_on_error("error allocating mesh pointer");
  *Mesh_pointer = (long)mp;

  // sets processes mpi rank
  mp->myrank = *h_myrank;

  // sets global parameters
  mp->NSPEC_AB = *NSPEC_AB;
  mp->NGLOB_AB = *NGLOB_AB;

  // constants
  mp->simulation_type = *SIMULATION_TYPE;
  mp->stacey_absorbing_conditions = *STACEY_BOUNDARY_CONDITIONS;
  mp->pml_boundary_conditions = *PML_BOUNDARY_CONDITIONS;
  mp->save_forward = *SAVE_FORWARD;
  mp->p_sv = *P_SV;

  // safety check
  if (*h_NGLLX != NGLLX) {
    exit_on_error("make sure that the NGLL constants are equal in the two files:\n" \
                  "  setup/constants.h and src/gpu/mesh_constants_cuda.h\n" \
                  "and then please re-compile; also make sure that the value of NGLL3_PADDED " \
                  "is consistent with the value of NGLL\n");
  }

  // sets constant arrays
  setConst_hprime_xx(h_hprime_xx,mp);
  // setConst_hprime_zz(h_hprime_zz,mp); // only needed if NGLLX != NGLLY != NGLLZ

  setConst_hprimewgll_xx(h_hprimewgll_xx,mp);
  //setConst_hprimewgll_zz(h_hprimewgll_zz,mp); // only needed if NGLLX != NGLLY != NGLLZ

  setConst_wxgll(h_wxgll,mp);

  // Using texture memory for the hprime-style constants is slower on
  // Fermi generation hardware, but *may* be faster on Kepler
  // generation. We will reevaluate this again, so might as well leave
  // in the code with with #USE_TEXTURES_FIELDS not-defined.
  #ifdef USE_TEXTURES_CONSTANTS
  {
    // checks that realw is a float
    if (sizeof(realw) != sizeof(float)) exit_on_error("TEXTURES only work with realw selected as float");

    // note: device memory returned by hipMalloc guarantees that the offset is 0,
    //       however here we use the global memory array d_hprime_xx and need to provide an offset variable for the function call
    // binds texture
    #ifdef USE_OLDER_CUDA4_GPU
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<realw>();

      const textureReference* d_hprime_xx_tex_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_hprime_xx_tex_ptr, HIP_SYMBOL("d_hprime_xx_tex")), 1101);
      print_CUDA_error_if_any(hipBindTexture(&d_hprime_xx_tex_offset, d_hprime_xx_tex_ptr, mp->d_hprime_xx, &channelDesc, sizeof(realw)*(NGLL2)), 1102);

      const textureReference* d_wxgll_xx_tex_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_wxgll_xx_tex_ptr, HIP_SYMBOL("d_wxgll_xx_tex")), 1103);
      print_CUDA_error_if_any(hipBindTexture(&d_wxgll_xx_tex_offset, d_wxgll_xx_tex_ptr, mp->d_wxgll, &channelDesc, sizeof(realw)*(NGLL2)), 1104);

   #else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<realw>();

      print_CUDA_error_if_any(hipBindTexture(&d_hprime_xx_tex_offset, &d_hprime_xx_tex, mp->d_hprime_xx,
                                              &channelDesc, sizeof(realw)*(NGLL2)), 1105);
      //printf("Bind texture hprime_xx offset = %lu\n",d_hprime_xx_tex_offset);

      print_CUDA_error_if_any(hipBindTexture(&d_wxgll_xx_tex_offset, &d_wxgll_xx_tex, mp->d_wxgll,
                                              &channelDesc, sizeof(realw)*(NGLLX)), 1106);
      //printf("Bind texture wxgll_xx offset = %lu\n",d_wxgll_xx_tex_offset);
   #endif
  }
  #endif

  // mesh
  // Assuming NGLLX=5. Padded is then 32 (5^2+3)
  int size_padded = NGLL2_PADDED * (mp->NSPEC_AB);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xix, size_padded*sizeof(realw)),1001);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xiz, size_padded*sizeof(realw)),1002);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammax, size_padded*sizeof(realw)),1003);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammaz, size_padded*sizeof(realw)),1004);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_kappav, size_padded*sizeof(realw)),1005);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_muv, size_padded*sizeof(realw)),1006);

  print_CUDA_error_if_any(hipMemcpy2D(mp->d_xix, NGLL2_PADDED*sizeof(realw),
                                       h_xix, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1501);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_xiz, NGLL2_PADDED*sizeof(realw),
                                       h_xiz, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1503);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_gammax, NGLL2_PADDED*sizeof(realw),
                                       h_gammax, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1507);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_gammaz, NGLL2_PADDED*sizeof(realw),
                                       h_gammaz, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1509);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_kappav, NGLL2_PADDED*sizeof(realw),
                                       h_kappav, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1510);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_muv, NGLL2_PADDED*sizeof(realw),
                                       h_muv, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1511);

  // global indexing (padded)
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibool, size_padded*sizeof(int)),1600);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_ibool, NGLL2_PADDED*sizeof(int),
                                       h_ibool, NGLL2*sizeof(int), NGLL2*sizeof(int),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1601);

  // prepare interprocess-edge exchange information
  mp->num_interfaces_ext_mesh = *num_interfaces_ext_mesh;
  mp->max_nibool_interfaces_ext_mesh = *max_nibool_interfaces_ext_mesh;
  if (mp->num_interfaces_ext_mesh > 0) {
    copy_todevice_int((void**)&mp->d_nibool_interfaces_ext_mesh,h_nibool_interfaces_ext_mesh,
                      mp->num_interfaces_ext_mesh);
    copy_todevice_int((void**)&mp->d_ibool_interfaces_ext_mesh,h_ibool_interfaces_ext_mesh,
                      (mp->num_interfaces_ext_mesh)*(mp->max_nibool_interfaces_ext_mesh));
    //int blocksize = BLOCKSIZE_TRANSFER;
    //int size_padded = ((int)ceil(((double)(mp->max_nibool_interfaces_ext_mesh))/((double)blocksize)))*blocksize;
  }
  mp->size_mpi_buffer = 0;
  mp->size_mpi_buffer_potential = 0;

  // streams
  // setup two streams, one for compute and one for host<->device memory copies
  hipStreamCreate(&mp->compute_stream);
  // copy stream (needed to transfer mpi buffers)
  if (mp->num_interfaces_ext_mesh * mp->max_nibool_interfaces_ext_mesh > 0) {
    hipStreamCreate(&mp->copy_stream);
  }

  // inner elements
  copy_todevice_int((void**)&mp->d_ispec_is_inner,h_ispec_is_inner,mp->NSPEC_AB);

  // sources
  mp->nsources_local = *nsources_local_f;
  if (mp->nsources_local > 0){
    copy_todevice_realw((void**)&mp->d_source_time_function,h_source_time_function,(*NSTEP)*(mp->nsources_local));
    copy_todevice_realw((void**)&mp->d_sourcearrays,h_sourcearrays,NDIM*NGLL2*mp->nsources_local);
    copy_todevice_int((void**)&mp->d_ispec_selected_source,h_ispec_selected_source,mp->nsources_local);
  }

  // receiver stations
  mp->nrec_local = *nrec_local; // number of receiver located in this partition

  // Alexis Bottero (AB AB) defined all these arrays in order to be able to write several signal types with one simulation
  mp->h_NSIGTYPE = *h_NSIGTYPE;
  mp->h_seismotypeVec = h_seismotypeVec;

  //only in case needed on GPU..
  //copy_todevice_int((void**)&mp->d_seismotypeVec,h_seismotypeVec,mp->h_NSIGTYPE);

  // note that: size of size(ispec_selected_rec_loc) = nrec_local
  if (mp->nrec_local > 0){
    // pointer look-up table
    mp->h_seismograms = (realw**) malloc(sizeof(realw*) * mp->h_NSIGTYPE);
    mp->d_seismograms = (realw**) malloc(sizeof(realw*) * mp->h_NSIGTYPE);
    // allocates seismogram buffers
    for(int i_sig = 0; i_sig < mp->h_NSIGTYPE; i_sig++) {
      if (mp->h_seismotypeVec[i_sig] != 0){
        // buffer array on GPU
        print_CUDA_error_if_any(hipMalloc((void**)&mp->d_seismograms[i_sig],
                                           2*(*nlength_seismogram)*(mp->nrec_local)*sizeof(realw)),1303);
        // pinned memory on CPU (for async memory copies which are not used yet, but just in case..)
        print_CUDA_error_if_any(hipHostMalloc((void**)&(mp->h_seismograms[i_sig]),
                                               2*(*nlength_seismogram)*(mp->nrec_local)*sizeof(realw)),8004);
      }else{
        mp->h_seismograms[i_sig] = NULL;
        mp->d_seismograms[i_sig] = NULL;
      }
    }

    copy_todevice_realw((void**)&mp->d_cosrot,h_cosrot,mp->nrec_local);
    copy_todevice_realw((void**)&mp->d_sinrot,h_sinrot,mp->nrec_local);

    copy_todevice_realw((void**)&mp->d_xir_store_loc,h_xir_store,(mp->nrec_local)*NGLLX);
    copy_todevice_realw((void**)&mp->d_gammar_store_loc,h_gammar_store,(mp->nrec_local)*NGLLX);

    copy_todevice_int((void**)&mp->d_ispec_selected_rec_loc,h_ispec_selected_rec_loc,mp->nrec_local);
  }

  // number of elements per domain
  mp->nspec_acoustic = *nspec_acoustic;
  mp->nspec_elastic  = *nspec_elastic;

  // element domain flags (needed for seismogram outputs)
  copy_todevice_int((void**)&mp->d_ispec_is_acoustic,ispec_is_acoustic,mp->NSPEC_AB);
  copy_todevice_int((void**)&mp->d_ispec_is_elastic,ispec_is_elastic,mp->NSPEC_AB);

  // JC JC here we will need to add GPU support for the new C-PML routines

  GPU_ERROR_CHECKING ("prepare_constants_device");
}


/* ----------------------------------------------------------------------------------------------- */

// for ACOUSTIC simulations

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(prepare_fields_acoustic_device,
              PREPARE_FIELDS_ACOUSTIC_DEVICE)(long* Mesh_pointer,
                                              realw* rmass_acoustic, realw* rhostore, realw* kappastore,
                                              int* num_phase_ispec_acoustic, int* phase_ispec_inner_acoustic,
                                              int* num_free_surface_faces,
                                              int* free_surface_ispec,
                                              int* free_surface_ijk,
                                              int* ELASTIC_SIMULATION,
                                              int* num_coupling_ac_el_faces,
                                              int* coupling_ac_el_ispec,
                                              int* coupling_ac_el_ijk,
                                              realw* coupling_ac_el_normal,
                                              realw* coupling_ac_el_jacobian2Dw,
                                              int * h_ninterface_acoustic,int * h_inum_interfaces_acoustic,
                                              int* ATTENUATION_VISCOACOUSTIC,
                                              realw* h_A_newmark,realw* h_B_newmark,
                                              int* NO_BACKWARD_RECONSTRUCTION,realw* h_no_backward_acoustic_buffer) {

  TRACE("prepare_fields_acoustic_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer);

  // allocates arrays on device (GPU)
  int size = mp->NGLOB_AB;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_acoustic),sizeof(realw)*size),2001);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_dot_acoustic),sizeof(realw)*size),2002);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_dot_dot_acoustic),sizeof(realw)*size),2003);
  // initializes values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_potential_acoustic,0,sizeof(realw)*size),2007);
  print_CUDA_error_if_any(hipMemset(mp->d_potential_dot_acoustic,0,sizeof(realw)*size),2007);
  print_CUDA_error_if_any(hipMemset(mp->d_potential_dot_dot_acoustic,0,sizeof(realw)*size),2007);

  #ifdef USE_TEXTURES_FIELDS
  {
    #ifdef USE_OLDER_CUDA4_GPU
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      const textureReference* d_potential_tex_ref_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_potential_tex_ref_ptr, HIP_SYMBOL("d_potential_tex")), 2001);
      print_CUDA_error_if_any(hipBindTexture(0, d_potential_tex_ref_ptr, mp->d_potential_acoustic, &channelDesc, sizeof(realw)*size), 2001);

      const textureReference* d_potential_dot_dot_tex_ref_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_potential_dot_dot_tex_ref_ptr, HIP_SYMBOL("d_potential_dot_dot_tex")), 2003);
      print_CUDA_error_if_any(hipBindTexture(0, d_potential_dot_dot_tex_ref_ptr, mp->d_potential_dot_dot_acoustic, &channelDesc, sizeof(realw)*size), 2003);
    #else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      print_CUDA_error_if_any(hipBindTexture(0, &d_potential_tex, mp->d_potential_acoustic, &channelDesc, sizeof(realw)*size), 2001);
      print_CUDA_error_if_any(hipBindTexture(0, &d_potential_dot_dot_tex, mp->d_potential_dot_dot_acoustic, &channelDesc, sizeof(realw)*size), 2003);
    #endif
  }
  #endif

  // mpi buffer
  mp->size_mpi_buffer_potential = (mp->num_interfaces_ext_mesh) * (mp->max_nibool_interfaces_ext_mesh);
  if (mp->size_mpi_buffer_potential > 0) {
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_send_potential_dot_dot_buffer),mp->size_mpi_buffer_potential *sizeof(realw)),2004);
  }

  // mass matrix
  copy_todevice_realw((void**)&mp->d_rmass_acoustic,rmass_acoustic,mp->NGLOB_AB);

  // density
  // padded array
  // Assuming NGLLX==5. Padded is then 32 (5^2+3)
  int size_padded = NGLL2_PADDED * mp->NSPEC_AB;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rhostore),size_padded*sizeof(realw)),2006);
  // transfer constant element data with padding
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_rhostore, NGLL2_PADDED*sizeof(realw),
                                       rhostore, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),2106);

  // non-padded array
  copy_todevice_realw((void**)&mp->d_kappastore,kappastore,NGLL2*mp->NSPEC_AB);

  // phase elements
  mp->num_phase_ispec_acoustic = *num_phase_ispec_acoustic;
  copy_todevice_int((void**)&mp->d_phase_ispec_inner_acoustic,phase_ispec_inner_acoustic,
                    2*mp->num_phase_ispec_acoustic);

  // allocate surface arrays
  mp->num_free_surface_faces = *num_free_surface_faces;
  if (mp->num_free_surface_faces > 0) {
    copy_todevice_int((void**)&mp->d_free_surface_ispec,free_surface_ispec,mp->num_free_surface_faces);
    copy_todevice_int((void**)&mp->d_free_surface_ijk,free_surface_ijk,2*NGLLX*mp->num_free_surface_faces);
  }

  // coupling with elastic parts
  if (*ELASTIC_SIMULATION && *num_coupling_ac_el_faces > 0) {
    copy_todevice_int((void**)&mp->d_coupling_ac_el_ispec,coupling_ac_el_ispec,(*num_coupling_ac_el_faces));
    copy_todevice_int((void**)&mp->d_coupling_ac_el_ijk,coupling_ac_el_ijk,2*NGLLX*(*num_coupling_ac_el_faces));
    copy_todevice_realw((void**)&mp->d_coupling_ac_el_normal,coupling_ac_el_normal,
                        2*NGLLX*(*num_coupling_ac_el_faces));
    copy_todevice_realw((void**)&mp->d_coupling_ac_el_jacobian2Dw,coupling_ac_el_jacobian2Dw,
                        NGLLX*(*num_coupling_ac_el_faces));
  }

  mp->ninterface_acoustic = *h_ninterface_acoustic;
  copy_todevice_int((void**)&mp->d_inum_interfaces_acoustic,h_inum_interfaces_acoustic,mp->num_interfaces_ext_mesh);

  // attenuation
  if (*ATTENUATION_VISCOACOUSTIC) {
    copy_todevice_realw((void**)&mp->d_A_newmark_acous,h_A_newmark,NGLL2*mp->NSPEC_AB*N_SLS);
    copy_todevice_realw((void**)&mp->d_B_newmark_acous,h_B_newmark,NGLL2*mp->NSPEC_AB*N_SLS);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_e1_acous,mp->NSPEC_AB*sizeof(realw)*NGLL2*N_SLS),2202);
    print_CUDA_error_if_any(hipMemset(mp->d_e1_acous,0,mp->NSPEC_AB*sizeof(realw)*NGLL2*N_SLS),2203);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_sum_forces_old,mp->NSPEC_AB*sizeof(realw)*NGLL2),2204);
    print_CUDA_error_if_any(hipMemset(mp->d_sum_forces_old,0,mp->NSPEC_AB*sizeof(realw)*NGLL2),2205);
  }

  if (*NO_BACKWARD_RECONSTRUCTION){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_acoustic_buffer),mp->NGLOB_AB*sizeof(realw)),2206);
    hipStreamCreateWithFlags(&mp->copy_stream_no_backward,hipStreamNonBlocking);
    hipHostRegister(h_no_backward_acoustic_buffer,3*mp->NGLOB_AB*sizeof(realw),0);
    hipEventCreate(&mp->transfer_is_complete1);
    hipEventCreate(&mp->transfer_is_complete2);
  }

  GPU_ERROR_CHECKING ("prepare_fields_acoustic_device");
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_acoustic_adj_dev,
              PREPARE_FIELDS_ACOUSTIC_ADJ_DEV)(long* Mesh_pointer,
                                               int* APPROXIMATE_HESS_KL,
                                               int* ATTENUATION_VISCOACOUSTIC,
                                               int* NO_BACKWARD_RECONSTRUCTION) {

  TRACE("prepare_fields_acoustic_adj_dev");

  Mesh* mp = (Mesh*)(*Mesh_pointer);

  // kernel simulations
  if (mp->simulation_type != 3 ) return;

  // allocates backward/reconstructed arrays on device (GPU)
  int size = mp->NGLOB_AB * sizeof(realw);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_acoustic),size),3014);
  // initializes values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_b_potential_acoustic,0,size),3007);

  if (! *NO_BACKWARD_RECONSTRUCTION){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_dot_acoustic),size),3015);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_dot_dot_acoustic),size),3016);
    // initializes values to zero
    print_CUDA_error_if_any(hipMemset(mp->d_b_potential_dot_acoustic,0,size),3007);
    print_CUDA_error_if_any(hipMemset(mp->d_b_potential_dot_dot_acoustic,0,size),3007);
  }

  #ifdef USE_TEXTURES_FIELDS
  {
    #ifdef USE_OLDER_CUDA4_GPU
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      const textureReference* d_b_potential_tex_ref_ptr;

      print_CUDA_error_if_any(hipGetTextureReference(&d_b_potential_tex_ref_ptr, HIP_SYMBOL("d_b_potential_tex")), 3001);
      print_CUDA_error_if_any(hipBindTexture(0, d_b_potential_tex_ref_ptr, mp->d_b_potential_acoustic, &channelDesc, size), 3001);

      if (! *NO_BACKWARD_RECONSTRUCTION){
        const textureReference* d_b_potential_dot_dot_tex_ref_ptr;
        print_CUDA_error_if_any(hipGetTextureReference(&d_b_potential_dot_dot_tex_ref_ptr, HIP_SYMBOL("d_b_potential_dot_dot_tex")),3003);
        print_CUDA_error_if_any(hipBindTexture(0, d_b_potential_dot_dot_tex_ref_ptr, mp->d_b_potential_dot_dot_acoustic, &channelDesc, size), 3003);
      }
    #else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      print_CUDA_error_if_any(hipBindTexture(0, &d_b_potential_tex, mp->d_b_potential_acoustic, &channelDesc, size), 3001);
      if (! *NO_BACKWARD_RECONSTRUCTION) print_CUDA_error_if_any(hipBindTexture(0, &d_b_potential_dot_dot_tex, mp->d_b_potential_dot_dot_acoustic, &channelDesc, size), 3003);
    #endif
  }
  #endif

  // allocates kernels
  size = NGLL2 * mp->NSPEC_AB * sizeof(realw);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_ac_kl),size),3017);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_kappa_ac_kl),size),3018);
  // initializes kernel values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_rho_ac_kl,0,size),3019);
  print_CUDA_error_if_any(hipMemset(mp->d_kappa_ac_kl,0,size),3020);

  // preconditioner
  if (*APPROXIMATE_HESS_KL) {
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_ac_kl),size),3030);
    // initializes with zeros
    print_CUDA_error_if_any(hipMemset(mp->d_hess_ac_kl,0,size),3031);
  }

  if (*ATTENUATION_VISCOACOUSTIC && (! *NO_BACKWARD_RECONSTRUCTION) ) {
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_sum_forces_old),size),3040);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_e1_acous),size*N_SLS),3041);
  }

  // mpi buffer
  if (mp->size_mpi_buffer_potential > 0 && (! *NO_BACKWARD_RECONSTRUCTION)) {
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_send_potential_dot_dot_buffer),mp->size_mpi_buffer_potential*sizeof(realw)),3014);
  }

  GPU_ERROR_CHECKING ("prepare_fields_acoustic_adj_dev");
}


/* ----------------------------------------------------------------------------------------------- */

// for ELASTIC simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_elastic_device,
              PREPARE_FIELDS_ELASTIC_DEVICE)(long* Mesh_pointer,
                                             realw* rmassx, realw* rmassz,
                                             int* num_phase_ispec_elastic,
                                             int* phase_ispec_inner_elastic,
                                             int* ispec_is_anisotropic,
                                             int* ANISOTROPY,
                                             realw *c11store,realw *c12store,realw *c13store,
                                             realw *c15store,
                                             realw *c23store,
                                             realw *c25store,realw *c33store,
                                             realw *c35store,
                                             realw *c55store,
                                             int* h_ninterface_elastic,int * h_inum_interfaces_elastic,
                                             int* ATTENUATION_VISCOELASTIC,
                                             realw* h_A_newmark_mu,realw* h_B_newmark_mu,
                                             realw* h_A_newmark_kappa,realw* h_B_newmark_kappa) {

  TRACE("prepare_fields_elastic_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer);
  int size,size_padded;

  // debug
  //printf("prepare_fields_elastic_device: rank %d - wavefield setup\n",mp->myrank);
  //synchronize_mpi();

  // elastic wavefields
  size = NDIM * mp->NGLOB_AB;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_displ),sizeof(realw)*size),4001);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_veloc),sizeof(realw)*size),4002);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_accel),sizeof(realw)*size),4003);
  // initializes values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_displ,0,sizeof(realw)*size),4007);
  print_CUDA_error_if_any(hipMemset(mp->d_veloc,0,sizeof(realw)*size),4007);
  print_CUDA_error_if_any(hipMemset(mp->d_accel,0,sizeof(realw)*size),4007);

  #ifdef USE_TEXTURES_FIELDS
  {
    #ifdef USE_OLDER_CUDA4_GPU
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      const textureReference* d_displ_tex_ref_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_displ_tex_ref_ptr, HIP_SYMBOL("d_displ_tex")), 4004);
      print_CUDA_error_if_any(hipBindTexture(0, d_displ_tex_ref_ptr, mp->d_displ, &channelDesc, sizeof(realw)*size), 4005);
    #else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      print_CUDA_error_if_any(hipBindTexture(0, &d_displ_tex, mp->d_displ, &channelDesc, sizeof(realw)*size), 4008);
    #endif
  }
  #endif


  // debug
  //synchronize_mpi();

  // MPI buffer
  mp->size_mpi_buffer = NDIM * (mp->num_interfaces_ext_mesh) * (mp->max_nibool_interfaces_ext_mesh);
  if (mp->size_mpi_buffer > 0) {
    // note: Allocate pinned mpi-buffers.
    //       MPI buffers use pinned memory allocated by hipHostMalloc, which
    //       enables the use of asynchronous memory copies from host <-> device
    // send buffer
    print_CUDA_error_if_any(hipHostMalloc((void**)&(mp->h_send_accel_buffer),sizeof(realw)*(mp->size_mpi_buffer)),8004);
    //mp->send_buffer = (float*)malloc((mp->size_mpi_buffer)*sizeof(float));
    // adjoint
    //print_CUDA_error_if_any(hipHostMalloc((void**)&(mp->h_send_b_accel_buffer),sizeof(float)*(mp->size_mpi_buffer)),8004);
    // mp->b_send_buffer = (float*)malloc((size_mpi_buffer)*sizeof(float));
    // receive buffer
    print_CUDA_error_if_any(hipHostMalloc((void**)&(mp->h_recv_accel_buffer),sizeof(realw)*(mp->size_mpi_buffer)),8004);
    //mp->recv_buffer = (float*)malloc((mp->size_mpi_buffer)*sizeof(float));

    // non-pinned buffer
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_send_accel_buffer),mp->size_mpi_buffer*sizeof(realw)),4004);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_recv_accel_buffer),mp->size_mpi_buffer*sizeof(realw)),4004);

    // adjoint
    if (mp->simulation_type == 3) {
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_send_accel_buffer),mp->size_mpi_buffer*sizeof(realw)),4004);
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_recv_accel_buffer),mp->size_mpi_buffer*sizeof(realw)),4004);
    }
  }

  // debug
  //printf("prepare_fields_elastic_device: rank %d - mass matrix\n",mp->myrank);
  //synchronize_mpi();

  // mass matrix
  copy_todevice_realw((void**)&mp->d_rmassx,rmassx,mp->NGLOB_AB);
  copy_todevice_realw((void**)&mp->d_rmassz,rmassz,mp->NGLOB_AB);

  // anisotropy flag
  copy_todevice_int((void**)&mp->d_ispec_is_anisotropic,ispec_is_anisotropic,mp->NSPEC_AB);

  // phase elements
  mp->num_phase_ispec_elastic = *num_phase_ispec_elastic;

  copy_todevice_int((void**)&mp->d_phase_ispec_inner_elastic,phase_ispec_inner_elastic,2*mp->num_phase_ispec_elastic);

  // debug
  //synchronize_mpi();

  // anisotropy
  if (*ANISOTROPY) {
    // debug
    //printf("prepare_fields_elastic_device: rank %d - attenuation setup\n",mp->myrank);
    //synchronize_mpi();

    // Assuming NGLLX==5. Padded is then 32 (5^2+3)
    size_padded = NGLL2_PADDED * (mp->NSPEC_AB);

    // allocates memory on GPU
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c11store),size_padded*sizeof(realw)),4700);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c12store),size_padded*sizeof(realw)),4701);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c13store),size_padded*sizeof(realw)),4702);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c15store),size_padded*sizeof(realw)),4704);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c23store),size_padded*sizeof(realw)),4707);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c25store),size_padded*sizeof(realw)),4709);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c33store),size_padded*sizeof(realw)),4711);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c35store),size_padded*sizeof(realw)),4711);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c55store),size_padded*sizeof(realw)),4718);

    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c11store, NGLL2_PADDED*sizeof(realw),
                                         c11store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c12store, NGLL2_PADDED*sizeof(realw),
                                         c12store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c13store, NGLL2_PADDED*sizeof(realw),
                                         c13store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c15store, NGLL2_PADDED*sizeof(realw),
                                         c15store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c23store, NGLL2_PADDED*sizeof(realw),
                                         c23store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c25store, NGLL2_PADDED*sizeof(realw),
                                         c25store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c33store, NGLL2_PADDED*sizeof(realw),
                                         c33store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c35store, NGLL2_PADDED*sizeof(realw),
                                         c35store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c55store, NGLL2_PADDED*sizeof(realw),
                                         c55store, NGLL2*sizeof(realw), NGLL2*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
  }

  mp->ninterface_elastic = *h_ninterface_elastic;
  copy_todevice_int((void**)&mp->d_inum_interfaces_elastic,h_inum_interfaces_elastic,mp->num_interfaces_ext_mesh);

  // attenuation
  if (*ATTENUATION_VISCOELASTIC) {
    copy_todevice_realw((void**)&mp->d_A_newmark_mu,h_A_newmark_mu,NGLL2*mp->NSPEC_AB*N_SLS);
    copy_todevice_realw((void**)&mp->d_B_newmark_mu,h_B_newmark_mu,NGLL2*mp->NSPEC_AB*N_SLS);
    copy_todevice_realw((void**)&mp->d_A_newmark_kappa,h_A_newmark_kappa,NGLL2*mp->NSPEC_AB*N_SLS);
    copy_todevice_realw((void**)&mp->d_B_newmark_kappa,h_B_newmark_kappa,NGLL2*mp->NSPEC_AB*N_SLS);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_e1,mp->NSPEC_AB*sizeof(realw)*NGLL2*N_SLS),4801);
    print_CUDA_error_if_any(hipMemset(mp->d_e1,0,mp->NSPEC_AB*sizeof(realw)*NGLL2*N_SLS),4802);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_e11,mp->NSPEC_AB*sizeof(realw)*NGLL2*N_SLS),4803);
    print_CUDA_error_if_any(hipMemset(mp->d_e11,0,mp->NSPEC_AB*sizeof(realw)*NGLL2*N_SLS),4804);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_e13,mp->NSPEC_AB*sizeof(realw)*NGLL2*N_SLS),4805);
    print_CUDA_error_if_any(hipMemset(mp->d_e13,0,mp->NSPEC_AB*sizeof(realw)*NGLL2*N_SLS),4806);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_dux_dxl_old,mp->NSPEC_AB*sizeof(realw)*NGLL2),4807);
    print_CUDA_error_if_any(hipMemset(mp->d_dux_dxl_old,0,mp->NSPEC_AB*sizeof(realw)*NGLL2),4808);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_duz_dzl_old,mp->NSPEC_AB*sizeof(realw)*NGLL2),4809);
    print_CUDA_error_if_any(hipMemset(mp->d_duz_dzl_old,0,mp->NSPEC_AB*sizeof(realw)*NGLL2),4810);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_dux_dzl_plus_duz_dxl_old,mp->NSPEC_AB*sizeof(realw)*NGLL2),4811);
    print_CUDA_error_if_any(hipMemset(mp->d_dux_dzl_plus_duz_dxl_old,0,mp->NSPEC_AB*sizeof(realw)*NGLL2),4812);
  }

  // debug
  //printf("prepare_fields_elastic_device: rank %d - done\n",mp->myrank);
  //synchronize_mpi();

  GPU_ERROR_CHECKING ("prepare_fields_elastic_device");
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_elastic_adj_dev,
              PREPARE_FIELDS_ELASTIC_ADJ_DEV)(long* Mesh_pointer,
                                              int* size_f,
                                              int* APPROXIMATE_HESS_KL,
                                              int* ATTENUATION_VISCOELASTIC,
                                              int* NO_BACKWARD_RECONSTRUCTION){

  TRACE("prepare_fields_elastic_adj_dev");

  Mesh* mp = (Mesh*)(*Mesh_pointer);
  int size;

  // checks if kernel simulation
  if (mp->simulation_type != 3 ) return;

  // kernel simulations
  // debug
  //printf("prepare_fields_elastic_adj_dev: rank %d - kernel setup\n",mp->myrank);
  //synchronize_mpi();

  // backward/reconstructed wavefields
  // allocates backward/reconstructed arrays on device (GPU)
  size = *size_f;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_displ),sizeof(realw)*size),5201);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_veloc),sizeof(realw)*size),5202);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_accel),sizeof(realw)*size),5203);
  // initializes values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_b_displ,0,sizeof(realw)*size),5207);
  print_CUDA_error_if_any(hipMemset(mp->d_b_veloc,0,sizeof(realw)*size),5207);
  print_CUDA_error_if_any(hipMemset(mp->d_b_accel,0,sizeof(realw)*size),5207);

 #ifdef USE_TEXTURES_FIELDS
  {
    #ifdef USE_OLDER_CUDA4_GPU
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      const textureReference* d_b_displ_tex_ref_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_b_displ_tex_ref_ptr, HIP_SYMBOL("d_b_displ_tex")), 5204);
      print_CUDA_error_if_any(hipBindTexture(0, d_b_displ_tex_ref_ptr, mp->d_b_displ, &channelDesc, sizeof(realw)*size), 5205);
    #else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      print_CUDA_error_if_any(hipBindTexture(0, &d_b_displ_tex, mp->d_b_displ, &channelDesc, sizeof(realw)*size), 5208);
    #endif
  }
  #endif

  // anisotropic/isotropic kernels
  // debug
  //printf("prepare_fields_elastic_adj_dev: rank %d -  anisotropic/isotropic kernels\n",mp->myrank);
  //synchronize_mpi();

  // allocates kernels
  size = NGLL2 * mp->NSPEC_AB; // note: non-aligned; if align, check memcpy below and indexing
  // density kernel
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_kl),size*sizeof(realw)),5211);
  // isotropic kernels
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_mu_kl),size*sizeof(realw)),5213);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_kappa_kl),size*sizeof(realw)),5214);

  // initializes kernel values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_rho_kl,0,size*sizeof(realw)),5212);
  print_CUDA_error_if_any(hipMemset(mp->d_mu_kl,0,size*sizeof(realw)),5216);
  print_CUDA_error_if_any(hipMemset(mp->d_kappa_kl,0,size*sizeof(realw)),5217);

  // approximate hessian kernel
  if (*APPROXIMATE_HESS_KL) {
    // debug
    //printf("prepare_fields_elastic_adj_dev: rank %d - hessian kernel\n",mp->myrank);
    //synchronize_mpi();

    size = NGLL2 * mp->NSPEC_AB; // note: non-aligned; if align, check memcpy below and indexing
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_el_kl),size*sizeof(realw)),5450);
    print_CUDA_error_if_any(hipMemset(mp->d_hess_el_kl,0,size*sizeof(realw)),5451);
  }

  // attenuation
  if (*ATTENUATION_VISCOELASTIC && (! *NO_BACKWARD_RECONSTRUCTION) ) {
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_e1,mp->NSPEC_AB*sizeof(realw)*NGLL2*N_SLS),4801);
    print_CUDA_error_if_any(hipMemset(mp->d_b_e1,0,mp->NSPEC_AB*sizeof(realw)*NGLL2*N_SLS),4802);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_e11,mp->NSPEC_AB*sizeof(realw)*NGLL2*N_SLS),4803);
    print_CUDA_error_if_any(hipMemset(mp->d_b_e11,0,mp->NSPEC_AB*sizeof(realw)*NGLL2*N_SLS),4804);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_e13,mp->NSPEC_AB*sizeof(realw)*NGLL2*N_SLS),4805);
    print_CUDA_error_if_any(hipMemset(mp->d_b_e13,0,mp->NSPEC_AB*sizeof(realw)*NGLL2*N_SLS),4806);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_dux_dxl_old,mp->NSPEC_AB*sizeof(realw)*NGLL2),4807);
    print_CUDA_error_if_any(hipMemset(mp->d_b_dux_dxl_old,0,mp->NSPEC_AB*sizeof(realw)*NGLL2),4808);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_duz_dzl_old,mp->NSPEC_AB*sizeof(realw)*NGLL2),4809);
    print_CUDA_error_if_any(hipMemset(mp->d_b_duz_dzl_old,0,mp->NSPEC_AB*sizeof(realw)*NGLL2),4810);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_dux_dzl_plus_duz_dxl_old,mp->NSPEC_AB*sizeof(realw)*NGLL2),4811);
    print_CUDA_error_if_any(hipMemset(mp->d_b_dux_dzl_plus_duz_dxl_old,0,mp->NSPEC_AB*sizeof(realw)*NGLL2),4812);
  }

  // debug
  //printf("prepare_fields_elastic_adj_dev: rank %d - done\n",mp->myrank);
  //synchronize_mpi();

  GPU_ERROR_CHECKING ("prepare_fields_elastic_adj_dev");
}

/* ----------------------------------------------------------------------------------------------- */

// purely adjoint & kernel simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_sim2_or_3_const_device,
              PREPARE_SIM2_OR_3_CONST_DEVICE)(long* Mesh_pointer,
                                              int* nadj_rec_local,
                                              realw* h_source_adjoint,
                                              int* NSTEP) {

  TRACE("prepare_sim2_or_3_const_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer);

  // adjoint source arrays
  mp->nadj_rec_local = *nadj_rec_local;
  if (mp->nadj_rec_local > 0) {
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_adj_sourcearrays,(mp->nadj_rec_local)*2*NGLL2*sizeof(realw)),6003);

    copy_todevice_realw((void**)&mp->d_source_adjoint,h_source_adjoint,(*NSTEP)*(*nadj_rec_local)*NDIM);
  }

  GPU_ERROR_CHECKING ("prepare_sim2_or_3_const_device");
}

/* ----------------------------------------------------------------------------------------------- */

// PML boundary conditions

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_pml_device,
              PREPARE_PML_DEVICE)(long* Mesh_pointer,
                                  int* NSPEC_PML,
                                  int* NSPEC_PML_X,
                                  int* NSPEC_PML_Z,
                                  int* NSPEC_PML_XZ,
                                  int* h_spec_to_pml,
                                  realw* h_abs_normalized,
                                  realw* ALPHA_MAX_PML,
                                  realw* d0_max,
                                  realw* deltat,
                                  realw* h_alphax_store,
                                  realw* h_alphaz_store,
                                  realw* h_betax_store,
                                  realw* h_betaz_store,
                                  int *PML_nglob_abs_acoustic_f,
                                  int *h_PML_abs_points_acoustic){

  TRACE("prepare_PML_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer);

  if (mp->pml_boundary_conditions){
    mp->deltat = *deltat;
    mp->nspec_pml    = *NSPEC_PML;
    mp->nspec_pml_x  = *NSPEC_PML_X;
    mp->nspec_pml_z  = *NSPEC_PML_Z;
    mp->ALPHA_MAX_PML = *ALPHA_MAX_PML;
    mp->d0_max = *d0_max;

    copy_todevice_int((void**)&mp->d_spec_to_pml,h_spec_to_pml,mp->NSPEC_AB);

    // PML wavefields
    print_CUDA_error_if_any(hipMalloc((void**)&mp->PML_dpotentialdxl_old,NGLL2*mp->nspec_pml*sizeof(realw)),1301);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->PML_dpotentialdzl_old,NGLL2*mp->nspec_pml*sizeof(realw)),1302);
    // initializes
    print_CUDA_error_if_any(hipMemset(mp->PML_dpotentialdxl_old,0,sizeof(realw)*NGLL2*mp->nspec_pml),2007);
    print_CUDA_error_if_any(hipMemset(mp->PML_dpotentialdzl_old,0,sizeof(realw)*NGLL2*mp->nspec_pml),2007);

    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_potential_old,NGLL2*mp->nspec_pml*sizeof(realw)),1303);
    // initializes
    print_CUDA_error_if_any(hipMemset(mp->d_potential_old,0,sizeof(realw)*NGLL2*mp->nspec_pml),2007);

    copy_todevice_realw((void**)&mp->abscissa_norm,h_abs_normalized,NGLL2*mp->nspec_pml);

    // PML memory variables
    print_CUDA_error_if_any(hipMalloc((void**)&mp->rmemory_acoustic_dux_dx,NGLL2*mp->nspec_pml*sizeof(realw)),1290);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->rmemory_acoustic_dux_dz,NGLL2*mp->nspec_pml*sizeof(realw)),1291);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->rmemory_acoustic_dux_dx2,NGLL2*(*NSPEC_PML_XZ)*sizeof(realw)),1292);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->rmemory_acoustic_dux_dz2,NGLL2*(*NSPEC_PML_XZ)*sizeof(realw)),1292);
    // initializes
    print_CUDA_error_if_any(hipMemset(mp->rmemory_acoustic_dux_dx,0,sizeof(realw)*NGLL2*mp->nspec_pml),2007);
    print_CUDA_error_if_any(hipMemset(mp->rmemory_acoustic_dux_dz,0,sizeof(realw)*NGLL2*mp->nspec_pml),2007);
    print_CUDA_error_if_any(hipMemset(mp->rmemory_acoustic_dux_dx2,0,sizeof(realw)*NGLL2*(*NSPEC_PML_XZ)),2007);
    print_CUDA_error_if_any(hipMemset(mp->rmemory_acoustic_dux_dz2,0,sizeof(realw)*NGLL2*(*NSPEC_PML_XZ)),2007);

    print_CUDA_error_if_any(hipMalloc((void**)&mp->rmemory_pot_acoustic,NGLL2*mp->nspec_pml*sizeof(realw)),1293);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->rmemory_pot_acoustic2,NGLL2*(*NSPEC_PML_XZ)*sizeof(realw)),1294);
    // initializes
    print_CUDA_error_if_any(hipMemset(mp->rmemory_pot_acoustic,0,sizeof(realw)*NGLL2*(mp->nspec_pml)),2007);
    print_CUDA_error_if_any(hipMemset(mp->rmemory_pot_acoustic2,0,sizeof(realw)*NGLL2*(*NSPEC_PML_XZ)),2007);

    // PML coefficients
    copy_todevice_realw((void**)&mp->alphax_store,h_alphax_store,NGLL2*(*NSPEC_PML_XZ));
    copy_todevice_realw((void**)&mp->alphaz_store,h_alphaz_store,NGLL2*(*NSPEC_PML_XZ));
    copy_todevice_realw((void**)&mp->betax_store,h_betax_store,NGLL2*(*NSPEC_PML_XZ));
    copy_todevice_realw((void**)&mp->betaz_store,h_betaz_store,NGLL2*(*NSPEC_PML_XZ));

    // acoustic boundary
    mp->pml_nglob_abs_acoustic = *PML_nglob_abs_acoustic_f;
    copy_todevice_int((void**)&mp->d_pml_abs_points_acoustic,h_PML_abs_points_acoustic,mp->pml_nglob_abs_acoustic);
  }

  GPU_ERROR_CHECKING ("prepare_PML_device");
}

/* ----------------------------------------------------------------------------------------------- */

// Stacey boundary conditions

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_stacey_device,
              PREPARE_STACEY_DEVICE)(long* Mesh_pointer,
                                     int* ACOUSTIC_SIMULATION,
                                     int* ELASTIC_SIMULATION,
                                     realw* rho_vp, realw* rho_vs,
                                     int* h_nspec_bottom,
                                     int* h_nspec_left,
                                     int* h_nspec_right,
                                     int* h_nspec_top,
                                     int* h_abs_boundary_ispec, int* h_abs_boundary_ij,
                                     realw* h_abs_boundary_normal,
                                     realw* h_abs_boundary_jacobian1Dw,
                                     int* h_num_abs_boundary_faces,
                                     int* h_edge_abs,
                                     int* h_ib_bottom,
                                     int* h_ib_left,
                                     int* h_ib_right,
                                     int* h_ib_top){

  TRACE("prepare_Stacey_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer);

  // checks consistency
  if (! mp->stacey_absorbing_conditions)
    exit_on_error("Stacey absorbing condition flag inconsistent with prepare_stacey_device() call");

  // Stacey absorbing boundaries
  mp->d_num_abs_boundary_faces = *h_num_abs_boundary_faces;

  // Stacey absorbing conditions
  if (mp->stacey_absorbing_conditions && mp->d_num_abs_boundary_faces > 0) {
    mp->d_nspec_left = *h_nspec_left;
    mp->d_nspec_right = *h_nspec_right;
    mp->d_nspec_top = *h_nspec_top;
    mp->d_nspec_bottom = *h_nspec_bottom;

    // debug
    //printf("debug: stacey prepare faces %d %d\n",mp->stacey_absorbing_conditions,mp->d_num_abs_boundary_faces);
    //printf("debug: stacey prepare %d %d %d %d\n",mp->d_nspec_left,mp->d_nspec_right,mp->d_nspec_top,mp->d_nspec_bottom);

    copy_todevice_int((void**)&mp->d_abs_boundary_ispec,h_abs_boundary_ispec,mp->d_num_abs_boundary_faces);
    copy_todevice_int((void**)&mp->d_abs_boundary_ijk,h_abs_boundary_ij,
                      2*NGLLX*(mp->d_num_abs_boundary_faces));
    copy_todevice_realw((void**)&mp->d_abs_boundary_normal,h_abs_boundary_normal,
                        NDIM*NGLLX*(mp->d_num_abs_boundary_faces));
    copy_todevice_realw((void**)&mp->d_abs_boundary_jacobian2Dw,h_abs_boundary_jacobian1Dw,
                        NGLLX*(mp->d_num_abs_boundary_faces));

    copy_todevice_int((void**)&mp->d_edge_abs,h_edge_abs,(mp->d_num_abs_boundary_faces));
    copy_todevice_int((void**)&mp->d_ib_left,h_ib_left,(mp->d_num_abs_boundary_faces));
    copy_todevice_int((void**)&mp->d_ib_right,h_ib_right,(mp->d_num_abs_boundary_faces));
    copy_todevice_int((void**)&mp->d_ib_top,h_ib_top,(mp->d_num_abs_boundary_faces));
    copy_todevice_int((void**)&mp->d_ib_bottom,h_ib_bottom,(mp->d_num_abs_boundary_faces));

    // elastic domains
    if (*ELASTIC_SIMULATION){
      // debug
      //printf("prepare_fields_elastic_device: rank %d - absorbing boundary setup\n",mp->myrank);

      // non-padded arrays
      // rho_vp, rho_vs non-padded; they are needed for stacey boundary condition
      copy_todevice_realw((void**)&mp->d_rho_vp,rho_vp,NGLL2*mp->NSPEC_AB);
      copy_todevice_realw((void**)&mp->d_rho_vs,rho_vs,NGLL2*mp->NSPEC_AB);

      // absorb_field array used for file i/o
      if (mp->simulation_type == 3 || ( mp->simulation_type == 1 && mp->save_forward )){
        print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_elastic_left,NDIM*mp->d_nspec_left*sizeof(realw)*NGLLX),2201);
        print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_elastic_right,NDIM*mp->d_nspec_right*sizeof(realw)*NGLLX),2202);
        print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_elastic_top,NDIM*mp->d_nspec_top*sizeof(realw)*NGLLX),2203);
        print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_elastic_bottom,NDIM*mp->d_nspec_bottom*sizeof(realw)*NGLLX),2204);
        // initializes values to zero
        print_CUDA_error_if_any(hipMemset(mp->d_b_absorb_elastic_left,0,NDIM*mp->d_nspec_left*sizeof(realw)*NGLLX),2221);
        print_CUDA_error_if_any(hipMemset(mp->d_b_absorb_elastic_right,0,NDIM*mp->d_nspec_right*sizeof(realw)*NGLLX),2222);
        print_CUDA_error_if_any(hipMemset(mp->d_b_absorb_elastic_top,0,NDIM*mp->d_nspec_top*sizeof(realw)*NGLLX),2223);
        print_CUDA_error_if_any(hipMemset(mp->d_b_absorb_elastic_bottom,0,NDIM*mp->d_nspec_bottom*sizeof(realw)*NGLLX),2224);
      }
    } // ELASTIC_SIMULATION

    // acoustic domains
    if (*ACOUSTIC_SIMULATION){
      // absorb_field array used for file i/o
      if (mp->simulation_type == 3 || ( mp->simulation_type == 1 && mp->save_forward )){
        print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_potential_left,mp->d_nspec_left*sizeof(realw)*NGLLX),2211);
        print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_potential_right,mp->d_nspec_right*sizeof(realw)*NGLLX),2212);
        print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_potential_top,mp->d_nspec_top*sizeof(realw)*NGLLX),2213);
        print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_potential_bottom,mp->d_nspec_bottom*sizeof(realw)*NGLLX),2214);
        // initializes values to zero
        print_CUDA_error_if_any(hipMemset(mp->d_b_absorb_potential_left,0,mp->d_nspec_left*sizeof(realw)*NGLLX),2221);
        print_CUDA_error_if_any(hipMemset(mp->d_b_absorb_potential_right,0,mp->d_nspec_right*sizeof(realw)*NGLLX),2222);
        print_CUDA_error_if_any(hipMemset(mp->d_b_absorb_potential_top,0,mp->d_nspec_top*sizeof(realw)*NGLLX),2223);
        print_CUDA_error_if_any(hipMemset(mp->d_b_absorb_potential_bottom,0,mp->d_nspec_bottom*sizeof(realw)*NGLLX),2224);
      }
    } // ACOUSTIC_SIMULATION
  }

  GPU_ERROR_CHECKING ("prepare_Stacey_device");
}


/* ----------------------------------------------------------------------------------------------- */

// For moving sources

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(prepare_moving_sources_cuda,
              PREPARE_MOVING_SOURCES_CUDA)(long* Mesh_pointer,
                                           int* h_nsources_local_f_moving,
                                           int* NSOURCES,
                                           realw* h_sourcearrays_moving,
                                           int* h_ispec_selected_source_moving,
                                           int* NSTEP,
                                           realw* h_source_time_function_moving) {

  TRACE("prepare_moving_sources_cuda");
  // Pointers received are int* (not int** or int*** etc which would allow
  // using array[i][j] etc ) we use INDEX2, INDEX3 etc defined in mesh_constants_cuda
  // to access to the elements. Do not forget printf("%f", float); to print a
  // float !!
  // Example of partial loop over 5 dimensional array size:(3,1,3,3,?) from real*
  // for (int i = 0; i < 3; i++) {
  //   for (int n = 0; n < 3; n++) {
  //     printf("Example %d, 1, %d: ", i + 1, n + 1);
  //     printf("%f\n", array[INDEX5(3,1,3,3, i, 0, n,0,0)]);
  //   }
  //   printf("\n");
  // }
  // Beware: h_sourcearrays_moving[INDEX5(3,5,5,1, 0, 1, 1,1,i)]
  //
  // Example of partial loop over 2d array from real*
  // for (int i = 0; i < 3; i++) {
  //   for (int n = 0; n < 3; n++) {
  //     printf("%f\n", array[INDEX2(3, i, n)]);
  //   }
  //   printf("\n");
  // }
  //
  // printf("nsources:\n");
  // for (int i = 0; i < NSTEP_int; i++) {
  //     printf("%d\n", h_nsources_local_f_moving[i]);
  // }
  int NSTEP_int = *NSTEP;
  int nsources = *NSOURCES;

  Mesh* mp = (Mesh*)(*Mesh_pointer); // get mesh pointer out of fortran integer container

  // printf("This also works:\n");
  // int* p1 = h_ispec_selected_source_moving;
  // for (int i = 0; i < nsources*NSTEP_int; i++) {
  //     printf("%d\n", *p1);
  //     p1++;
  // }
  //
  // printf("This as well:\n");
  // int** p = &h_ispec_selected_source_moving;
  // for (int i_source = 0; i_source < nsources; ++i_source) {
  //     for (int it = 0; it < NSTEP_int; ++it) {
  //         printf("%d %d %d \n",i_source,it,*(*(p + i_source) + it));
  //     }
  //     printf("");
  // }

  copy_todevice_realw((void**)&mp->d_sourcearrays_moving,h_sourcearrays_moving,NDIM*NGLL2*nsources*NSTEP_int);
  copy_todevice_int((void**)&mp->d_ispec_selected_source_moving,h_ispec_selected_source_moving,nsources*NSTEP_int);
  // When the source is moving we don't know where it is going: all the slices
  // need to know the source_time_function
  // If the source is not moving only the slice containing the source knows the source_time_function
  copy_todevice_realw((void**)&mp->d_source_time_function_moving,h_source_time_function_moving,nsources*NSTEP_int);

  GPU_ERROR_CHECKING ("prepare_moving_sources_cuda");
}

/* ----------------------------------------------------------------------------------------------- */

// Old function for moving sources
// AB AB Let it here please it may be useful
// It is used when compute_add_sources_acoustic_GPU_moving_sources_old is used
// instead of compute_add_sources_acoustic_GPU_moving_sources in compute_gpu_acoustic.f90
// Read the comments there
//
//
//extern "C"
//void FC_FUNC_(recompute_source_position_cuda,
//              RECOMPUTE_SOURCE_POSITION_CUDA)(long* Mesh_pointer,
//                                        int* nsources_local_f,
//                                        realw* h_sourcearrays,
//                                        int* h_ispec_selected_source) {
//
//  TRACE("recompute_source_position_cuda");
//
//  Mesh* mp = (Mesh*)(*Mesh_pointer); // get mesh pointer out of fortran integer container
//
//  // sources
//  mp->nsources_local = *nsources_local_f;
//  if (mp->nsources_local > 0){
//    copy_todevice_realw((void**)&mp->d_sourcearrays,h_sourcearrays,NDIM*NGLL2*mp->nsources_local);
//    copy_todevice_int((void**)&mp->d_ispec_selected_source,h_ispec_selected_source,mp->nsources_local);
//  }
//
//  GPU_ERROR_CHECKING ("recompute_source_position_cuda");
//}


/* ----------------------------------------------------------------------------------------------- */

// cleanup

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_cleanup_device,
              PREPARE_CLEANUP_DEVICE)(long* Mesh_pointer,
                                      int* ACOUSTIC_SIMULATION,
                                      int* ELASTIC_SIMULATION,
                                      int* ANISOTROPY,
                                      int* APPROXIMATE_HESS_KL,
                                      int* ATTENUATION_VISCOACOUSTIC,
                                      int* ATTENUATION_VISCOELASTIC,
                                      int* NO_BACKWARD_RECONSTRUCTION,
                                      realw * h_no_backward_acoustic_buffer) {


TRACE("prepare_cleanup_device");

  // frees allocated memory arrays
  Mesh* mp = (Mesh*)(*Mesh_pointer);

#ifdef USE_TEXTURES_CONSTANTS
  hipUnbindTexture(d_hprime_xx_tex);
  hipUnbindTexture(d_wxgll_xx_tex);
#endif

  // frees memory on GPU
  // mesh
  hipFree(mp->d_xix);
  hipFree(mp->d_xiz);
  hipFree(mp->d_gammax);
  hipFree(mp->d_gammaz);
  hipFree(mp->d_muv);
  hipFree(mp->d_kappav);

  // Stacey absorbing boundaries
  if (mp->stacey_absorbing_conditions && mp->d_num_abs_boundary_faces > 0) {
    hipFree(mp->d_abs_boundary_ispec);
    hipFree(mp->d_abs_boundary_ijk);
    hipFree(mp->d_abs_boundary_normal);
    hipFree(mp->d_abs_boundary_jacobian2Dw);
    hipFree(mp->d_edge_abs);
    hipFree(mp->d_ib_left);
    hipFree(mp->d_ib_right);
    hipFree(mp->d_ib_top);
    hipFree(mp->d_ib_bottom);
  }

  // interfaces
  if (mp->num_interfaces_ext_mesh > 0) {
    hipFree(mp->d_nibool_interfaces_ext_mesh);
    hipFree(mp->d_ibool_interfaces_ext_mesh);
  }

  // global indexing
  hipFree(mp->d_ispec_is_inner);
  hipFree(mp->d_ibool);

  hipFree(mp->d_ispec_is_acoustic);
  hipFree(mp->d_ispec_is_elastic);

  // sources
  if (mp->nsources_local > 0){
    hipFree(mp->d_sourcearrays);
    hipFree(mp->d_source_time_function);
    hipFree(mp->d_ispec_selected_source);
  }

  if (mp->source_is_moving) {
    hipFree(mp->d_sourcearrays_moving);
    hipFree(mp->d_ispec_selected_source_moving);
    hipFree(mp->d_source_time_function_moving);
  }

  // receivers
  if (mp->nrec_local > 0) {
    // clear pointer look-up table
    for(int i_sig = 0; i_sig < mp->h_NSIGTYPE; i_sig++) {
      if (mp->d_seismograms[i_sig] != NULL) hipFree(mp->d_seismograms[i_sig]);
      if (mp->h_seismograms[i_sig] != NULL) hipHostFree(mp->h_seismograms[i_sig]);
    }
    free(mp->d_seismograms);
    free(mp->h_seismograms);

    hipFree(mp->d_cosrot),hipFree(mp->d_sinrot);
    hipFree(mp->d_gammar_store_loc);
    hipFree(mp->d_xir_store_loc);
    hipFree(mp->d_ispec_selected_rec_loc);
  }

  // PML
  if (mp->pml_boundary_conditions){
    hipFree(mp->d_spec_to_pml);
    hipFree(mp->PML_dpotentialdxl_old);
    hipFree(mp->PML_dpotentialdzl_old);
    hipFree(mp->d_potential_old);
    hipFree(mp->abscissa_norm);
    hipFree(mp->rmemory_acoustic_dux_dx);
    hipFree(mp->rmemory_acoustic_dux_dz);
    hipFree(mp->rmemory_acoustic_dux_dx2);
    hipFree(mp->rmemory_acoustic_dux_dz2);
    hipFree(mp->rmemory_pot_acoustic);
    hipFree(mp->rmemory_pot_acoustic2);
    hipFree(mp->alphax_store);
    hipFree(mp->alphaz_store);
    hipFree(mp->betax_store);
    hipFree(mp->betaz_store);
    hipFree(mp->d_pml_abs_points_acoustic);
  }

  // ACOUSTIC arrays
  if (*ACOUSTIC_SIMULATION) {
    hipFree(mp->d_potential_acoustic);
    hipFree(mp->d_potential_dot_acoustic);
    hipFree(mp->d_potential_dot_dot_acoustic);
    if (mp->size_mpi_buffer_potential > 0 ) hipFree(mp->d_send_potential_dot_dot_buffer);
    hipFree(mp->d_rmass_acoustic);
    hipFree(mp->d_rhostore);
    hipFree(mp->d_kappastore);
    hipFree(mp->d_phase_ispec_inner_acoustic);
    hipFree(mp->d_inum_interfaces_acoustic);

    if (*NO_BACKWARD_RECONSTRUCTION){
      hipFree(mp->d_potential_acoustic_buffer);
      hipHostUnregister(h_no_backward_acoustic_buffer);
      hipEventDestroy(mp->transfer_is_complete1);
      hipEventDestroy(mp->transfer_is_complete2);

    }
    if (mp->simulation_type == 3) {
      hipFree(mp->d_b_potential_acoustic);
      if (! *NO_BACKWARD_RECONSTRUCTION){
        hipFree(mp->d_b_potential_dot_acoustic);
        hipFree(mp->d_b_potential_dot_dot_acoustic);
      }
      hipFree(mp->d_rho_ac_kl);
      hipFree(mp->d_kappa_ac_kl);
      if (*APPROXIMATE_HESS_KL) hipFree(mp->d_hess_ac_kl);
      if (mp->size_mpi_buffer_potential > 0 && ! *NO_BACKWARD_RECONSTRUCTION) hipFree(mp->d_b_send_potential_dot_dot_buffer);
      if (*ATTENUATION_VISCOACOUSTIC && ! *NO_BACKWARD_RECONSTRUCTION) {
        hipFree(mp->d_b_sum_forces_old);
        hipFree(mp->d_b_e1_acous);
      }
    }

    if (mp->stacey_absorbing_conditions && mp->d_num_abs_boundary_faces > 0){
      if (mp->simulation_type == 3 || ( mp->simulation_type == 1 && mp->save_forward )){
        hipFree(mp->d_b_absorb_potential_bottom);
        hipFree(mp->d_b_absorb_potential_left);
        hipFree(mp->d_b_absorb_potential_right);
        hipFree(mp->d_b_absorb_potential_top);
      }
    }
    if (*ATTENUATION_VISCOACOUSTIC){
      hipFree(mp->d_e1_acous);
      hipFree(mp->d_A_newmark_acous);
      hipFree(mp->d_B_newmark_acous);
      hipFree(mp->d_sum_forces_old);
    }

  } // ACOUSTIC_SIMULATION

  // ELASTIC arrays
  if (*ELASTIC_SIMULATION) {
    hipFree(mp->d_displ);
    hipFree(mp->d_veloc);
    hipFree(mp->d_accel);

    if (mp->size_mpi_buffer > 0){
      hipFree(mp->d_send_accel_buffer);
      hipFree(mp->d_recv_accel_buffer);
      hipHostFree(mp->h_send_accel_buffer);
      hipHostFree(mp->h_recv_accel_buffer);
      if (mp->simulation_type == 3){
        hipFree(mp->d_b_send_accel_buffer);
        hipFree(mp->d_b_recv_accel_buffer);
      }
    }

    hipFree(mp->d_rmassx);
    hipFree(mp->d_rmassz);

    hipFree(mp->d_phase_ispec_inner_elastic);
    hipFree(mp->d_ispec_is_anisotropic);
    hipFree(mp->d_inum_interfaces_elastic);

    if (mp->stacey_absorbing_conditions && mp->d_num_abs_boundary_faces > 0){
      hipFree(mp->d_rho_vp);
      hipFree(mp->d_rho_vs);
      if (mp->simulation_type == 3 || ( mp->simulation_type == 1 && mp->save_forward )){
        hipFree(mp->d_b_absorb_elastic_bottom);
        hipFree(mp->d_b_absorb_elastic_left);
        hipFree(mp->d_b_absorb_elastic_right);
        hipFree(mp->d_b_absorb_elastic_top);
      }
    }

    if (mp->simulation_type == 3) {
      hipFree(mp->d_b_displ);
      hipFree(mp->d_b_veloc);
      hipFree(mp->d_b_accel);
      hipFree(mp->d_rho_kl);
      hipFree(mp->d_mu_kl);
      hipFree(mp->d_kappa_kl);
      if (*APPROXIMATE_HESS_KL ) hipFree(mp->d_hess_el_kl);
      if (*ATTENUATION_VISCOELASTIC && ! *NO_BACKWARD_RECONSTRUCTION) {
        hipFree(mp->d_b_e1);
        hipFree(mp->d_b_e11);
        hipFree(mp->d_b_e13);
        hipFree(mp->d_b_dux_dxl_old);
        hipFree(mp->d_b_duz_dzl_old);
        hipFree(mp->d_b_dux_dzl_plus_duz_dxl_old);
      }
    }

    if (*ANISOTROPY) {
      hipFree(mp->d_c11store);
      hipFree(mp->d_c12store);
      hipFree(mp->d_c13store);
      hipFree(mp->d_c15store);
      hipFree(mp->d_c23store);
      hipFree(mp->d_c25store);
      hipFree(mp->d_c33store);
      hipFree(mp->d_c35store);
      hipFree(mp->d_c55store);
    }

    if (*ATTENUATION_VISCOELASTIC) {
      hipFree(mp->d_A_newmark_mu);
      hipFree(mp->d_B_newmark_mu);
      hipFree(mp->d_A_newmark_kappa);
      hipFree(mp->d_B_newmark_kappa);
      hipFree(mp->d_e1);
      hipFree(mp->d_e11);
      hipFree(mp->d_e13);
      hipFree(mp->d_dux_dxl_old);
      hipFree(mp->d_duz_dzl_old);
      hipFree(mp->d_dux_dzl_plus_duz_dxl_old);
    }

  } // ELASTIC_SIMULATION

  // purely adjoint & kernel array
  if (mp->simulation_type == 3) {
    if (mp->nadj_rec_local > 0) {
      hipFree(mp->d_adj_sourcearrays);
    }
  }

  // mesh pointer - not needed anymore
  free(mp);
}
