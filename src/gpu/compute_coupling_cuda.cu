/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                              CNRS, France
!                       and Princeton University, USA
!                 (there are currently many more authors!)
!                           (c) October 2017
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/

#include "mesh_constants_cuda.h"



/* ----------------------------------------------------------------------------------------------- */

// ACOUSTIC - ELASTIC coupling

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_coupling_ac_el_cuda,
              COMPUTE_COUPLING_AC_EL_CUDA)(long* Mesh_pointer,
                                           int* iphasef,
                                           int* num_coupling_ac_el_facesf) {
  TRACE("compute_coupling_ac_el_cuda");
  //double start_time = get_time();

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  int iphase            = *iphasef;

  // only adds this contribution for first pass
  if (iphase != 1) return;

  int num_coupling_ac_el_faces  = *num_coupling_ac_el_facesf;

  // way 1: exact blocksize to match NGLLSQUARE
  int blocksize = NGLLX;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(num_coupling_ac_el_faces,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // launches GPU kernel
  compute_coupling_acoustic_el_kernel<<<grid,threads>>>(mp->d_displ,
                                                       mp->d_potential_dot_dot_acoustic,
                                                       num_coupling_ac_el_faces,
                                                       mp->d_coupling_ac_el_ispec,
                                                       mp->d_coupling_ac_el_ijk,
                                                       mp->d_coupling_ac_el_normal,
                                                       mp->d_coupling_ac_el_jacobian2Dw,
                                                       mp->d_ibool);

  //  adjoint simulations
  if (mp->simulation_type == 3) {
    compute_coupling_acoustic_el_kernel<<<grid,threads>>>(mp->d_b_displ,
                                                          mp->d_b_potential_dot_dot_acoustic,
                                                          num_coupling_ac_el_faces,
                                                          mp->d_coupling_ac_el_ispec,
                                                          mp->d_coupling_ac_el_ijk,
                                                          mp->d_coupling_ac_el_normal,
                                                          mp->d_coupling_ac_el_jacobian2Dw,
                                                          mp->d_ibool);
  }


  //double end_time = get_time();
  //printf("Elapsed time: %e\n",end_time-start_time);

  GPU_ERROR_CHECKING ("compute_coupling_acoustic_el_kernel");
}


/* ----------------------------------------------------------------------------------------------- */

// ELASTIC - ACOUSTIC coupling

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(compute_coupling_el_ac_cuda,
              COMPUTE_COUPLING_EL_AC_CUDA)(long* Mesh_pointer,
                                           int* iphasef,
                                           int* num_coupling_ac_el_facesf) {
  TRACE("compute_coupling_el_ac_cuda");
  //double start_time = get_time();

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  int iphase            = *iphasef;

  // only adds this contribution for first pass
  if (iphase != 1) return;

  int num_coupling_ac_el_faces  = *num_coupling_ac_el_facesf;

  // way 1: exact blocksize to match NGLLX
  int blocksize = NGLLX;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(num_coupling_ac_el_faces,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // launches GPU kernel
  compute_coupling_elastic_ac_kernel<<<grid,threads>>>(mp->d_potential_dot_dot_acoustic,
                                                       mp->d_accel,
                                                       num_coupling_ac_el_faces,
                                                       mp->d_coupling_ac_el_ispec,
                                                       mp->d_coupling_ac_el_ijk,
                                                       mp->d_coupling_ac_el_normal,
                                                       mp->d_coupling_ac_el_jacobian2Dw,
                                                       mp->d_ibool);

  //  adjoint simulations
  if (mp->simulation_type == 3) {
    compute_coupling_elastic_ac_kernel<<<grid,threads>>>(mp->d_b_potential_dot_dot_acoustic,
                                                         mp->d_b_accel,
                                                         num_coupling_ac_el_faces,
                                                         mp->d_coupling_ac_el_ispec,
                                                         mp->d_coupling_ac_el_ijk,
                                                         mp->d_coupling_ac_el_normal,
                                                         mp->d_coupling_ac_el_jacobian2Dw,
                                                         mp->d_ibool);
  }

  //double end_time = get_time();
  //printf("Elapsed time: %e\n",end_time-start_time);

  GPU_ERROR_CHECKING ("compute_coupling_el_ac_cuda");
}
