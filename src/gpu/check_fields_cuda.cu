#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                              CNRS, France
!                       and Princeton University, USA
!                 (there are currently many more authors!)
!                           (c) October 2017
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/

#include "mesh_constants_cuda.h"

/* ----------------------------------------------------------------------------------------------- */

// Helper functions

/* ----------------------------------------------------------------------------------------------- */

void cudaMemoryTest(int posId)
{
    // call this function at different places to locate a bug
    const unsigned int N = 1048576;
    const unsigned int bytes = N * sizeof(int);
    int *h_a = (int*)malloc(bytes);
    int *d_a;
    cudaSafeCall(hipMalloc((int**)&d_a, bytes), posId);
    memset(h_a, 0, bytes);
    cudaSafeCall(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice), posId);
    cudaSafeCall(hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost), posId);
}

/* ----------------------------------------------------------------------------------------------- */

/*
 __global__ void check_field(int* ibool,int* nibool, int max_nibool,int num_interfaces_ext_mesh)
 {


 int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;
 int ientry,iglob;

 for(int iinterface=0; iinterface < num_interfaces_ext_mesh; iinterface++) {
 if (id<nibool[iinterface]) {

 // entry in interface array
 ientry = id + max_nibool*iinterface;
 // global index in wavefield
 iglob = ibool[ientry] - 1;

 cuPrintf("valeurs de iglob %d, de l'indice d'entree %d,du nombre de points dans l'interface %d : %d\n", iglob,ientry,iinterface,nibool[iinterface]);
 }}
 }*/


/* ----------------------------------------------------------------------------------------------- */

// GPU device memory functions

/* ----------------------------------------------------------------------------------------------- */

void get_free_memory(double* free_db, double* used_db, double* total_db) {

  TRACE("get_free_memory");

  // gets memory usage in byte
  size_t free_byte ;
  size_t total_byte ;
  hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
  if (hipSuccess != cuda_status) {
    printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
    exit(EXIT_FAILURE);
  }

  *free_db = (double)free_byte ;
  *total_db = (double)total_byte ;
  *used_db = *total_db - *free_db ;
  return;
}

/* ----------------------------------------------------------------------------------------------- */

// Saves GPU memory usage to file
void output_free_memory(int myrank,char* info_str) {

  TRACE("output_free_memory");

  FILE* fp;
  char filename[BUFSIZ];
  double free_db,used_db,total_db;
  int do_output_info;

  // by default, only main process outputs device infos to avoid file cluttering
  do_output_info = 0;
  if (myrank == 0) {
    do_output_info = 1;
    sprintf(filename,OUTPUT_FILES_PATH"/gpu_device_mem_usage.txt");
  }
  // debugging
  if (DEBUG) {
    do_output_info = 1;
    sprintf(filename,OUTPUT_FILES_PATH"/gpu_device_mem_usage_proc_%06d.txt",myrank);
  }

  // outputs to file
  if (do_output_info) {

    // gets memory usage
    get_free_memory(&free_db,&used_db,&total_db);

    // file output
    fp = fopen(filename,"a+");
    if (fp != NULL) {
      fprintf(fp,"%d: @%s GPU memory usage: used = %f MB, free = %f MB, total = %f MB\n", myrank, info_str,
              used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
      fclose(fp);
    }
  }
}

/* ----------------------------------------------------------------------------------------------- */

// Fortran-callable version of above method
extern "C"
void FC_FUNC_(output_free_device_memory,
              OUTPUT_FREE_DEVICE_MEMORY)(int* myrank_f) {
  TRACE("output_free_device_memory");

  char info[64];
  int myrank = *myrank_f;

  sprintf(info,"f %d:",myrank);
  output_free_memory(myrank,info);
}


/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(get_free_device_memory,
              get_FREE_DEVICE_MEMORY)(realw* free, realw* used, realw* total) {
  TRACE("get_free_device_memory");

  double free_db,used_db,total_db;

  get_free_memory(&free_db,&used_db,&total_db);

  // converts to MB
  *free = (realw) free_db/1024.0/1024.0;
  *used = (realw) used_db/1024.0/1024.0;
  *total = (realw) total_db/1024.0/1024.0;
  return;
}



/* ----------------------------------------------------------------------------------------------- */

// Auxiliary functions

/* ----------------------------------------------------------------------------------------------- */

/*
__global__ void memset_to_realw_kernel(realw* array, int size, realw value){

  unsigned int tid = threadIdx.x;
  unsigned int bx = blockIdx.y*gridDim.x+blockIdx.x;
  unsigned int i = tid + bx*blockDim.x;

  if (i < size) {
    array[i] = *value;
  }
}
*/

/* ----------------------------------------------------------------------------------------------- */

realw get_device_array_maximum_value(realw* array, int size){

// get maximum of array on GPU by copying over to CPU and handle it there

  realw max = 0.0f;

  // checks if anything to do
  if (size > 0) {
    realw* h_array;

    // explicitly wait for cuda kernels to finish
    // (hipMemcpy implicitly synchronizes all other cuda operations)
    synchronize_cuda();

    h_array = (realw*)calloc(size,sizeof(realw));
    print_CUDA_error_if_any(hipMemcpy(h_array,array,sizeof(realw)*size,hipMemcpyDeviceToHost),33001);

    // finds maximum value in array
    max = h_array[0];
    for( int i=1; i < size; i++){
      if (abs(h_array[i]) > max ) max = abs(h_array[i]);
    }
    free(h_array);
  }
  return max;
}



/* ----------------------------------------------------------------------------------------------- */

// ACOUSTIC simulations

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(get_norm_acoustic_from_device,
              GET_NORM_ACOUSTIC_FROM_DEVICE)(realw* norm,long* Mesh_pointer,const int* FORWARD_OR_ADJOINT) {

  TRACE("get_norm_acoustic_from_device");
  //double start_time = get_time();

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  realw max1 = 0.0;
  //realw max2 = 0.0;
  //realw max3 = 0.0;
  realw *d_max1;
  //realw *d_max2;
  //realw *d_max3;

  //initializes
  *norm = 0.0f;

  // way 2 b: timing Elapsed time: 1.236916e-03
  // launch simple reduction kernel
  int blocksize = BLOCKSIZE_TRANSFER;

  int size = mp->NGLOB_AB;
  int size_padded = ((int)ceil(((double)size)/((double)blocksize)))*blocksize;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // on host (allocates & initializes to zero)
  realw* h_max1 = (realw*) calloc(num_blocks_x*num_blocks_y,sizeof(realw));
  //realw* h_max2 = (realw*) calloc(num_blocks_x*num_blocks_y,sizeof(realw));
  //realw* h_max3 = (realw*) calloc(num_blocks_x*num_blocks_y,sizeof(realw));

  // allocates memory on device
  print_CUDA_error_if_any(hipMalloc((void**)&d_max1,num_blocks_x*num_blocks_y*sizeof(realw)),78001);
  // initializes values to zero
  print_CUDA_error_if_any(hipMemset(d_max1,0,num_blocks_x*num_blocks_y*sizeof(realw)),77002);

  //print_CUDA_error_if_any(hipMalloc((void**)&d_max2,num_blocks_x*num_blocks_y*sizeof(realw)),78001);
  //print_CUDA_error_if_any(hipMemset(d_max2,0,num_blocks_x*num_blocks_y*sizeof(realw)),77002);

  //print_CUDA_error_if_any(hipMalloc((void**)&d_max3,num_blocks_x*num_blocks_y*sizeof(realw)),78001);
  //print_CUDA_error_if_any(hipMemset(d_max3,0,num_blocks_x*num_blocks_y*sizeof(realw)),77002);

  if (*FORWARD_OR_ADJOINT == 1) {
    get_maximum_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_potential_acoustic,size,d_max1);
    //get_maximum_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_potential_dot_acoustic,size,d_max2);
    //get_maximum_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_potential_dot_dot_acoustic,size,d_max3);
  }else if (*FORWARD_OR_ADJOINT == 3) {
    get_maximum_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_potential_acoustic,size,d_max1);
    //get_maximum_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_potential_dot_acoustic,size,d_max2);
    //get_maximum_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_potential_dot_dot_acoustic,size,d_ma3);
  }

  GPU_ERROR_CHECKING ("get_maximum_kernel");

  // synchronizes
  //synchronize_cuda();
  // explicitly waits for stream to finish
  // (hipMemcpy implicitly synchronizes all other cuda operations)
  hipStreamSynchronize(mp->compute_stream);

  print_CUDA_error_if_any(hipMemcpy(h_max1,d_max1,num_blocks_x*num_blocks_y*sizeof(realw),
                                     hipMemcpyDeviceToHost),222);

  //print_CUDA_error_if_any(hipMemcpy(h_max2,d_max2,num_blocks_x*num_blocks_y*sizeof(realw),
  //                                   hipMemcpyDeviceToHost),222);

  //print_CUDA_error_if_any(hipMemcpy(h_max3,d_max3,num_blocks_x*num_blocks_y*sizeof(realw),
  //                                   hipMemcpyDeviceToHost),222);

  // determines max for all blocks
  max1 = h_max1[0];
  for(int i=1;i<num_blocks_x*num_blocks_y;i++) {
    if (max1 < h_max1[i]) max1 = h_max1[i];
  }
  /*
  max2 = h_max2[0];
  for(int i=1;i<num_blocks_x*num_blocks_y;i++) {
    if (max2 < h_max2[i]) max2 = h_max2[i];
  }
  */
  /*
  max3 = h_max3[0];
  for(int i=1;i<num_blocks_x*num_blocks_y;i++) {
    if (max3 < h_max3[i]) max3 = h_max3[i];
  }
  */
  hipFree(d_max1);
  free(h_max1);
  //hipFree(d_max2);
  //free(h_max2);
  //hipFree(d_max3);
  //free(h_max3);

  // return result
  //norm[0] = max1;
  //norm[1] = max2;
  //norm[2] = max3;
  // only potential_acoustic for now:
  *norm = max1;

  //double end_time = get_time();
  //printf("Elapsed time: %e\n",end_time-start_time);

  GPU_ERROR_CHECKING ("get_norm_acoustic_from_device");
}

/* ----------------------------------------------------------------------------------------------- */

// ELASTIC simulations

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(get_norm_elastic_from_device,
              GET_NORM_ELASTIC_FROM_DEVICE)(realw* norm,long* Mesh_pointer,const int* FORWARD_OR_ADJOINT) {

  TRACE("\tget_norm_elastic_from_device");
  //double start_time = get_time();

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  realw max1; //,max2,max3;
  realw res1; //,res2,res3;
  realw *d_max1;
  //realw *d_max2;
  //realw *d_max3;

  //initializes
  *norm = 0.0f;

  // launch simple reduction kernel
  int blocksize = BLOCKSIZE_TRANSFER;

  int size = mp->NGLOB_AB;
  int size_padded = ((int)ceil(((double)size)/((double)blocksize)))*blocksize;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // on host (allocates & initializes to zero)
  realw* h_max1 = (realw*) calloc(num_blocks_x*num_blocks_y,sizeof(realw));
  //realw* h_max2 = (realw*) calloc(num_blocks_x*num_blocks_y,sizeof(realw));
  //realw* h_max3 = (realw*) calloc(num_blocks_x*num_blocks_y,sizeof(realw));

  // allocates memory on device
  // and initializes values to zero
  print_CUDA_error_if_any(hipMalloc((void**)&d_max1,num_blocks_x*num_blocks_y*sizeof(realw)),77001);
  print_CUDA_error_if_any(hipMemset(d_max1,0,num_blocks_x*num_blocks_y*sizeof(realw)),77002);

  //print_CUDA_error_if_any(hipMalloc((void**)&d_max2,num_blocks_x*num_blocks_y*sizeof(realw)),77001);
  //print_CUDA_error_if_any(hipMemset(d_max2,0,num_blocks_x*num_blocks_y*sizeof(realw)),77002);

  //print_CUDA_error_if_any(hipMalloc((void**)&d_max3,num_blocks_x*num_blocks_y*sizeof(realw)),77001);
  //print_CUDA_error_if_any(hipMemset(d_max3,0,num_blocks_x*num_blocks_y*sizeof(realw)),77002);

  if (*FORWARD_OR_ADJOINT == 1) {
    get_maximum_vector_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_displ,size,d_max1);
    //get_maximum_vector_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_veloc,size,d_max2);
    //get_maximum_vector_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_accel,size,d_max3);
  }else if (*FORWARD_OR_ADJOINT == 3) {
    get_maximum_vector_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_displ,size,d_max1);
    //get_maximum_vector_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_veloc,size,d_max2);
    //get_maximum_vector_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_accel,size,d_max3);
  }

  //double end_time = get_time();
  //printf("Elapsed time: %e\n",end_time-start_time);

  GPU_ERROR_CHECKING ("get_norm_elastic_from_device");

  // synchronizes
  //synchronize_cuda();
  // explicitly waits for stream to finish
  // (hipMemcpy implicitly synchronizes all other cuda operations)
  hipStreamSynchronize(mp->compute_stream);

  // copies reduction array back to CPU
  print_CUDA_error_if_any(hipMemcpy(h_max1,d_max1,num_blocks_x*num_blocks_y*sizeof(realw),
                                     hipMemcpyDeviceToHost),222);
  //print_CUDA_error_if_any(hipMemcpy(h_max2,d_max2,num_blocks_x*num_blocks_y*sizeof(realw),
  //                                   hipMemcpyDeviceToHost),222);
  //print_CUDA_error_if_any(hipMemcpy(h_max3,d_max3,num_blocks_x*num_blocks_y*sizeof(realw),
  //                                   hipMemcpyDeviceToHost),222);

  // determines max for all blocks
  max1 = h_max1[0];
  for(int i=1;i<num_blocks_x*num_blocks_y;i++) {
    if (max1 < h_max1[i]) max1 = h_max1[i];
  }
  /*
  max2 = h_max2[0];
  for(int i=1;i<num_blocks_x*num_blocks_y;i++) {
    if (max2 < h_max2[i]) max2 = h_max2[i];
  }
  */
  /*
  max3 = h_max3[0];
  for(int i=1;i<num_blocks_x*num_blocks_y;i++) {
    if (max3 < h_max3[i]) max3 = h_max3[i];
  }
  */
  res1 = sqrt(max1);
  //res2 = sqrt(max2);
  //res3 = sqrt(max3);
  // debug
  // int it = 0; // not available.
  //printf("iteration %d  Valeur max de displ : %.12f  processus %d \n",it,res1,mp->myrank);
  //printf("iteration %d  Valeur max de veloc : %.12f  processus %d \n",it,res2,mp->myrank);
  //printf("iteration %d  Valeur max de accel : %.12f  processus %d \n",it,res3,mp->myrank);

  // return result
  *norm = res1;

  // debug
  //printf("rank % d - type: %d norm: %e \n",mp->myrank,*type,*norm);

  hipFree(d_max1);
  free(h_max1);
  //hipFree(d_max2);
  //free(h_max2);
  //hipFree(d_max3);
  //free(h_max3);

  //double end_time = get_time();
  //printf("Elapsed time: %e\n",end_time-start_time);

  GPU_ERROR_CHECKING ("get_norm_elastic_from_device");
}

/* ----------------------------------------------------------------------------------------------- */

// unused ...

/* ----------------------------------------------------------------------------------------------- */

/*
extern "C"
void FC_FUNC_(get_max_accel,
              GET_MAX_ACCEL)(int* itf,int* sizef,long* Mesh_pointer) {

TRACE("get_max_accel");

  Mesh* mp = (Mesh*)(*Mesh_pointer);
  int procid;
#ifdef WITH_MPI
  MPI_Comm_rank(MPI_COMM_WORLD,&procid);
#else
  procid = 0;
#endif
  int size = *sizef;
  int it = *itf;
  realw* accel_cpy = (realw*)malloc(size*sizeof(realw));
  hipMemcpy(accel_cpy,mp->d_accel,size*sizeof(realw),hipMemcpyDeviceToHost);
  realw maxval=0;
  for(int i=0;i<size;++i) {
    maxval = MAX(maxval,accel_cpy[i]);
  }
  printf("%d/%d: max=%e\n",it,procid,maxval);
  free(accel_cpy);
}
*/


/* ----------------------------------------------------------------------------------------------- */
//daniel: helper function
/*
 __global__ void check_phase_ispec_kernel(int num_phase_ispec,
 int* phase_ispec,
 int NSPEC_AB,
 int* ier) {

 int i,ispec,iphase,count0,count1;
 *ier = 0;

 for(iphase=0; iphase < 2; iphase++){
 count0 = 0;
 count1 = 0;

 for(i=0; i < num_phase_ispec; i++){
 ispec = phase_ispec[iphase*num_phase_ispec + i] - 1;
 if (ispec < -1 || ispec >= NSPEC_AB) {
 printf("Error in d_phase_ispec_inner_elastic %d %d\n",i,ispec);
 *ier = 1;
 return;
 }
 if (ispec >= 0) { count0++;}
 if (ispec < 0) { count1++;}
 }

 printf("check_phase_ispec done: phase %d, count = %d %d \n",iphase,count0,count1);

 }
 }

 void check_phase_ispec(long* Mesh_pointer,int type){

 Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

 printf("check phase_ispec for type=%d\n",type);

 dim3 grid(1,1);
 dim3 threads(1,1,1);

 int* h_debug = (int*) calloc(1,sizeof(int));
 int* d_debug;
 hipMalloc((void**)&d_debug,sizeof(int));

 if (type == 1) {
 check_phase_ispec_kernel<<<grid,threads>>>(mp->num_phase_ispec_elastic,
 mp->d_phase_ispec_inner_elastic,
 mp->NSPEC_AB,
 d_debug);
 }else if (type == 2) {
 check_phase_ispec_kernel<<<grid,threads>>>(mp->num_phase_ispec_acoustic,
 mp->d_phase_ispec_inner_acoustic,
 mp->NSPEC_AB,
 d_debug);
 }

 hipMemcpy(h_debug,d_debug,1*sizeof(int),hipMemcpyDeviceToHost);
 hipFree(d_debug);
 if (*h_debug != 0) {printf("error for type=%d\n",type); exit(1);}
 free(h_debug);
 fflush(stdout);

 GPU_ERROR_CHECKING ("check_phase_ispec");
 }
*/

/* ----------------------------------------------------------------------------------------------- */
//daniel: helper function
/*
 __global__ void check_ispec_is_kernel(int NSPEC_AB,
 int* ispec_is,
 int* ier) {

 int ispec,count0,count1;

 *ier = 0;
 count0 = 0;
 count1 = 0;
 for(ispec=0; ispec < NSPEC_AB; ispec++){
 if (ispec_is[ispec] < -1 || ispec_is[ispec] > 1) {
 printf("Error in ispec_is %d %d\n",ispec,ispec_is[ispec]);
 *ier = 1;
 return;
 //exit(1);
 }
 if (ispec_is[ispec] == 0) {count0++;}
 if (ispec_is[ispec] != 0) {count1++;}
 }
 printf("check_ispec_is done: count = %d %d\n",count0,count1);
 }

 void check_ispec_is(long* Mesh_pointer,int type){

 Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

 printf("check ispec_is for type=%d\n",type);

 dim3 grid(1,1);
 dim3 threads(1,1,1);

 int* h_debug = (int*) calloc(1,sizeof(int));
 int* d_debug;
 hipMalloc((void**)&d_debug,sizeof(int));

 if (type == 0) {
 check_ispec_is_kernel<<<grid,threads>>>(mp->NSPEC_AB,
 mp->d_ispec_is_inner,
 d_debug);
 }else if (type == 1) {
 check_ispec_is_kernel<<<grid,threads>>>(mp->NSPEC_AB,
 mp->d_ispec_is_elastic,
 d_debug);
 }else if (type == 2) {
 check_ispec_is_kernel<<<grid,threads>>>(mp->NSPEC_AB,
 mp->d_ispec_is_acoustic,
 d_debug);
 }

 hipMemcpy(h_debug,d_debug,1*sizeof(int),hipMemcpyDeviceToHost);
 hipFree(d_debug);
 if (*h_debug != 0) {printf("error for type=%d\n",type); exit(1);}
 free(h_debug);
 fflush(stdout);

 GPU_ERROR_CHECKING ("check_ispec_is");
 }
*/
/* ----------------------------------------------------------------------------------------------- */
//daniel: helper function
/*
 __global__ void check_array_ispec_kernel(int num_array_ispec,
 int* array_ispec,
 int NSPEC_AB,
 int* ier) {

 int i,ispec,count0,count1;

 *ier = 0;
 count0 = 0;
 count1 = 0;

 for(i=0; i < num_array_ispec; i++){
 ispec = array_ispec[i] - 1;
 if (ispec < -1 || ispec >= NSPEC_AB) {
 printf("Error in d_array_ispec %d %d\n",i,ispec);
 *ier = 1;
 return;
 }
 if (ispec >= 0) { count0++;}
 if (ispec < 0) { count1++;}
 }

 printf("check_array_ispec done: count = %d %d \n",count0,count1);
 }

 void check_array_ispec(long* Mesh_pointer,int type){

 Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

 printf("check array_ispec for type=%d\n",type);

 dim3 grid(1,1);
 dim3 threads(1,1,1);

 int* h_debug = (int*) calloc(1,sizeof(int));
 int* d_debug;
 hipMalloc((void**)&d_debug,sizeof(int));

 if (type == 1) {
 check_array_ispec_kernel<<<grid,threads>>>(mp->d_num_abs_boundary_faces,
 mp->d_abs_boundary_ispec,
 mp->NSPEC_AB,
 d_debug);
 }

 hipMemcpy(h_debug,d_debug,1*sizeof(int),hipMemcpyDeviceToHost);
 hipFree(d_debug);
 if (*h_debug != 0) {printf("error for type=%d\n",type); exit(1);}
 free(h_debug);
 fflush(stdout);

 GPU_ERROR_CHECKING ("check_array_ispec");
 }
*/

/* ----------------------------------------------------------------------------------------------- */

// Check functions

/* ----------------------------------------------------------------------------------------------- */

//max: helper functions

/*
extern "C"
void FC_FUNC_(check_max_norm_displ_gpu,
              CHECK_MAX_NORM_DISPL_GPU)(int* size, realw* displ,long* Mesh_pointer,int* announceID) {

  TRACE("check_max_norm_displ_gpu");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  hipMemcpy(displ, mp->d_displ,*size*sizeof(realw),hipMemcpyDeviceToHost);
  realw maxnorm=0;

  for(int i=0;i<*size;i++) {
    maxnorm = MAX(maxnorm,fabsf(displ[i]));
  }
  printf("%d: maxnorm of forward displ = %e\n",*announceID,maxnorm);
}
*/

/* ----------------------------------------------------------------------------------------------- */
/*
extern "C"
void FC_FUNC_(check_max_norm_vector,
              CHECK_MAX_NORM_VECTOR)(int* size, realw* vector1, int* announceID) {

TRACE("check_max_norm_vector");

  int procid;
#ifdef WITH_MPI
  MPI_Comm_rank(MPI_COMM_WORLD,&procid);
#else
  procid = 0;
#endif
  realw maxnorm=0;
  int maxloc;
  for(int i=0;i<*size;i++) {
    if (maxnorm<fabsf(vector1[i])) {
      maxnorm = vector1[i];
      maxloc = i;
    }
  }
  printf("%d:maxnorm of vector %d [%d] = %e\n",procid,*announceID,maxloc,maxnorm);
}
*/

/* ----------------------------------------------------------------------------------------------- */

/*
extern "C"
void FC_FUNC_(check_max_norm_displ,
              CHECK_MAX_NORM_DISPL)(int* size, realw* displ, int* announceID) {

TRACE("check_max_norm_displ");

  realw maxnorm=0;

  for(int i=0;i<*size;i++) {
    maxnorm = MAX(maxnorm,fabsf(displ[i]));
  }
  printf("%d: maxnorm of forward displ = %e\n",*announceID,maxnorm);
}
*/

/* ----------------------------------------------------------------------------------------------- */

/*
extern "C"
void FC_FUNC_(check_max_norm_b_displ_gpu,
              CHECK_MAX_NORM_B_DISPL_GPU)(int* size, realw* b_displ,long* Mesh_pointer,int* announceID) {

  TRACE("check_max_norm_b_displ_gpu");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  realw* b_accel = (realw*)malloc(*size*sizeof(realw));

  hipMemcpy(b_displ, mp->d_b_displ,*size*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(b_accel, mp->d_b_accel,*size*sizeof(realw),hipMemcpyDeviceToHost);

  realw maxnorm=0;
  realw maxnorm_accel=0;

  for(int i=0;i<*size;i++) {
    maxnorm = MAX(maxnorm,fabsf(b_displ[i]));
    maxnorm_accel = MAX(maxnorm,fabsf(b_accel[i]));
  }
  free(b_accel);
  printf("%d: maxnorm of backward displ = %e\n",*announceID,maxnorm);
  printf("%d: maxnorm of backward accel = %e\n",*announceID,maxnorm_accel);
}
*/

/* ----------------------------------------------------------------------------------------------- */

/*
extern "C"
void FC_FUNC_(check_max_norm_b_accel_gpu,
              CHECK_MAX_NORM_B_ACCEL_GPU)(int* size, realw* b_accel,long* Mesh_pointer,int* announceID) {

  TRACE("check_max_norm_b_accel_gpu");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  hipMemcpy(b_accel, mp->d_b_accel,*size*sizeof(realw),hipMemcpyDeviceToHost);

  realw maxnorm=0;

  for(int i=0;i<*size;i++) {
    maxnorm = MAX(maxnorm,fabsf(b_accel[i]));
  }
  printf("%d: maxnorm of backward accel = %e\n",*announceID,maxnorm);
}
*/

/* ----------------------------------------------------------------------------------------------- */

/*
extern "C"
void FC_FUNC_(check_max_norm_b_veloc_gpu,
              CHECK_MAX_NORM_B_VELOC_GPU)(int* size, realw* b_veloc,long* Mesh_pointer,int* announceID) {

  TRACE("check_max_norm_b_veloc_gpu");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  hipMemcpy(b_veloc, mp->d_b_veloc,*size*sizeof(realw),hipMemcpyDeviceToHost);

  realw maxnorm=0;

  for(int i=0;i<*size;i++) {
    maxnorm = MAX(maxnorm,fabsf(b_veloc[i]));
  }
  printf("%d: maxnorm of backward veloc = %e\n",*announceID,maxnorm);
}
*/

/* ----------------------------------------------------------------------------------------------- */

/*
extern "C"
void FC_FUNC_(check_max_norm_b_displ,
              CHECK_MAX_NORM_B_DISPL)(int* size, realw* b_displ,int* announceID) {

TRACE("check_max_norm_b_displ");

  realw maxnorm=0;

  for(int i=0;i<*size;i++) {
    maxnorm = MAX(maxnorm,fabsf(b_displ[i]));
  }
  printf("%d:maxnorm of backward displ = %e\n",*announceID,maxnorm);
}
*/

/* ----------------------------------------------------------------------------------------------- */

/*
extern "C"
void FC_FUNC_(check_max_norm_b_accel,
              CHECK_MAX_NORM_B_ACCEL)(int* size, realw* b_accel,int* announceID) {

TRACE("check_max_norm_b_accel");

  realw maxnorm=0;

  for(int i=0;i<*size;i++) {
    maxnorm = MAX(maxnorm,fabsf(b_accel[i]));
  }
  printf("%d:maxnorm of backward accel = %e\n",*announceID,maxnorm);
}
*/

/* ----------------------------------------------------------------------------------------------- */

/*
extern "C"
void FC_FUNC_(check_error_vectors,
              CHECK_ERROR_VECTORS)(int* sizef, realw* vector1,realw* vector2) {

TRACE("check_error_vectors");

  int size = *sizef;

  double diff2 = 0;
  double sum = 0;
  double temp;
  double maxerr=0;
  int maxerrorloc;

  for(int i=0;i<size;++i) {
    temp = vector1[i]-vector2[i];
    diff2 += temp*temp;
    sum += vector1[i]*vector1[i];
    if (maxerr < fabsf(temp)) {
      maxerr = abs(temp);
      maxerrorloc = i;
    }
  }

  printf("rel error = %f, maxerr = %e @ %d\n",diff2/sum,maxerr,maxerrorloc);
  int myrank;
#ifdef WITH_MPI
  MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
#else
  myrank = 0;
#endif
  if (myrank == 0) {
    for(int i=maxerrorloc;i>maxerrorloc-5;i--) {
      printf("[%d]: %e vs. %e\n",i,vector1[i],vector2[i]);
    }
  }

}
*/

